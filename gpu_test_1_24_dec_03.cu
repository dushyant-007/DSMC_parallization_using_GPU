
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <stdio.h>
#include <fstream>
#include <cstdlib>
#include <iomanip>
#include <bits/stdc++.h>

#include <string>
#include <cmath>

using namespace std ;
fstream file_9 ;
fstream file_4 ;
fstream file_3 ;
fstream file_18 ;
fstream file_13;

void ALLOCATE_GAS();
void HARD_SPHERE();
void ARGON();
void IDEAL_NITROGEN();
void REAL_OXYGEN();
void IDEAL_AIR();
void REMOVE_MOL(int &) ;
void SAMPLE_FLOW() ;
void OUTPUT_RESULTS() ;
void SETXT() ;
void cuda_collisions(int) ;
void COLLISIONS() ;
void DISSOCIATION() ;
void EXTEND_MNM(double ) ;
void MOLECULES_ENTER_1D() ;
void INDEX_MOLS() ;
void AIFX(double &,double &, double &, double &, double &, double &, double &, double &) ;
void RBC(double &, double &, double &,double &, double &,double &) ;
void REAL_AIR();
void HELIUM_ARGON_XENON();
void OXYGEN_HYDROGEN();
void MOLECULES_MOVE_1D() ;
void DERIVED_GAS_DATA() ;
void INITIALISE_SAMPLES();
void SET_INITIAL_STATE_1D();
void FIND_CELL_MB_1D(double& , int& ,int& , double&) ;
void FIND_CELL_1D(double &,int &,int &);
void RVELC(double &,double &,double &);
void SROT(int &,double &,double &);
void SVIB(int &,double &,int &, int&);
void SELE(int &,double &,double &);
void CQAX(double&,double &,double&);
void LBS(double,double,double&);
void ENERGY(int ,double &);
void REFLECT_1D(int&,int,double&);
void READ_DATA();


//modules calc
int NVER ,MVER,IMEG,NREL,MOLSC,ISF,ISAD,ISECS,IGS,IREM,NNC,IMTS,ERROR,NLINE,ICLASS, 
           NCLASS,NMCC,NMI,NMP,ICN ;
double FTIME,TLIM,PI,SPI,DPI,BOLTZ,FNUM,DTM,TREF,TSAMP,TOUT,SAMPRAT,OUTRAT,RANF,TOTCOLI,TOTMOVI,TENERGY,
                 DTSAMP,DTOUT,TPOUT,FRACSAM,TOTMOV,TOTCOL,ENTMASS,ENTREM,CPDTM,TPDTM,AVOG,TNORM,FNUMF;
double *VNMAX , *TDISS , *TRECOMB , *ALOSS , *EME , *AJM , *COLL_TOTCOL;
double *TCOL ; 

//module molecs
int *IPCELL,*IPSP,*ICREF,*IPCP ;
int *IPVIB ;
int NM , MNM ;
double *PX , *PV ;
double *PTIM , *PROT , *PELE  ;

//module gas
double RMAS,CXSS,RGFS,VMPM,FDEN,FPR,FMA,FPM,CTM ;
double FND[3],FTMP[3],FVTMP[3],VFX[3],VFY[3],TSURF[3],FSPEC[3],VSURF[3] ;
double *ERS,*CR,*TNEX,*PSF,*SLER,*FP ;
double *FSP,*SP,*SPR,*SPV,*VMP ;
double *SPM,*SPVM,*ENTR,*QELC,*SPRT ;
double *SPEX,*SPRC,*SPRP ;
double *SPREX ;

int MSP,MMVM,MMRM,MNSR,IGAS,MMEX,MEX,MELE,MVIBL ;
int *ISP,*ISPV,*NELL ;
int *ISPR,*LIS,*LRS,*ISRCD,*ISPRC,*ISPRK,*TREACG,*TREACL,*NSPEX,*NSLEV ;
int *ISPVM,*NEX ;
int *ISPEX ;

//module OUTPUT
int NSAMP,NMISAMP,NOUT,NDISSOC,NRECOMB,NTSAMP ;
int *NDISSL ;
double TISAMP , XVELS , YVELS , AVDTM ;
double *COLLS,*WCOLLS,*CLSEP,*SREAC,*STEMP,*TRANSTEMP,*ROTTEMP,*VIBTEMP,*ELTEMP ;
double *VAR,*VARS,*CSSS,*SUMVIB ;
double *CS,*VARSP,*VIBFRAC ;
double *CSS ;


//module GEOM_1D
int NCELLS,NCCELLS,NCIS,NDIV,MDIV,ILEVEL,IFX,JFX,IVB,IWF ;
int *ITYPE ;
int *ICELL ;
int *ICCELL , *JDIV ;
double DDIV,XS,VELOB,WFM,AWF,FREM,XREM ;
double *XB ;
double *CELL , *CCELL ;


void i_allocate( int x , int y , int *&b)
{
        
        
        hipMallocManaged(&b , (x*y+6)*sizeof(int) ) ;
        //b = new int[x*y+6] ;
        hipMemset(b, 0 , (x*y+6)*sizeof(int)) ;
        b[0] = x*y ;
        b[b[0]+1] = x ;
        b[b[0]+2] = y ;
}

void i_allocate( int x , int *&b )
{
      
        hipMallocManaged(&b , (x+6)*sizeof(int) ) ;
        //b = new int[x+6] ;
        hipMemset(b, 0 , (x+1)*sizeof(int)) ;
        b[0] = x ;
        b[b[0]+1]=x ;
}

void i_allocate( int x , int y , int z , int *&b )
{
        
        hipMallocManaged(&b , (x*y*z+6)*sizeof(int) ) ;    
        //b= new int[x*y*z+6] ;
        hipMemset(b, 0 , (x*y*z+6)*sizeof(int)) ;
        b[0] = x*y*z ;
        b[b[0]+1] = x ;
        b[b[0]+2] = y ;
        b[b[0]+3] = z ;
}

void i_allocate( int x , int y , int z , int w , int *&b  )
{
        
        hipMallocManaged(&b , (x*y*z*w+6)*sizeof(int) ) ;
        //b = new int[x*y*z*w+6] ;
        hipMemset(b, 0 , (x*y*z*w+6)*sizeof(int)) ;
        b[0] = x*y*z*w ;
        b[b[0]+1] = x ;
        b[b[0]+2] = y ;
        b[b[0]+3] = z ;
        b[b[0]+4] = w ;
}

void i_allocate( int x , int y , int z , int w ,int u , int *&b )
{
        
        hipMallocManaged(&b , (x*y*z*w*u+6)*sizeof(int) ) ;
        //b = new int[x*y*z*w*u+6] ;
        hipMemset(b, 0 , (x*y*z*w*u+6)*sizeof(int)) ;
        b[0] = x*y*z*w*u ;
        b[b[0]+1] = x ;
        b[b[0]+2] = y ;
        b[b[0]+3] = z ;
        b[b[0]+4] = w ;
        b[b[0]+5] = u ;
}

//__device__ __host__ 


//__device__ __host__

int& get(int* &b , int x)
{
    return b[x] ;
} 
int& get(int *&b , int x , int y)
{
        int r1 = b[b[0]+2] ;
        return b[(x-1)*r1 + y] ;
}

//__device__ __host__ 
int& get(int *&b , int x , int y , int z)
{
        int r1 = b[b[0]+2] ;
        int r2 = b[b[0]+3] ;
        return b[((x-1)*r1 + y-1)*r2+z] ;
}

//__device__ __host__ 
int& get(int *&b , int x , int y , int z , int w)
{
        int r1 = b[b[0]+2] ;
        int r2 = b[b[0]+3] ;
        int r3 = b[b[0]+4] ;
        return b[(((x-1)*r1 + y-1)*r2+z-1)*r3+w] ;
}

//__device__ __host__ 
int& get(int *&b , int x , int y , int z , int w , int u)
{
        int r1 = b[b[0]+2] ;
        int r2 = b[b[0]+3] ;
        int r3 = b[b[0]+4] ;
        int r4 = b[b[0]+5] ;
        return b[((((x-1)*r1 + y-1)*r2+z-1)*r3+w-1)*r4+u] ;
}


void d_allocate( int x , int y , double *&b )
{
        
        hipMallocManaged(&b , (x*y+6)*sizeof(double) ) ;
        //b = new double[x*y+6] ;
        hipMemset(b, 0 , (x*y+6)*sizeof(double)) ;
        b[0] = (double)x*y ;
        b[(int)b[0]+1] = x ;
        b[(int)b[0]+2] = y ;
}

void d_allocate( int x  , double *&b )
{
       
        hipMallocManaged(&b , (x+1)*sizeof(double) ) ;
        //b = new double[x+6] ;
        hipMemset(b, 0 , (x+1)*sizeof(double)) ;
        b[0] = x ;
        b[(int)b[0]+1] = x ;
}

void d_allocate( int x , int y , int z , double *&b )
{
       
        hipMallocManaged(&b , (x*y*z+6)*sizeof(double) ) ;
        //b = new double[x*y*z+6] ;
        hipMemset(b, 0 , (x*y*z+6)*sizeof(double)) ;
        b[0] = x*y*z ;
        b[(int)b[0]+1] = x ;
        b[(int)b[0]+2] = y ;
        b[(int)b[0]+3] = z ;
}

void d_allocate( int x , int y , int z , int w ,double *&b )
{
      
        hipMallocManaged(&b , (x*y*z*w+6)*sizeof(double) ) ;
        //b = new double[x*y*z*w+6] ;
        hipMemset(b, 0 , (x*y*z*w+6)*sizeof(double)) ;
        b[0] = x*y*z*w ;
        b[(int)b[0]+1] = x ;
        b[(int)b[0]+2] = y ;
        b[(int)b[0]+3] = z ;
        b[(int)b[0]+4] = w ;
}

void d_allocate(int x , int y , int z , int w ,int u , double *&b )
{
        hipMallocManaged(&b , (x*y*z*w*u+6)*sizeof(double) ) ;
        //b = new double[x*y*z*w*u+6] ;
        hipMemset(b, 0 , (x*y*z*w*u+6)*sizeof(double)) ;
        b[0] = x*y*z*w*u ;
        b[(int)b[0]+1] = x ;
        b[(int)b[0]+2] = y ;
        b[(int)b[0]+3] = z ;
        b[(int)b[0]+4] = w ;
        b[(int)b[0]+5] = u ;
}

//__device__ __host__ 
double& get(double *&b , int x)
{
        return b[x] ;
}

//__device__ __host__ 
double& get(double *&b , int x , int y)
{
        int r1 = b[(int)b[0]+2] ;
        return b[(x-1)*r1 + y] ;
}

//__device__ __host__ 
double& get(double *&b , int x , int y , int z)
{
        int r1 = b[(int)b[0]+2] ;
        int r2 = b[(int)b[0]+3] ;
        return b[((x-1)*r1 + y-1)*r2+z] ;
}

//__device__ __host__ 
double& get(double *&b , int x , int y , int z , int w)
{
        int r1 = b[(int)b[0]+2] ;
        int r2 = b[(int)b[0]+3] ;
        int r3 = b[(int)b[0]+4] ;
        return b[(((x-1)*r1 + y-1)*r2+z-1)*r3+w] ;
}

//__device__ __host__ 
double& get(double *&b , int x , int y , int z , int w , int u)
{
        int r1 = b[(int)b[0]+2] ;
        int r2 = b[(int)b[0]+3] ;
        int r3 = b[(int)b[0]+4] ;
        int r4 = b[(int)b[0]+5] ;
        return b[((((x-1)*r1 + y-1)*r2+z-1)*r3+w-1)*r4+u] ;
}

         


void READ_DATA()
{
    int NVERD , MVERD , N , K ;
    d_allocate(2 , XB ) ;
    i_allocate(2 , ITYPE ) ;
    i_allocate(201 , NDISSL ) ;
    cout << "ICLASS = "<< ICLASS << endl ;
    if(ICLASS==0)
        {
            cout << "Reading the data file DS0D.DAT\n";
            file_4.open("DS0D.DAT" , ios::in) ;
            file_3.open("DS0D.TXT" , ios::out) ;
            file_3 << "Data summary for program DSMC" << endl ;
            cout  << "DS0D.TXT opened \n" ;
        }

    if(ICLASS==1)
    {
        cout<<"Reading the data file DS1D.DAT"<<endl;
        file_4.open("DS1D.DAT", ios::in);
        file_3.open("DS1D.TXT", ios::out );
        file_3<<"Data summary for program DSMC"<<endl;
        // OPEN (4,FILE='DS1D.DAT')
        // OPEN (3,FILE='DS1D.TXT')
        // WRITE (3,*) 'Data summary for program DSMC'
    }

    file_4 >> IMEG ;
    file_3 << "The approximate number of megabytes for the calculation is" << IMEG  << endl ;
    file_4 >> IGAS ;
    file_3<< IGAS << endl;

    if(IGAS==1)
    {
        file_3<<" Hard sphere gas "<<endl;
        // WRITE (3,*) 'Hard sphere gas'
        HARD_SPHERE();
    }
    else if(IGAS==2)
    {
        file_3<<"Argon "<<endl;
        // WRITE (3,*) 'Argon'
        ARGON();
    }
    else if(IGAS==3)
    {
        file_3<<"Ideal nitrogen"<<endl;
        // WRITE (3,*) 'Ideal nitrogen'
        IDEAL_NITROGEN();
        
    }
    else if(IGAS==4)
    {
        file_3<<"Real oxygen "<<endl;
        // WRITE (3,*) 'Real oxygen'
        REAL_OXYGEN();
    }
    else if(IGAS==5)
    {
        file_3<<"Ideal air "<<endl;
        // TE (3,*) 'Ideal air'
        IDEAL_AIR();
    }
    else if(IGAS==6)
    {
        file_3<<"Real air @ 7.5 km/s "<<endl;
        // RITE (3,*) 'Real air @ 7.5 km/s'
        REAL_AIR();
    }
    else if(IGAS==7)
    {
        file_3<<"Helium-argon-xenon mixture "<<endl;
        // WRITE (3,*) 'Helium-argon-xenon mixture'
        HELIUM_ARGON_XENON();
    }
    else if(IGAS==8)
    {
        file_3<<"Oxygen-hydrogen "<<endl;
        // WRRITE (3,*) 'Oxygen-hydrogen'
        OXYGEN_HYDROGEN();
    }

  
    

    
    file_3<<"The gas properties are:- "<<endl;
    file_4>>FND[1];
    file_3<<"The stream number density is "<<FND[1]<<endl;
    file_4>>FTMP[1];
    file_3<<"The stream temperature is "<<FTMP[1]<<endl;

    

    if(MMVM>0)
    {
      
        file_4>>FVTMP[1];
        file_3<<"The stream vibrational and electronic temperature is "<<FVTMP[1]<<endl;
        // READ (4,*) FVTMP(1) //FVTMP;
        // WRITE (3,*) '    The stream vibrational and electronic temperature is',FVTMP(1) //FVTMP[1]
    }

    if(ICLASS==1)
    {
        file_4>>VFX[1];
        file_3<<"The stream velocity in the x direction is "<<VFX[1]<<endl;
        file_4>>VFY[1];
        file_3<<"The stream velocity in the y direction is "<<VFY[1]<<endl;
        // WRITE (3,*) '    The stream velocity in the y direction is',VFY(1) ////VFY[1]
    }

    if(MSP>1)
    {
        for(N=1;N<=MSP;N++)
        {   
            int in ;
            file_4 >> in;
            get(FSP ,N,1) =  in ;
            file_3 << " The fraction of species " << N <<" is "<<get(FSP ,N,1)<<endl;
            // WRITE (3,*) '    The fraction of species',N,' is',FSP(N,1) //get(FSP ,N,1]
        }
    }
    else get(FSP ,1,1) = 1 ;

    if(ICLASS==0){
        //       !--a homogeneous gas case is calculated as a one-dimensional flow with a single sampling cell
        // !--set the items that are required in the DS1D.DAT specification
        IFX=0;
        JFX=1;
        XB[1]=0.e00;
        XB[2]=0.0001e00*1.e25/FND[1];
        ITYPE[1]=1;
        ITYPE[2]=1;
        VFX[1]=0.e00;
        IGS=1;
        ISECS=0;
        IREM=0;
        MOLSC=10000*IMEG; //a single sampling cell
    }
    else if(ICLASS==1)
    {
        file_4>>IFX;
        
        if(IFX==0)
            file_3<<"Plane Flow"<<endl;
        
        if(IFX==1)
            file_3<<"Cylindrical flow"<<endl;
        
        if(IFX==2)
            file_3<<"Spherical flow"<<endl;
        
        JFX=IFX+1;
        file_4>>XB[1];
        
        file_3<<"The minimum x coordinate is "<<XB[1]<<endl;
        file_4>>ITYPE[1];
        if(ITYPE[1]==0)
            file_3<<"The minimum x coordinate is a stream boundary"<<endl;
        if(ITYPE[1]==1)
            file_3<<"The minimum x coordinate is a plane of symmetry"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is a plane of symmetry'
        if(ITYPE[1]==2)
            file_3<<"The minimum x coordinate is a solid surface"<<endl;
        if(ITYPE[1]==3)
            file_3<<"The minimum x coordinate is a vacuum"<<endl;
        if(ITYPE[1]==4)
            file_3<<"The minimum x coordinate is an axis or center"<<endl;
        if(ITYPE[1]==2)
        {
            file_3<<"The minimum x boundary is a surface with the following properties"<<endl;
            file_4>>TSURF[1];
            file_3<<"The temperature of the surface is "<<TSURF[1]<<endl;
            file_4>>FSPEC[1];
            file_3<<"The fraction of specular reflection is "<<FSPEC[1]<<endl;
            file_4>>VSURF[1];
            file_3<<"The velocity in the y direction of this surface is "<<VSURF[1] << endl ;
        }
        file_4>>XB[2];
        file_3<<"The maximum x coordinate is "<<XB[2]<<endl;
        file_4>>ITYPE[2];
        if(ITYPE[2]==0)
            file_3<<"The mmaximum  x coordinate is a stream boundary"<<endl;
        if(ITYPE[2]==1)
            file_3<<"The maximum x coordinate is a plane of symmetry"<<endl;
        if(ITYPE[2]==2)
            file_3<<"The maximum  x coordinate is a solid surface"<<endl;
        if(ITYPE[2]==3)
            file_3<<"The maximum  x coordinate is a vacuum"<<endl;
        ICN=0;
        if(ITYPE[2]==4)
        {
            file_3<<"The maximum x coordinate is a stream boundary with a fixed number of simulated molecules"<<endl;
            // WRITE (3,*) 'The maximum x coordinate is a stream boundary with a fixed number of simulated molecules'
            if(MSP==1)
                ICN=1;
        }
        if(ITYPE[2]==2)
        {
            file_3<<"The maximum  x boundary is a surface with the following properties"<<endl;
            file_4>>TSURF[1];
            file_3<<"The temperature of the surface is "<<TSURF[1]<<endl;
            file_4>>FSPEC[1];
            file_3<<"The fraction of specular reflection is "<<FSPEC[1]<<endl;
            file_4>>VSURF[1];
            file_3<<"The velocity in the y direction of this surface is "<<VSURF[1]<<endl;
        }
        if(IFX>0)
        {
            file_4>>IWF;
            if(IWF==0)
                file_3<<"There are no radial weighting factors"<<endl;
            if(IWF==1)
                file_3<<"There are radial weighting factors"<<endl;
            if(IWF==1)
            {
                file_4>>WFM;
                file_3<<"The maximum value of the weighting factor is  "<<WFM<<endl;
                WFM=(WFM-1)/XB[2];
            }
        }
        file_4>>IGS;
        // READ (4,*) IGS //IGS
        if(IGS==0)
            file_3<<"The flowfield is initially a vacuum "<<endl;
        // WRITE (3,*) 'The flowfield is initially a vacuum'
        if(IGS==1)
            file_3<<"The flowfield is initially the stream(s) or reference gas"<<endl;
        // WRITE (3,*) 'The flowfield is initially the stream(s) or reference gas'
        file_4>>ISECS;
        // READ (4,*) ISECS //ISECS
        if(ISECS==0)
            file_3<<"There is no secondary stream initially at x > 0"<<endl;
        // WRITE (3,*) 'There is no secondary stream initially at x > 0'
        if(ISECS==1 && IFX==0)
            file_3<<"There is a secondary stream applied initially at x = 0 (XB(2) must be > 0)"<<endl;
        // WRITE (3,*) 'There is a secondary stream applied initially at x = 0 (XB(2) must be > 0)'
        if(ISECS==1 && IFX>0)
        {
            if(IWF==1)
            {
                file_3<<"There cannot be a secondary stream when weighting factors are present"<<endl;
                // WRITE (3,*) 'There cannot be a secondary stream when weighting factors are present'
                return;//STOP//dout
            }
            file_3<<"There is a secondary stream"<<endl;
            // WRITE (3,*) 'There is a secondary stream'
            file_4>>XS;
            // READ (4,*) XS //XS
            file_3<<"The secondary stream boundary is at r= "<<XS<<endl;
            // WRITE (3,*) 'The secondary stream boundary is at r=',XS //XS
        }
        if(ISECS==1)
        {
            file_3<<"The secondary stream (at x>0 or X>XS) properties are:-"<<endl;
            file_4>>FND[2];
            file_3<<"The stream number density is "<<FND[2]<<endl;
            file_4>>FTMP[2];
            file_3<<"The stream temperature is "<<FTMP[2]<<endl;
            // WRITE (3,*) 'The secondary stream (at x>0 or X>XS) properties are:-'
            // READ (4,*) FND(2) //FND
            // WRITE (3,*) '    The stream number density is',FND(2) //FND
            // READ (4,*) FTMP(2) //FTMP
            // WRITE (3,*) '    The stream temperature is',FTMP(2) //FTMP
            if(MMVM>0)
            {
                file_4>>FVTMP[2];
                file_3<<"The stream vibrational and electronic temperature is "<<FVTMP[2]<<endl;
                // READ (4,*) FVTMP(2) //FVTMP[2]
                // WRITE (3,*) '    The stream vibrational and electronic temperature is',FVTMP(2) //FVTMP[2]
            }
            file_4>>VFX[2];
            file_3<<"The stream velocity in the x direction is "<<VFX[2]<<endl;
            file_4>>VFY[2];
            file_3<<"The stream velocity in the y direction is "<<VFY[2]<<endl;
            // READ (4,*) VFX(2) //VFX
            // WRITE (3,*) '    The stream velocity in the x direction is',VFX(2) //VFX
            // READ (4,*) VFY(2) //VFY
            // WRITE (3,*) '    The stream velocity in the y direction is',VFY(2) //VFY
            if(MSP>1)
            {
                for(N=1;N<=MSP;N++)
                {
                    int in ;
                    file_4>>in;
                    get(FSP ,N,2 )= in ;
                    file_3<<"The fraction of species "<<N<<" is "<<get(FSP ,N,2)<<endl;
                    // READ (4,*) FSP(N,2) //FSP
                    // WRITE (3,*) '    The fraction of species',N,' is',FSP(N,2) //FSP
                }
            }
            else
            {
                get(FSP ,1,2)=1;
            }
        }
        if(IFX==0 && ITYPE[1]==0)
        {
            file_4>>IREM;
            // READ (4,*) IREM //IREM
            if(IREM==0)
            {
                file_3<<"There is no molecule removal"<<endl;
                // WRITE (3,*) 'There is no molecule removal'
                XREM=XB[1]-1.e00;
                FREM=0.e00;
            }
            else if(IREM==1)
            {
                file_4>>XREM;
                file_3<<"There is full removal of the entering (at XB(1)) molecules between "<<XREM<<" and "<<XB[2]<<endl;
                // READ (4,*) XREM //XREM
                // WRITE (3,*) ' There is full removal of the entering (at XB(1)) molecules between',XREM,' and',XB(2) //XREM ,XB[2]
                FREM=1.e00;
            }
            else if(IREM==2)
            {
                file_3<<"Molecule removal is specified whenever the program is restarted"<<endl;
                // WRITE (3,*) ' Molecule removal is specified whenever the program is restarted'
                XREM=XB[1]-1.e00;
                FREM=0.e00;
            }
            else
            {
                XREM=XB[1]-1.e00;
                FREM=0.e00;
            }
        }
        //IVB=0;
        //VELOB=0.e00;
        if(ITYPE[2]==1)
        {
            file_4>>IVB;
            // READ (4,*) IVB
            if(IVB==0)
                file_3<<"The outer boundary is stationary"<<endl;
            // WRITE (3,*) ' The outer boundary is stationary'
            if(IVB==1)
            {
                file_3<<"The outer boundary moves with a constant speed"<<endl;
                file_4>>VELOB;
                file_3<<" The speed of the outer boundary is "<<VELOB<<endl;
                // WRITE (3,*) ' The outer boundary moves with a constant speed'
                // READ (4,*) VELOB //VELOB
                // WRITE (3,*) ' The speed of the outer boundary is',VELOB //VELOB
            }
        }
        file_4>>MOLSC;
        file_3<<"The desired number of molecules in a sampling cell is "<<MOLSC<<endl;
        // READ (4,*) MOLSC //MOLSC
        // WRITE (3,*) 'The desired number of molecules in a sampling cell is',MOLSC ////MOLSC
    }
    //set the speed of the outer boundary
    file_3.close();
    file_4.close();
    // CLOSE (3)
    // CLOSE (4)
    // set the stream at the maximum x boundary if there is no secondary stream
    if(ISECS==0 && ITYPE[2]==0)
    {
        FND[2]=FND[1];
        FTMP[2]=FTMP[1];
        if(MMVM>0)
            FVTMP[2]=FVTMP[1];
        VFX[2]=VFX[1];
        if(MSP>1)
        {
            for(N=1;N<=MSP;N++)
            {
                get(FSP ,N,2)=get(FSP ,N,1);
            }
        }
        else
            get(FSP ,1,2)=1;
    }

    cout << "READ_DATA functin finished . . . " ;
}   

void HARD_SPHERE()
{
    ////GAS gas;
    ////CALC calc;
    cout<<"Reading HARD_SPHERE Data"<<endl;
    MSP=1;
    MMRM=0;
    MMVM=0;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=1;
    MVIBL=0;
    
    ALLOCATE_GAS();
    
    get(SP ,1,1)=4.0e-10;    //reference diameter
    get(SP ,2,1)=273.0;       //reference temperature
    get(SP ,3,1)=0.5;        //viscosity-temperature index
    get(SP ,4,1)=1.0;         //reciprocal of VSS scattering parameter (1 for VHS)
    get(SP ,5,1)=5.e-26;     //mass
    get(ISPR ,1,1)=0;        //number of rotational degrees of freedom
    cout<<"Hard Sphere data done"<<endl;
    return;
}


void ARGON()
{
    // //GAS gas;
    // //CALC calc;
    cout<<"Reading Argon Data"<<endl;
    MSP=1;
    MMRM=0;
    MMVM=0;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=1;
    MVIBL=0;
    ALLOCATE_GAS();
    get(SP ,1,1)=4.17e-10;
    get(SP ,2,1)=273.15;
    get(SP ,3,1)=0.81;
    get(SP ,4,1)=1.0;
    get(SP ,5,1)=6.63e-26;
    get(ISPR ,1,1)=0;
    get(ISPR ,2,1)=0;
    cout<<"Argon Data done"<<endl;
    return;
}
//
void IDEAL_NITROGEN()
{
    // //GAS gas;
    // //CALC calc;
    cout<<"Reading IDEAL_NITROGEN data"<<endl;
    MSP=1;
    MMRM=1;
    MMVM=0;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=0;
    MVIBL=0;
    //cout << "initaial values set\n" ; // dsuedit
    ALLOCATE_GAS();
    
    //cout << "allocation finished \n" ; // dsuedit
    get(SP ,1,1)=4.17e-10;
    get(SP ,2,1)=273.0;
    get(SP ,3,1)=0.74;
    
    get(SP ,4,1)=1.0;
    get(SP ,5,1)=4.65e-26;
    get(ISPR ,1,1)=2;
    get(ISPR ,2,1)=0;

    get(SPR ,1,1)=5.0;
    cout << "ideal_nitrogen data done\n" ;
    return;
}
//
void REAL_OXYGEN()
{
    //
    //GAS gas;
    //CALC calc;
    cout<<"Reading Real_Oxygen data"<<endl;
    MSP=2;
    MMRM=1;
    MMVM=1;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=5;
    MVIBL=26;
    ALLOCATE_GAS();
    get(SP ,1,1)=4.07e-10;
    get(SP ,2,1)=273.00;
    get(SP ,3,1)=0.77e00;
    get(SP ,4,1)=1.e00;
    get(SP ,5,1)=5.312e-26;
    get(SP ,6,1)=0.e00;
    get(ISPR ,1,1)=2;
    get(ISPR ,2,1)=0 ;            //0,1 for constant,polynomial rotational relaxation collision number
    get(SPR ,1,1)=5.0;             // the collision number or the coefficient of temperature in the polynomial (if a polynomial, the coeff. of T^2 is in spr_db(3  )
    
    get(ISPV  ,1)=1   ;            // the number of vibrational modes
    get(SPVM ,1,1,1)=2256.e00  ;        // the characteristic vibrational temperature
    get(SPVM ,2,1,1)=90000.e00;        // a constant Zv, or the reference Zv
    get(SPVM ,3,1,1)=2256.e00;        // -1 for a constant Zv, or the reference temperature
    get(SPVM ,5,1,1)=1.0;            //arbitrary reduction factor
    get( ISPVM ,1,1,1)=2;
    get( ISPVM ,2,1,1)=2;
    get(NELL  ,1)=3;
    if(MELE > 1) {
        //*
        get(QELC ,1,1,1)=3.0;
        get(QELC ,2,1,1)=0.0;
        get(QELC ,3,1,1)=50.0;  //500.
        get(QELC ,1,2,1)=2.0;
        get(QELC ,2,2,1)=11393.0;
        get(QELC ,3,2,1)=50.0;  //500         //for equipartition, the cross-section ratios must be the same for all levels
        get(QELC ,1,3,1)=1.0;
        get(QELC ,3,3,1)=50.0;  //500.
    }
    //
    //species 2 is atomic oxygen
    get(SP ,1,2)=3.e-10;
    get(SP ,2,2)=273.e00;
    get(SP ,3,2)=0.8e00;
    get(SP ,4,2)=1.e00;
    get(SP ,5,2)=2.656e-26;
    get(SP ,6,2)=4.099e-19;
    get(ISPR ,1,2)=0;
    get(ISPV  ,2)=0;     //must be set//
    //set electronic information
    if(MELE > 1){
        get(NELL  ,2)=5;
        get(QELC ,1,1,2)=5.0;
        get(QELC ,2,1,2)=0.0;
        get(QELC ,2,3,1)=18985.0;
        get(QELC ,3,1,2)=50.0;
        get(QELC ,1,2,2)=3.0;
        get(QELC ,2,2,2)=228.9;
        get(QELC ,3,2,2)=50.0;
        get(QELC ,1,3,2)=1.0;
        get(QELC ,2,3,2)=325.9;
        get(QELC ,3,3,2)=50.0;
        get(QELC ,1,4,2)=5.0;
        get(QELC ,2,4,2)=22830.0;
        get(QELC ,3,4,2)=50.0;
        get(QELC ,1,5,2)=1.0;
        get(QELC ,2,5,2)=48621.0;
        get(QELC ,3,5,2)=50.0;
    }
    //set data needed for recombination
    //
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(ISPRC ,i,j)=0;
            get(ISPRK ,i,j)=0;
        }
    }
    // ISPRC=0;
    // ISPRK=0;
    get(ISPRC ,2,2)=1;    //O+O -> O2  recombined species code for an O+O recombination
    get(ISPRK ,2,2)=1 ;     //the relevant vibrational mode of this species
    get( SPRC ,1,2,2,1)=0.04;
    get( SPRC ,2,2,2,1)=-1.3;
    get( SPRC ,1,2,2,2)=0.05;
    get( SPRC ,2,2,2,2)=-1.1;
    get( SPRT ,1,2,2)=5000.e00;
    get( SPRT ,2,2,2)=15000.e00;
    //
    //memget(NSPEX,0,sizeof(*NSPEX));
    //memget(SPEX,0.e00,sizeof(*SPEX));
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(NSPEX  ,i,j)=0;
        }
    }
    for(int i=1;i<7;i++){
        for(int j=1;j<MMEX+1;j++){
            for(int k=1;k<MSP+1;k++){
                for(int l=1;l<MSP+1;l++)
                    get(SPEX  ,i,j,k,l)=0.e00;
            }
        }
    }
    //SPEX=0.e00;
    //ISPEX=0;
    //
    DERIVED_GAS_DATA();
    //
    cout<<"Real_Oxygen data done"<<endl;
    return;
}
//
void IDEAL_AIR()
{
    //GAS gas;
    //CALC calc;
    cout<<"Reading IDEAL_AIR data"<<endl;
    MSP=2;
    MMRM=1;
    MMVM=0;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=1;
    MVIBL=0;
    //
    ALLOCATE_GAS();
    //
    get(SP ,1,1)=4.07e-10;
    get(SP ,2,1)=273.0;
    get(SP ,3,1)=0.77;
    get(SP ,4,1)=1.0;
    get(SP ,5,1)=5.312e-26;
    get(ISPR ,1,1)=2;
    get(ISPR ,2,1)=0;
    get(SPR ,1,1)=5.0;
    get(SP ,1,2)=4.17e-10;
    get(SP ,2,2)=273.0;
    get(SP ,3,2)=0.74;
    get(SP ,4,2)=1.0;
    get(SP ,5,2)=4.65e-26;
    get(ISPR ,1,2)=2;
    get(ISPR ,2,2)=0;
    get(SPR ,1,2)=5.0;
    cout<<"IDEAL_AIR data done"<<endl;
    return;
}
//
void REAL_AIR()
{
    //GAS gas;
    //CALC calc;
    cout<<"REAL_AIR data done"<<endl;
    MSP=5;
    MMRM=1;
    MMVM=1;
    MELE=5;
    MVIBL=40;  //?
    //
    MEX=4;
    MMEX=1;
    //
    MNSR=0;
    ALLOCATE_GAS();
    //species 1 is oxygen
    get(SP ,1,1)=4.07e-10;
    get(SP ,2,1)=273.e00;
    get(SP ,3,1)=0.77e00;
    get(SP ,4,1)=1.e00;
    get(SP ,5,1)=5.312e-26;
    get(SP ,6,1)=0.e00;
    get(ISPR ,1,1)=2;
    get(ISPR ,2,1)=0;
    get(SPR ,1,1)=5.e00;
    get(ISPV  ,1)=1;               // the number of vibrational modes
    get(SPVM ,1,1,1)=2256.e00;          // the characteristic vibrational temperature
    get(SPVM ,2,1,1)=18000.e00;  //90000.D00        // a constant Zv, or the reference Zv
    get(SPVM ,3,1,1)=2256.e00;       // -1 for a constant Zv, or the reference temperature
    get(SPVM ,5,1,1)=1.0;
    get( ISPVM ,1,1,1)=3;
    get( ISPVM ,2,1,1)=3;
    get(NELL  ,1)=3;
    get(QELC ,1,1,1)=3.0;
    get(QELC ,2,1,1)=0.0;
    get(QELC ,3,1,1)=50.0;
    get(QELC ,1,2,1)=2.0;
    get(QELC ,2,2,1)=11393.0;
    get(QELC ,3,2,1)=50.0;
    get(QELC ,1,3,1)=1.0;
    
    get(QELC ,2,3,1)=18985.0;
    get(QELC ,3,3,1)=50.0;
    //species 2 is nitrogen
    get(SP ,1,2)=4.17e-10;
    get(SP ,2,2)=273.e00;
    get(SP ,3,2)=0.74e00;
    get(SP ,4,2)=1.e00;
    get(SP ,5,2)=4.65e-26;
    get(SP ,6,2)=0.e00;
    get(ISPR ,1,2)=2;
    get(ISPR ,2,2)=0;
    get(SPR ,1,2)=5.e00;
    get(ISPV  ,2)=1;
    get(SPVM ,1,1,2)=3371.e00;
    get(SPVM ,2,1,2)=52000.e00;     //260000.D00
    get(SPVM ,3,1,2)=3371.e00;
    get(SPVM ,5,1,2)=0.3;
    get( ISPVM ,1,1,2)=4;
    get( ISPVM ,2,1,2)=4;
    get(NELL  ,2)=1;
    get(QELC ,1,1,2)=1.0;
    get(QELC ,2,1,2)=0.0;
    get(QELC ,3,1,2)=100.0;
    //species 3 is atomic oxygen
    get(SP ,1,3)=3.e-10;
    get(SP ,2,3)=273.e00;
    get(SP ,3,3)=0.8e00;
    get(SP ,4,3)=1.e00;
    get(SP ,5,3)=2.656e-26;
    get(SP ,6,3)=4.099e-19;
    get(ISPR ,1,3)=0;
    get(ISPV  ,3)=0;
    get(NELL  ,3)=5;
    get(QELC ,1,1,3)=5.0;
    get(QELC ,2,1,3)=0.0;
    get(QELC ,3,1,3)=50.0;
    get(QELC ,1,2,3)=3.0;
    get(QELC ,2,2,3)=228.9;
    get(QELC ,3,2,3)=50.0;
    get(QELC ,1,3,3)=1.0;
    get(QELC ,2,3,3)=325.9;
    get(QELC ,3,3,3)=50.0;
    get(QELC ,1,4,3)=5.0;
    get(QELC ,2,4,3)=22830.0;
    get(QELC ,3,4,3)=50.0;
    get(QELC ,1,5,3)=1.0;
    get(QELC ,2,5,3)=48621.0;
    get(QELC ,3,5,3)=50.0;
    //species 4 is atomic nitrogen
    get(SP ,1,4)=3.e-10;
    get(SP ,2,4)=273.e00;
    get(SP ,3,4)=0.8e00;
    get(SP ,4,4)=1.0e00;
    get(SP ,5,4)=2.325e-26;
    get(SP ,6,4)=7.849e-19;
    get(ISPR ,1,4)=0;
    get(ISPV  ,4)=0;
    get(NELL  ,4)=3;
    get(QELC ,1,1,4)=4.0;
    get(QELC ,2,1,4)=0.0;
    get(QELC ,3,1,4)=50.0;
    get(QELC ,1,2,4)=10.0;
    get(QELC ,2,2,4)=27658.0;
    get(QELC ,3,2,4)=50.0;
    get(QELC ,1,3,4)=6.0;
    get(QELC ,2,3,4)=41495.0;
    get(QELC ,3,3,4)=50.0;
    //species 5 is NO
    get(SP ,1,5)=4.2e-10;
    get(SP ,2,5)=273.e00;
    get(SP ,3,5)=0.79e00;
    get(SP ,4,5)=1.0e00;
    get(SP ,5,5)=4.98e-26;
    get(SP ,6,5)=1.512e-19;
    get(ISPR ,1,5)=2;
    get(ISPR ,2,5)=0;
    get(SPR ,1,5)=5.e00;
    get(ISPV  ,5)=1;
    get(SPVM ,1,1,5)=2719.e00;
    get(SPVM ,2,1,5)=14000.e00;   //70000.D00
    get(SPVM ,3,1,5)=2719.e00;
    get(SPVM ,5,1,5)=0.2;
    get( ISPVM ,1,1,5)=3;
    get( ISPVM ,2,1,5)=4;
    get(NELL  ,5)=2;
    get(QELC ,1,1,5)=2.0;
    get(QELC ,2,1,5)=0.0;
    get(QELC ,3,1,5)=50.0;
    get(QELC ,1,2,5)=2.0;
    get(QELC ,2,2,5)=174.2;
    get(QELC ,3,2,5)=50.0;
    //set the recombination data for the molecule pairs
    //memget(ISPRC,0,sizeof(*ISPRC));//ISPRC=0;    //data os zero unless explicitly set
    //memget(ISPRK,0,sizeof(*ISPRK));//ISPRK=0;
    //memget(SPRC,0,sizeof(*SPRC));//SPRC=0.e00;
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(ISPRC ,i,j)=0;
        }
    }
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(ISPRK ,i,j)=0;
        }
    }
    for(int i=1;i<5;i++){
        for(int j=1;j<MSP+1;j++){
            for(int k=1;k<MSP+1;k++){
                for(int l=1;l<MSP+1;l++)
                    get(SPEX  ,i,j,k,l)=0.e00;
            }
        }
    }
    get(ISPRC ,3,3)=1; //O+O -> O2  recombined species code for an O+O recombination
    get(ISPRK ,3,3)=1;
    get( SPRC ,1,3,3,1)=0.04e00;
    get( SPRC ,2,3,3,1)=-1.3e00;
    get( SPRC ,1,3,3,2)=0.07e00;
    get( SPRC ,2,3,3,2)=-1.2e00;
    get( SPRC ,1,3,3,3)=0.08e00;
    get( SPRC ,2,3,3,3)=-1.2e00;
    get( SPRC ,1,3,3,4)=0.09e00;
    get( SPRC ,2,3,3,4)=-1.2e00;
    get( SPRC ,1,3,3,5)=0.065e00;
    get( SPRC ,2,3,3,5)=-1.2e00;
    get( SPRT ,1,3,3)=5000.e00;
    get( SPRT ,2,3,3)=15000.e00;
    get(ISPRC ,4,4)=2;  //N+N -> N2
    get(ISPRK ,4,4)=1;
    get( SPRC ,1,4,4,1)=0.15e00;
    get( SPRC ,2,4,4,1)=-2.05e00;
    get( SPRC ,1,4,4,2)=0.09e00;
    get( SPRC ,2,4,4,2)=-2.1e00;
    get( SPRC ,1,4,4,3)=0.16e00;
    get( SPRC ,2,4,4,3)=-2.0e00;
    get( SPRC ,1,4,4,4)=0.17e00;
    get( SPRC ,2,4,4,4)=-2.0e00;
    get( SPRC ,1,4,4,5)=0.17e00;
    get( SPRC ,2,4,4,5)=-2.1e00;
    get( SPRT ,1,4,4)=5000.e00;
    get( SPRT ,2,4,4)=15000.e00;
    get(ISPRC ,3,4)=5;
    get(ISPRK ,3,4)=1;
    get( SPRC ,1,3,4,1)=0.3e00;
    get( SPRC ,2,3,4,1)=-1.9e00;
    get( SPRC ,1,3,4,2)=0.4e00;
    get( SPRC ,2,3,4,2)=-2.0e00;
    get( SPRC ,1,3,4,3)=0.3e00;
    get( SPRC ,2,3,4,3)=-1.75e00;
    get( SPRC ,1,3,4,4)=0.3e00;
    get( SPRC ,2,3,4,4)=-1.75e00;
    get( SPRC ,1,3,4,5)=0.15e00;
    get( SPRC ,2,3,4,5)=-1.9e00;
    get( SPRT ,1,3,4)=5000.e00;
    get( SPRT ,2,3,4)=15000.e00;
    //set the exchange reaction data
    //memget(SPEX,0,sizeof(*SPEX));//SPEX=0.e00;
    for(int i=1;i<7;i++){
        for(int j=1;j<MMEX+1;j++){
            for(int k=1;k<MSP+1;k++){
                for(int l=1;l<MSP+1;l++)
                    get(SPEX  ,i,j,k,l)=0.e00;
            }
        }
    }
    //ISPEX=0;
    //NSPEX=0;
    get(NSPEX  ,2,3)=1;
    get(NSPEX  ,4,5)=1;
    get(NSPEX  ,3,5)=1;
    get(NSPEX  ,1,4)=1;
    //N2+O->NO+N
    get(ISPEX  ,1,1,2,3)=2;
    get(ISPEX  ,1,2,2,3)=3;
    get(ISPEX  ,1,3,2,3)=5;
    get(ISPEX  ,1,4,2,3)=4;
    get(ISPEX  ,1,5,2,3)=1;
    get(ISPEX  ,1,6,2,3)=1;
    get(SPEX  ,6,1,2,3)=0.e00;
    get(NEX  ,1,2,3)=1;
    //NO+N->N2+0
    get(ISPEX  ,1,1,4,5)=5;
    get(ISPEX  ,1,2,4,5)=4;
    get(ISPEX  ,1,3,4,5)=2;
    get(ISPEX  ,1,4,4,5)=3;
    get(ISPEX  ,1,5,4,5)=1;
    get(ISPEX  ,1,6,4,5)=1;
    get(ISPEX  ,1,7,4,5)=1;
    get(SPEX  ,1,1,4,5)=0.8e00;
    get(SPEX  ,2,1,4,5)=-0.75e00;
    get(SPEX  ,4,1,4,5)=5000.e00;
    get(SPEX  ,5,1,4,5)=15000.e00;
    get(SPEX  ,6,1,4,5)=0.e00;
    get(NEX  ,1,4,5)=2;
    //NO+O->O2+N
    get(ISPEX  ,1,1,3,5)=5;
    get(ISPEX  ,1,2,3,5)=3;
    get(ISPEX  ,1,3,3,5)=1;
    get(ISPEX  ,1,4,3,5)=4;
    get(ISPEX  ,1,5,3,5)=1;
    get(ISPEX  ,1,6,3,5)=1;
    get(SPEX  ,6,1,3,5)=2.e-19;
    get(NEX  ,1,3,5)=3;
    //O2+N->NO+O
    get(ISPEX  ,1,1,1,4)=1;
    get(ISPEX  ,1,2,1,4)=4;
    get(ISPEX  ,1,3,1,4)=5;
    get(ISPEX  ,1,4,1,4)=3;
    get(ISPEX  ,1,5,1,4)=1;
    get(ISPEX  ,1,6,1,4)=1;
    get(ISPEX  ,1,7,1,4)=1 ;
    get(SPEX  ,1,1,1,4)=7.e00;
    get(SPEX  ,2,1,1,4)=-0.85e00;
    get(SPEX  ,4,1,1,4)=5000.e00;
    get(SPEX  ,5,1,1,4)=15000.e00;
    get(SPEX  ,6,1,1,4)=0.e00;
    get(NEX  ,1,1,4)=4;
    
    DERIVED_GAS_DATA();
    cout<<"REAL_AIR data done"<<endl;
    return;
}
//
void HELIUM_ARGON_XENON()
{
    //GAS gas;
    //CALC calc;
    cout<<"Reading HELIUM_ARGON_XENON data"<<endl;
    MSP=3;
    MMRM=0;
    MMVM=0;
    MNSR=0;
    MEX=0;
    MMEX=0;
    MELE=1;
    MVIBL=0;
    
    ALLOCATE_GAS();
    
    get(SP ,1,1)=2.30e-10;   //2.33D-10
    get(SP ,2,1)=273.0;
    get(SP ,3,1)=0.66;
    get(SP ,4,1)=0.794;   //1.
    get(SP ,5,1)=6.65e-27;
    get(ISPR ,1,1)=0;
    get(ISPR ,2,1)=0;
    //
    get(SP ,1,2)=4.11e-10;   //4.17D-10
    get(SP ,2,2)=273.15;
    get(SP ,3,2)=0.81;
    get(SP ,4,2)=0.714;    //1.
    get(SP ,5,2)=6.63e-26;
    get(ISPR ,1,2)=0;
    get(ISPR ,2,2)=0;
    //
    get(SP ,1,3)=5.65e-10;   //5.74D-10
    get(SP ,2,3)=273.0;
    get(SP ,3,3)=0.85;
    get(SP ,4,3)=0.694;   //1.
    get(SP ,5,3)=21.8e-26;
    get(ISPR ,1,3)=0;
    get(ISPR ,2,3)=0;
    cout<<"HELIUM_ARGON_XENON data done"<<endl;
    return;
}
//
void OXYGEN_HYDROGEN()
{
    //
    //GAS gas;
    //CALC calc;
    cout<<"Reading OXYGEN_HYDROGEN data"<<endl;
    MSP=8;
    MMRM=3;
    MMVM=3;
    MELE=1;
    MVIBL=40;  //the maximum number of vibrational levels before a cumulative level reaches 1
    //
    MEX=16;
    MMEX=3;
    //
    MNSR=0;
    //
    ALLOCATE_GAS();
    //
    //species 1 is hydrogen H2
    get(SP ,1,1)=2.92e-10;
    get(SP ,2,1)=273.e00;
    get(SP ,3,1)=0.67e00;
    get(SP ,4,1)=1.e00;
    get(SP ,5,1)=3.34e-27;
    get(SP ,6,1)=0.e00;
    get(ISPR ,1,1)=2;
    get(ISPR ,2,1)=0;
    get(SPR ,1,1)=5.e00;
    get(ISPV  ,1)=1;         // the number of vibrational modes
    get(SPVM ,1,1,1)=6159.e00;          // the characteristic vibrational temperature
    get(SPVM ,2,1,1)=20000.e00;  //estimate
    get(SPVM ,3,1,1)=2000.e00; //estimate
    get(SPVM ,5,1,1)=1.0;
    get( ISPVM ,1,1,1)=2;
    get( ISPVM ,2,1,1)=2;
    //species 2 is atomic hydrogen H
    get(SP ,1,2)=2.5e-10;      //estimate
    get(SP ,2,2)=273.e00;
    get(SP ,3,2)=0.8e00;
    get(SP ,4,2)=1.e00;
    get(SP ,5,2)=1.67e-27;
    get(SP ,6,2)=3.62e-19;
    get(ISPR ,1,2)=0;
    get(ISPV  ,2)=0;
    //species 3 is oxygen O2
    get(SP ,1,3)=4.07e-10;
    get(SP ,2,3)=273.e00;
    get(SP ,3,3)=0.77e00;
    get(SP ,4,3)=1.e00;
    get(SP ,5,3)=5.312e-26;
    get(SP ,6,3)=0.e00;
    get(ISPR ,1,3)=2;
    get(ISPR ,2,3)=0;
    get(SPR ,1,3)=5.e00;
    get(ISPV  ,3)=1;               // the number of vibrational modes
    get(SPVM ,1,1,3)=2256.e00;          // the characteristic vibrational temperature
    get(SPVM ,2,1,3)=18000.e00;  //90000.D00        // a constant Zv, or the reference Zv
    get(SPVM ,3,1,3)=2256.e00;       // -1 for a constant Zv, or the reference temperature
    get(SPVM ,5,1,3)=1.e00;
    get( ISPVM ,1,1,3)=4;
    get( ISPVM ,2,1,3)=4;
    //species 4 is atomic oxygen O
    get(SP ,1,4)=3.e-10;    //estimate
    get(SP ,2,4)=273.e00;
    get(SP ,3,4)=0.8e00;
    get(SP ,4,4)=1.e00;
    get(SP ,5,4)=2.656e-26;
    get(SP ,6,4)=4.099e-19;
    get(ISPR ,1,4)=0;
    get(ISPV  ,4)=0;
    //species 5 is hydroxy OH
    get(SP ,1,5)=4.e-10;       //estimate
    get(SP ,2,5)=273.e00;
    get(SP ,3,5)=0.75e00;      //-estimate
    get(SP ,4,5)=1.0e00;
    get(SP ,5,5)=2.823e-26;
    get(SP ,6,5)=6.204e-20;
    get(ISPR ,1,5)=2;
    get(ISPR ,2,5)=0;
    get(SPR ,1,5)=5.e00;
    get(ISPV  ,5)=1;
    get(SPVM ,1,1,5)=5360.e00;
    get(SPVM ,2,1,5)=20000.e00;   //estimate
    get(SPVM ,3,1,5)=2500.e00;    //estimate
    get(SPVM ,5,1,5)=1.0e00;
    get( ISPVM ,1,1,5)=2;
    get( ISPVM ,2,1,5)=4;
    //species 6 is water vapor H2O
    get(SP ,1,6)=4.5e-10;      //estimate
    get(SP ,2,6)=273.e00;
    get(SP ,3,6)=0.75e00 ;     //-estimate
    get(SP ,4,6)=1.0e00;
    get(SP ,5,6)=2.99e-26;
    get(SP ,6,6)=-4.015e-19;
    get(ISPR ,1,6)=3;
    get(ISPR ,2,6)=0;
    get(SPR ,1,6)=5.e00;
    get(ISPV  ,6)=3;
    get(SPVM ,1,1,6)=5261.e00;  //symmetric stretch mode
    get(SPVM ,2,1,6)=20000.e00;   //estimate
    get(SPVM ,3,1,6)=2500.e00;    //estimate
    get(SPVM ,5,1,6)=1.e00;
    get(SPVM ,1,2,6)=2294.e00;  //bend mode
    get(SPVM ,2,2,6)=20000.e00;   //estimate
    get(SPVM ,3,2,6)=2500.e00;    //estimate
    get(SPVM ,5,2,6)=1.0e00;
    get(SPVM ,1,3,6)=5432.e00;  //asymmetric stretch mode
    get(SPVM ,2,3,6)=20000.e00;   //estimate
    get(SPVM ,3,3,6)=2500.e00 ;   //estimate
    get(SPVM ,5,3,6)=1.e00;
    get( ISPVM ,1,1,6)=2;
    get( ISPVM ,2,1,6)=5;
    get( ISPVM ,1,2,6)=2;
    get( ISPVM ,2,2,6)=5;
    get( ISPVM ,1,3,6)=2;
    get( ISPVM ,2,3,6)=5;
    //species 7 is hydroperoxy HO2
    get(SP ,1,7)=5.5e-10;       //estimate
    get(SP ,2,7)=273.e00;
    get(SP ,3,7)=0.75e00 ;     //-estimate
    get(SP ,4,7)=1.0e00;
    get(SP ,5,7)=5.479e-26;
    get(SP ,6,7)=2.04e-20;
    get(ISPR ,1,7)=2;    //assumes that HO2 is linear
    get(ISPR ,2,7)=0;
    get(SPR ,1,7)=5.e00;
    get(ISPV  ,7)=3;
    get(SPVM ,1,1,7)=4950.e00;
    get(SPVM ,2,1,7)=20000.e00;   //estimate
    get(SPVM ,3,1,7)=2500.e00  ;  //estimate
    get(SPVM ,5,1,7)=1.e00;
    get(SPVM ,1,2,7)=2000.e00;
    get(SPVM ,2,2,7)=20000.e00;   //estimate
    get(SPVM ,3,2,7)=2500.e00;    //estimate
    get(SPVM ,5,2,7)=1.e00;
    get(SPVM ,1,3,7)=1580.e00;
    get(SPVM ,2,3,7)=20000.e00;   //estimate
    get(SPVM ,3,3,7)=2500.e00;    //estimate
    get(SPVM ,5,3,7)=1.e00;
    get( ISPVM ,1,1,7)=2;
    get( ISPVM ,2,1,7)=3;
    get( ISPVM ,1,2,7)=2;
    get( ISPVM ,2,2,7)=3;
    get( ISPVM ,1,3,7)=2;
    get( ISPVM ,2,3,7)=3;
    //Species 8 is argon
    get(SP ,1,8)=4.17e-10;
    get(SP ,2,8)=273.15;
    get(SP ,3,8)=0.81   ;
    get(SP ,4,8)=1.0;
    get(SP ,5,8)=6.63e-26;
    get(SP ,6,8)=0.e00;
    get(ISPR ,1,8)=0;
    get(ISPV  ,8)=0;
    //
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(ISPRC ,i,j)=0;
        }
    }
    //ISPRC=0;    //data is zero unless explicitly set
    //
    get(ISPRC ,4,4)=3;    //O+O+M -> O2+M  recombined species code for an O+O recombination
    get(ISPRK ,4,4)=1;
    get( SPRC ,1,4,4,1)=0.26e00;
    get( SPRC ,2,4,4,1)=-1.3e00;
    get( SPRC ,1,4,4,2)=0.29e00;
    get( SPRC ,2,4,4,2)=-1.3e00;
    get( SPRC ,1,4,4,3)=0.04e00;
    get( SPRC ,2,4,4,3)=-1.5e00;
    get( SPRC ,1,4,4,4)=0.1e00;
    get( SPRC ,2,4,4,4)=-1.4e00;
    get( SPRC ,1,4,4,5)=0.1e00;
    get( SPRC ,2,4,4,5)=-1.4e00;
    get( SPRC ,1,4,4,6)=0.1e00;
    get( SPRC ,2,4,4,6)=-1.4e00;
    get( SPRC ,1,4,4,7)=0.07e00;
    get( SPRC ,2,4,4,7)=-1.5e00;
    get( SPRC ,1,4,4,8)=0.07e00;
    get( SPRC ,2,4,4,8)=-1.5e00;
    get( SPRT ,1,4,4)=1000.e00;
    get( SPRT ,2,4,4)=3000.e00;
    //
    get(ISPRC ,2,2)=1;   //H+H+M -> H2+M
    get(ISPRK ,2,2)=1;
    get( SPRC ,1,2,2,1)=0.07e00;
    get( SPRC ,2,2,2,1)=-2.e00;
    get( SPRC ,1,2,2,2)=0.11e00;
    get( SPRC ,2,2,2,2)=-2.2e00;
    get( SPRC ,1,2,2,3)=0.052e00;
    get( SPRC ,2,2,2,3)=-2.5e00;
    get( SPRC ,1,2,2,4)=0.052e00;
    get( SPRC ,2,2,2,4)=-2.5e00;
    get( SPRC ,1,2,2,5)=0.052e00;
    get( SPRC ,2,2,2,5)=-2.5e00;
    get( SPRC ,1,2,2,6)=0.052e00;
    get( SPRC ,2,2,2,6)=-2.5e00;
    get( SPRC ,1,2,2,7)=0.052e00;
    get( SPRC ,2,2,2,7)=-2.5e00;
    get( SPRC ,1,2,2,8)=0.04e00;
    get( SPRC ,2,2,2,7)=-2.5e00;
    get( SPRT ,1,2,2)=1000.e00;
    get( SPRT ,2,2,2)=3000.e00;
    //
    get(ISPRC ,2,4)=5;    //H+0+M -> OH+M
    get(ISPRK ,2,4)=1;
    get( SPRC ,1,2,4,1)=0.15e00;
    get( SPRC ,2,2,4,1)=-2.e00;
    get( SPRC ,1,2,4,2)=0.04e00;
    get( SPRC ,2,2,4,2)=-1.3e00;
    get( SPRC ,1,2,4,3)=0.04e00;
    get( SPRC ,2,2,4,3)=-1.3e00;
    get( SPRC ,1,2,4,4)=0.04e00;
    get( SPRC ,2,2,4,4)=-1.3e00;
    get( SPRC ,1,2,4,5)=0.04e00;
    get( SPRC ,2,2,4,5)=-1.3e00;
    get( SPRC ,1,2,4,6)=0.21e00;
    get( SPRC ,2,2,4,6)=-2.1e00;
    get( SPRC ,1,2,4,7)=0.18e00;
    get( SPRC ,2,2,4,7)=-2.3e00;
    get( SPRC ,1,2,4,8)=0.16e00;
    get( SPRC ,2,2,4,8)=-2.3e00;
    get( SPRT ,1,2,4)=1000.e00;
    get( SPRT ,2,2,4)=3000.e00;
    //
    get(ISPRC ,2,5)=6;    //H+OH+M -> H2O+M
    get(ISPRK ,2,5)=1;
    get( SPRC ,1,2,5,1)=0.1e00;
    get( SPRC ,2,2,5,1)=-2.0e00;
    get( SPRC ,1,2,5,2)=0.1e00;
    get( SPRC ,2,2,5,2)=-2.0e00;
    get( SPRC ,1,2,5,3)=0.0025e00;
    get( SPRC ,2,2,5,3)=-2.2e00;
    get( SPRC ,1,2,5,4)=0.0025e00;
    get( SPRC ,2,2,5,4)=-2.2e00;
    get( SPRC ,1,2,5,5)=0.0025e00;
    get( SPRC ,2,2,5,5)=-2.2e00;
    get( SPRC ,1,2,5,6)=0.0015e00;
    get( SPRC ,2,2,5,6)=-2.2e00;
    get( SPRC ,1,2,5,7)=0.0027e00;
    get( SPRC ,2,2,5,7)=-2.e00;
    get( SPRC ,1,2,5,8)=0.0025e00;
    get( SPRC ,2,2,5,8)=-2.e00;
    get( SPRT ,1,2,5)=1000.e00;
    get( SPRT ,2,2,5)=3000.e00;
    //
    get(ISPRC ,2,3)=7;   //H+O2+M -> H02+M
    get(ISPRK ,2,3)=1;
    get( SPRC ,1,2,3,1)=0.0001e00;
    get( SPRC ,2,2,3,1)=-1.7e00;
    get( SPRC ,1,2,3,2)=0.0001e00;
    get( SPRC ,2,2,3,2)=-1.7e00;
    get( SPRC ,1,2,3,3)=0.00003e00;
    get( SPRC ,2,2,3,3)=-1.5e00;
    get( SPRC ,1,2,3,4)=0.00003e00;
    get( SPRC ,2,2,3,4)=-1.7e00;
    get( SPRC ,1,2,3,5)=0.00003e00;
    get( SPRC ,2,2,3,5)=-1.7e00;
    get( SPRC ,1,2,3,6)=0.00003e00;
    get( SPRC ,2,2,3,6)=-1.7e00;
    get( SPRC ,1,2,3,7)=0.000012e00;
    get( SPRC ,2,2,3,7)=-1.7e00;
    get( SPRC ,1,2,3,8)=0.00002e00;
    get( SPRC ,2,2,3,8)=-1.7e00;
    get( SPRT ,1,2,3)=1000.e00;
    get( SPRT ,2,2,3)=3000.e00;
    //
    //set the exchange reaction data
    //  memget(SPEX,0,sizeof(*SPEX));//SPEX=0.e00;    //all activation energies and heats of reaction are zero unless set otherwise
    for(int i=1;i<7;i++){
        for(int j=1;j<MMEX+1;j++){
            for(int k=1;k<MSP+1;k++){
                for(int l=1;l<MSP+1;l++)
                    get(SPEX  ,i,j,k,l)=0.e00;
            }
        }
    }
    //ISPEX=0;       // ISPEX is also zero unless set otherwise
    for(int i=1;i<MMEX+1;i++){
        for(int j=1;j<8;j++){
            for(int k=1;k<MSP+1;k++){
                for(int l=1;l<MSP+1;l++)
                    get(ISPEX  ,i,j,k,l)=0.e00;
            }
        }
    }
    //NSPEX=0;
    for(int i=1;i<MSP+1;i++){
        for(int j=1;j<MSP+1;j++){
            get(NSPEX  ,i,j)=0;
        }
    }
    //set the number of exchange reactions for each species pair
    get(NSPEX  ,1,3)=1;
    get(NSPEX  ,2,7)=3;
    get(NSPEX  ,2,3)=1;
    get(NSPEX  ,4,5)=1;
    get(NSPEX  ,1,4)=1;
    get(NSPEX  ,2,5)=1;
    get(NSPEX  ,1,5)=1;
    get(NSPEX  ,2,6)=1;
    get(NSPEX  ,4,6)=2;
    get(NSPEX  ,5,5)=2;
    get(NSPEX  ,4,7)=1;
    get(NSPEX  ,3,5)=1;
    //set the information on the chain reactions
    //
    //H2+O2 -> HO2+H
    get(ISPEX  ,1,1,1,3)=1;
    get(ISPEX  ,1,2,1,3)=3;
    get(ISPEX  ,1,3,1,3)=7;
    get(ISPEX  ,1,4,1,3)=2;
    get(ISPEX  ,1,5,1,3)=1;
    get(ISPEX  ,1,6,1,3)=1;
    get(SPEX  ,6,1,1,3)=0.e00;
    get(NEX  ,1,1,3)=1;
    //
    //HO2+H -> H2+02
    get(ISPEX  ,1,1,2,7)=7;
    get(ISPEX  ,1,2,2,7)=2;
    get(ISPEX  ,1,3,2,7)=1;
    get(ISPEX  ,1,4,2,7)=3;
    get(ISPEX  ,1,5,2,7)=1;
    get(ISPEX  ,1,6,2,7)=1;
    get(ISPEX  ,1,7,2,7)=1;
    //H02 is H-O-O so that not all vibrational modes contribute to this reaction, but the numbers here are guesses//
    get(SPEX  ,1,1,2,7)=20.e00;
    get(SPEX  ,2,1,2,7)=0.4e00;
    get(SPEX  ,4,1,2,7)=2000.e00;
    get(SPEX  ,5,1,2,7)=3000.e00;
    get(SPEX  ,6,1,2,7)=0.e00;
    get(NEX  ,1,2,7)=2;
    //
    //O2+H -> OH+O
    get(ISPEX  ,1,1,2,3)=3;
    get(ISPEX  ,1,2,2,3)=2;
    get(ISPEX  ,1,3,2,3)=5;
    get(ISPEX  ,1,4,2,3)=4;
    get(ISPEX  ,1,5,2,3)=1;
    get(ISPEX  ,1,6,2,3)=1;
    get(SPEX  ,6,1,2,3)=0.e00;
    get(NEX  ,1,2,3)=3;
    //
    //OH+O -> O2+H
    get(ISPEX  ,1,1,4,5)=5;
    get(ISPEX  ,1,2,4,5)=4;
    get(ISPEX  ,1,3,4,5)=3;
    get(ISPEX  ,1,4,4,5)=2;
    get(ISPEX  ,1,5,4,5)=1;
    get(ISPEX  ,1,6,4,5)=1;
    get(ISPEX  ,1,7,4,5)=1;
    get(SPEX  ,1,1,4,5)=0.65e00;
    get(SPEX  ,2,1,4,5)=-0.26;
    get(SPEX  ,4,1,4,5)=2000.e00;
    get(SPEX  ,5,1,4,5)=3000.e00;
    get(SPEX  ,6,1,4,5)=0.e00;
    get(NEX  ,1,4,5)=4;
    //
    //H2+O -> OH+H
    get(ISPEX  ,1,1,1,4)=1;
    get(ISPEX  ,1,2,1,4)=4;
    get(ISPEX  ,1,3,1,4)=5;
    get(ISPEX  ,1,4,1,4)=2;
    get(ISPEX  ,1,5,1,4)=1;
    get(ISPEX  ,1,6,1,4)=1;
    get(SPEX  ,6,1,1,4)=0.e00;
    get(NEX  ,1,1,4)=5;
    //
    //OH+H -> H2+O
    get(ISPEX  ,1,1,2,5)=5;
    get(ISPEX  ,1,2,2,5)=2;
    get(ISPEX  ,1,3,2,5)=1;
    get(ISPEX  ,1,4,2,5)=4;
    get(ISPEX  ,1,5,2,5)=1;
    get(ISPEX  ,1,6,2,5)=1;
    get(ISPEX  ,1,7,2,5)=1;
    get(SPEX  ,1,1,2,5)=0.5e00;
    get(SPEX  ,2,1,2,5)=-0.2e00;
    get(SPEX  ,4,1,2,5)=2000.e00;
    get(SPEX  ,5,1,2,5)=3000.e00;
    get(SPEX  ,6,1,2,5)=0.e00;
    get(NEX  ,1,2,5)=6;
    //
    //H20+H -> OH+H2
    get(ISPEX  ,1,1,2,6)=6;
    get(ISPEX  ,1,2,2,6)=2;
    get(ISPEX  ,1,3,2,6)=5;
    get(ISPEX  ,1,4,2,6)=1;
    get(ISPEX  ,1,5,2,6)=1;
    get(ISPEX  ,1,6,2,6)=1;
    get(SPEX  ,6,1,2,6)=2.0e-19;
    get(NEX  ,1,2,6)=7;
    
    //OH+H2 -> H2O+H
    get(ISPEX  ,1,1,1,5)=5;
    get(ISPEX  ,1,2,1,5)=1;
    get(ISPEX  ,1,3,1,5)=6;
    get(ISPEX  ,1,4,1,5)=2;
    get(ISPEX  ,1,5,1,5)=1;
    get(ISPEX  ,1,6,1,5)=1;
    get(ISPEX  ,1,7,1,5)=1;
    get(SPEX  ,1,1,1,5)=0.5;
    get(SPEX  ,2,1,1,5)=-0.2;
    get(SPEX  ,4,1,1,5)=2000.e00;
    get(SPEX  ,5,1,1,5)=3000.e00;
    get(SPEX  ,6,1,1,5)=0.e00;
    get(NEX  ,1,1,5)=8;
    //
    //H2O+O -> OH+OH
    get(ISPEX  ,1,1,4,6)=6;
    get(ISPEX  ,1,2,4,6)=4;
    get(ISPEX  ,1,3,4,6)=5;
    get(ISPEX  ,1,4,4,6)=5;
    get(ISPEX  ,1,5,4,6)=1;
    get(ISPEX  ,1,6,4,6)=1;
    get(SPEX  ,6,1,4,6)=0.e00;
    get(NEX  ,1,4,6)=9;
    //
    //0H+OH -> H2O+O
    get(ISPEX  ,1,1,5,5)=5;
    get(ISPEX  ,1,2,5,5)=5;
    get(ISPEX  ,1,3,5,5)=6;
    get(ISPEX  ,1,4,5,5)=4;
    get(ISPEX  ,1,5,5,5)=1;
    get(ISPEX  ,1,6,5,5)=1;
    get(ISPEX  ,1,7,5,5)=1;
    get(SPEX  ,1,1,5,5)=0.35;
    get(SPEX  ,2,1,5,5)=-0.2 ;
    get(SPEX  ,4,1,5,5)=2000.e00;
    get(SPEX  ,5,1,5,5)=3000.e00;
    get(SPEX  ,6,1,5,5)=0.e00;
    get(NEX  ,1,5,5)=10;
    //
    //OH+OH  -> HO2+H
    //
    get(ISPEX  ,2,1,5,5)=5;
    get(ISPEX  ,2,2,5,5)=5;
    get(ISPEX  ,2,3,5,5)=7;
    get(ISPEX  ,2,4,5,5)=2;
    get(ISPEX  ,2,5,5,5)=1;
    get(ISPEX  ,2,6,5,5)=1;
    get(SPEX  ,6,2,5,5)=0.e00;
    get(NEX  ,2,5,5)=11;
    //
    //H02+H -> 0H+OH
    get(ISPEX  ,2,1,2,7)=7;
    get(ISPEX  ,2,2,2,7)=2;
    get(ISPEX  ,2,3,2,7)=5;
    get(ISPEX  ,2,4,2,7)=5;
    get(ISPEX  ,2,5,2,7)=1;
    get(ISPEX  ,2,6,2,7)=1;
    get(ISPEX  ,2,7,2,7)=1;
    get(SPEX  ,1,2,2,7)=120.e00;
    get(SPEX  ,2,2,2,7)=-0.05e00;
    get(SPEX  ,4,2,2,7)=2000.e00;
    get(SPEX  ,5,2,2,7)=3000.e00;
    get(SPEX  ,6,2,2,7)=0.e00;
    get(NEX  ,2,2,7)=12;
    //
    //H2O+O -> HO2+H
    //
    get(ISPEX  ,2,1,4,6)=6;
    get(ISPEX  ,2,2,4,6)=4;
    get(ISPEX  ,2,3,4,6)=7;
    get(ISPEX  ,2,4,4,6)=2;
    get(ISPEX  ,2,5,4,6)=1;
    get(ISPEX  ,2,6,4,6)=1;
    get(SPEX  ,6,2,4,6)=0.e00;
    get(NEX  ,2,4,6)=13;
    //
    //H02+H -> H2O+O
    //
    get(ISPEX  ,3,1,2,7)=7;
    get(ISPEX  ,3,2,2,7)=2;
    get(ISPEX  ,3,3,2,7)=6;
    get(ISPEX  ,3,4,2,7)=4;
    get(ISPEX  ,3,5,2,7)=1;
    get(ISPEX  ,3,6,2,7)=1;
    get(ISPEX  ,3,7,2,7)=1;
    get(SPEX  ,1,3,2,7)=40.e00;
    get(SPEX  ,2,3,2,7)=-1.e00;
    get(SPEX  ,4,3,2,7)=2000.e00;
    get(SPEX  ,5,3,2,7)=3000.e00;
    get(SPEX  ,6,3,2,7)=0.e00;
    get(NEX  ,3,2,7)=14;
    //
    //OH+O2 -> HO2+O
    //
    get(ISPEX  ,1,1,3,5)=5;
    get(ISPEX  ,1,2,3,5)=3;
    get(ISPEX  ,1,3,3,5)=7;
    get(ISPEX  ,1,4,3,5)=4;
    get(ISPEX  ,1,5,3,5)=1;
    get(ISPEX  ,1,6,3,5)=1;
    get(SPEX  ,6,1,3,5)=0.e00;
    get(NEX  ,1,3,5)=15;
    //
    //H02+0 -> OH+O2
    //
    get(ISPEX  ,1,1,4,7)=7;
    get(ISPEX  ,1,2,4,7)=4;
    get(ISPEX  ,1,3,4,7)=5;
    get(ISPEX  ,1,4,4,7)=3;
    get(ISPEX  ,1,5,4,7)=1;
    get(ISPEX  ,1,6,4,7)=1;
    get(ISPEX  ,1,7,4,7)=1;
    get(SPEX  ,1,1,4,7)=100.e00;
    get(SPEX  ,2,1,4,7)=0.15e00;
    get(SPEX  ,4,1,4,7)=2000.e00;
    get(SPEX  ,5,1,4,7)=3000.e00;
    get(SPEX  ,6,1,4,7)=0.e00;
    get(NEX  ,1,4,7)=16;
    
    //
    DERIVED_GAS_DATA();
    //
    cout<<"OXYGEN_HYDROGEN data done"<<endl;
    return;
}
//****
//**END OF GAS DATABASE**
//****
//

//module 
int main()
{
    int IRUN,ICONF,N,M,IADAPT,IRETREM,ISET ;
    double A ;

    NVER =1 ;
    MVER =1 ;
    NREL = 1 ;

    //constants
    PI=3.1415926535897932E00 ;
    DPI=6.283185307179586E00 ;
    SPI=1.772453850905516E00 ; 
    BOLTZ=1.380658E-23 ;
    AVOG=6.022169E26     ;

    //adjustable computational parametres
    NMCC = 50 ;
    CPDTM = 0.2 ;
    TPDTM = 0.5 ;
    NNC = 1 ;
    SAMPRAT = 5 ;
    OUTRAT = 10 ;
    FRACSAM =1 ;
    ISAD = 0 ;
    IMTS =2 ;
    FNUMF =1 ;
    TLIM = 1.E20 ;

    
    file_9.open("DIAG.TXT" , ios::out ) ;
    int a ;
    if(file_9.is_open())
    {
        cout << "file_9 DIAG.TXT IS OPEN \n" ;
    }
    else cout << "file_9 DIAG.TXT COULDN'T BE OPEN OPENED" ;

    file_13.open("MolNum.DAT" , ios::out ) ;

    IVB =0 ;

    IRUN=2 ;
    if(IRUN == 1)   cout << "continuing an existing run . . .  \n";
    if(IRUN == 2)
    {
        cout << "enter 0 for a homogenous gas \n" ;
        cout << "Enter 1 for a one-dimensional flow, or\n" ;
        cout << "Enter 2 for a two-dimensional plane flow, or \n" ;
        cout << "Enter 3 for a three dimensional flow, or \n" ;
        cout << "Enter 4 for an axially-symmetric flow :- \n" ;
        ICLASS = 0 ;    // dsuedit
        //cin >> ICLASS ; //dsuedit ICLASS =0
        NCLASS =2 ;

        if (ICLASS < 2)
        {
            
            NCLASS=1 ;
        }
        if(ICLASS == 3) NCLASS =3 ;

        cout << "Enter 0 for an eventually steady flow, or\n" ;
        cout << "enter 1 for a continuing unsteady flow :-\n" ;
        cin >> ISF ;  //dsuedit   assuming steady state all the time .
        //ISF = 0 ;   //dsuedit
        file_9 << "Starting a new run with ICLASS, ISF" <<ICLASS<<ISF << endl ;
    }


    if( IRUN == 2 )
    {
        READ_DATA() ;
        if (ICLASS<2)
        {
            
            SET_INITIAL_STATE_1D() ;
        }
        if(ICLASS==0)   ENERGY(0,A) ;

    }
    int ch = 0 ;

    while(FTIME < TLIM) {
        

        clock_t t[8] ;
        FTIME = FTIME+DTM ;
        file_9 << "TIME " << FTIME << "\tNM -- \t" << NM << "\tCOLLA -- \t\t" << TOTCOL << endl ;
        file_13 << "FTIME/TNORM , FLOAT(NM)/FLOAT(NMI)  -- " << FTIME/TNORM << (double)(NM)/(NMI) << endl  ;
        t[0] = clock() ;
         //cout<< "  TIME --   "<<setw(20)<<setprecision(10)<<FTIME<<"  NM  "<<NM<<" \t COLLS --  "<<std::left<<setw(20)<<setprecision(10)<<TOTCOL<<"\tCollision_time : "<<endl;
         t[1] = clock() ;
        MOLECULES_MOVE_1D() ;
       
      
        //cout  << "ITYPE[1] = " << ITYPE[1] << "\t ITYPE[2] = " << ITYPE[2] << endl ;
        t[2]= clock() ;
        if((ITYPE[1] == 0)||(ITYPE[2] == 0)||(ITYPE[2]==4))
            MOLECULES_ENTER_1D() ;
        t[3] = clock() ;

        INDEX_MOLS() ;
        t[4] = clock() ;
        
        COLLISIONS() ;
        t[5] = clock() ;

        

        if (MMVM>0) DISSOCIATION() ;
        t[6] = clock() ;

        if (FTIME > TSAMP)
        {
            if(ISF==0)  SAMPLE_FLOW() ;

            if((ISF == 1) && (FTIME < TPOUT+(1e00-FRACSAM)*DTOUT))
            {
                TSAMP =TSAMP+DTSAMP ;
                INITIALISE_SAMPLES() ;
            }
            if((ISF == 1) && (FTIME >= TPOUT+(1-FRACSAM)*DTOUT))
            {
                SAMPLE_FLOW() ;
            }
        }
        t[7] = clock() ;
        if(FTIME >TOUT )
        {
            OUTPUT_RESULTS() ;
            TPOUT = FTIME ;
        }
        t[8] = clock() ;

        for(int i= 0 ; i<8 ; i++)
        {
            cout << t[i+1]-t[i] << "\t" ;
        }
        cout << endl ;
        ch++ ;
        if(ch>10000)   
            break ;
    }

    cin >> TOTCOL ;
    return 0;
}


void ALLOCATE_GAS()
{
    // //GAS gas;
    // //CALC calc;
    d_allocate(MSP,2,FSP);

    d_allocate(6,MSP,SP);
    d_allocate(3,MSP,SPR);

    d_allocate(8,MSP,MSP,SPM);
    
    i_allocate(2,MSP,ISPR);
    i_allocate(MSP,ISPV);
    d_allocate(6,MSP,2,ENTR);
    d_allocate(MSP,2,VMP);
    d_allocate(MSP,VNMAX);
    d_allocate(MSP,CR);
    d_allocate(MSP,MSP,TCOL);
    i_allocate(MSP,MSP,ISPRC);
    i_allocate(MSP,MSP,ISPRK);
    d_allocate(4,MSP,MSP,MSP,SPRC);
    i_allocate(MSP,NELL);
    d_allocate(3,MELE+1,MSP,QELC);
    d_allocate(2,MSP,MSP,MVIBL+1,SPRP);
    d_allocate(2,MSP,MSP,SPRT);
    d_allocate(MSP,AJM);
    d_allocate(MSP,FP);
    d_allocate(MSP,ALOSS);
    d_allocate(MSP,EME);
    /*ALLOCATE (FSP(MSP,2),SP(6,MSP),SPR(3,MSP),SPM(8,MSP,MSP),ISPR(2,MSP),ISPV(MSP),ENTR(6,MSP,2),      &
     VMP(MSP,2),VNMAX(MSP),CR(MSP),TCOL(MSP,MSP),ISPRC(MSP,MSP),ISPRK(MSP,MSP),SPRC(4,MSP,MSP,MSP),                        &
     NELL(MSP),QELC(3,MELE,MSP),SPRP(2,MSP,MSP,0:MVIBL),SPRT(2,MSP,MSP),AJM(MSP),FP(MSP),    &
     ALOSS(MSP),EME(MSP),STAT=ERROR)
     //
     IF (ERROR /= 0) THEN
     WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SPECIES VARIABLES',ERROR
     END IF
     //*/
    i_allocate(MMEX,MSP,MSP,NEX);
    i_allocate(MSP,MSP,NSPEX);
    d_allocate(6,MMEX,MSP,MSP,SPEX);
    i_allocate(MMEX,7,MSP,MSP,ISPEX);
    i_allocate(4,MSP,TREACG);
    d_allocate(MMEX,PSF);
    i_allocate(4,MSP,TREACL);
    d_allocate(MEX,TNEX);
    d_allocate(2,MMEX,MSP,MSP,MVIBL+1,SPREX);
    i_allocate(2,MSP,NSLEV);
    d_allocate(MSP,SLER);
    // ALLOCATE (NEX(MMEX,MSP,MSP),NSPEX(MSP,MSP),SPEX(6,MMEX,MSP,MSP),ISPEX(MMEX,7,MSP,MSP),TREACG(4,MSP),         &
    //           PSF(MMEX),TREACL(4,MSP),TNEX(MEX),SPREX(2,MMEX,MSP,MSP,0:MVIBL),NSLEV(2,MSP),SLER(MSP),STAT=ERROR)
    // //
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE Q-K REACTION VARIABLES',ERROR
    // END IF
    // //
    

    if(MMVM >= 0){
        d_allocate(5,MMVM,MSP,SPVM);
        i_allocate(2,MMVM,MSP,ISPVM);
        d_allocate(MSP,TDISS);
        d_allocate(MSP,TRECOMB);
        //ALLOCATE (SPVM(5,MMVM,MSP),ISPVM(2,MMVM,MSP),TDISS(MSP),TRECOMB(MSP),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE VIBRATION VARIABLES',ERROR
    }
    
    //N.B. surface reactions are not yet implemented
    if(MNSR > 0){
        d_allocate(MNSR,ERS);
        i_allocate(2,MNSR,LIS);
        i_allocate(6,MNSR,LRS);
        i_allocate(MNSR,MSP,ISRCD);
        //ALLOCATE (ERS(MNSR),LIS(2,MNSR),LRS(6,MNSR),ISRCD(MNSR,MSP),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SURFACE REACTION VARIABLES',ERROR
    }
   

     //AJM=0.e00;
    //memget(AJM,0.e00,sizeof(*AJM));
    for(int i=0 ; i<MSP+1 ; i++ ) {
        AJM[i] = 0 ;
    }
    
    
    return;
    
}

void ENERGY(int I,double &TOTEN)
{
    //calculate the total energy (all molecules if I=0, otherwise molecule I)
    //I>0 used for dianostic purposes only
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,L,N,II,M,IV,KV,J;
    double TOTENI,TOTELE;
    //
    TOTEN=0.0;
    TOTELE=0;

    //
    
    if(I == 0){
        for(N=1;N<=NM;N++) {
            if( get(IPCELL  ,N) > 0){
                L=get(IPSP ,N) ;
                TOTENI=TOTEN;
                TOTEN=TOTEN+get(SP ,6,L);
                TOTEN=TOTEN+0.5e00*get(SP ,5,L)*(pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2)+pow(get(PV  ,3,N),2));
                if(get(ISPR ,1,L) > 0) TOTEN=TOTEN+PROT[N];
                if(get(ISPV  ,L) > 0){
                    for(KV=1;KV<=get(ISPV  ,L);KV++){
                        J=get (IPVIB , KV,N);
                        //         IF (J <0) THEN
                        //           J=-J
                        //           IF (J == 99999) J=0
                        //         END IF
                        TOTEN=TOTEN+double(J)*BOLTZ*get(SPVM ,1,KV,L);
                    }
                }
            }
            if(MELE > 1){
                TOTEN=TOTEN+PELE[N];
                TOTELE=TOTELE+PELE[N];
            }
            //if((TOTEN-TOTENI) > 1.e-16) cout<<"MOL "<<N<<" ENERGY "<<TOTEN-TOTENI<<endl;
        }
        //
        //WRITE (9,*) 'Total Energy =',TOTEN,NM
        //WRITE (*,*) 'Total Energy =',TOTEN,NM
        file_9<<"Total Energy =  "<<setprecision(25)<<TOTEN<<"\t"<<NM<<endl;
        cout<<"Total Energy =  "<<setprecision(20)<<TOTEN<<"\t"<<NM<<endl;
        //  WRITE (*,*) 'Electronic Energy =',TOTELE
    }
    else{
        N=I;
        if(get(IPCELL  ,N) > 0){
            L=get(IPSP ,N);
            TOTEN=TOTEN+get(SP ,6,L);
            TOTEN=TOTEN+0.5e00*get(SP ,5,L)*(pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2)+pow(get(PV  ,3,N),2));
            if(get(ISPR ,1,L) > 0) TOTEN=TOTEN+PROT[N];
            if(get(ISPV  ,L) > 0){
                for(KV=1;KV<=get(ISPV  ,L);KV++){
                    J=get (IPVIB , KV,N);
                    //         IF (J <0) THEN
                    //           J=-J
                    //           IF (J == 99999) J=0
                    //         END IF
                    TOTEN=TOTEN+double(J)*BOLTZ*get(SPVM ,1,KV,L);
                }
            }
        }
    }
    
    //
    return;   //
}

void INITIALISE_SAMPLES()
{
    file_9 << " INITIALISE SAMPLES IS RUNNING \n" ;
    file_3 << "INITIALISE SAMPLES IS running\n" ;
    cout << "INITIALISE SAMPLES IS running\n" ;
    int N;
    //
    NSAMP=0.0;
    TISAMP=FTIME;
    NMISAMP=NM;
    //memget(COLLS,0.e00,sizeof(*COLLS));memget(WCOLLS,0.e00,sizeof(*WCOLLS));memget(CLSEP,0.e00,sizeof(*CLSEP));
   
    for(int i=0;i<NCELLS+1;i++)
        COLLS[i]=0.e00;
    for(int i=0;i<NCELLS+1;i++)
       WCOLLS[i]=0.e00;
    for(int i=0;i<NCELLS+1;i++)
        CLSEP[i]=0.e00;
    //COLLS=0.e00 ; WCOLLS=0.e00 ; CLSEP=0.e00;
    //memget(TCOL,0.0,sizeof(*TCOL));//TCOL=0.0;
    for(int i=0;i<MSP+1;i++){
        for(int j=0;j<MSP+1;j++){
            get (TCOL , i,j)=0.0;
        }
    }
    //TREACG=0;
    //TREACL=0;
    for(int i=0;i<5;i++){
        for(int j=0;j<MSP+1;j++){
            get (TREACG , i,j)=0;
        }
    }
    for(int i=0;i<5;i++){
        for(int j=0;j<MSP+1;j++){
            get (TREACL , i,j)=0;
        }
    }
    //memget(CS,0.0,sizeof(*CS));memget(CSS,0.0,sizeof(*CSS));memget(CSSS,0.0,sizeof(*CSSS));
    for(int j=0;j<MSP+10;j++){
        for(int k=0;k<NCELLS+1;k++){
            for(int l=0;l<MSP+1;l++)
                get (CS,1+ j,k,l)=0.0;
        }
    }
    for(int i=0;i<9;i++){
        for(int j=0;j<3;j++){
            for(int k=0;k<MSP+1;k++){
                for(int l=0;l<3;l++)
                    get (CSS , 1+ i,j,k,l)=0.0;
            }
        }
    }
    for(int k=0;k<7;k++){
        for(int l=0;l<3;l++)
            get (CSSS , k,l)=0.0;
    }
    //CS=0.0 ; CSS=0.0 ; CSSS=0.0;
    //memget(VIBFRAC,0.e00,sizeof(*VIBFRAC));//VIBFRAC=0.e00;
    //memget(SUMVIB,0.e00,sizeof(*SUMVIB));//SUMVIB=0.e00;
    for(int j=0;j<MSP+1;j++){
        for(int k=0;k<MMVM+1;k++){
            for(int l=0;l<151;l++)
                get (VIBFRAC , j,k,l+1)=0.0;
        }
    }
    for(int k=0;k<MSP+1;k++){
        for(int l=0;l<MMVM+1;l++)
            get (SUMVIB , k,l)=0.0;
    }
    
}

void SET_INITIAL_STATE_1D()
{
    //set the initial state of a homogeneous or one-dimensional flow
    //
    //MOLECS molecs;
    //GEOM_1D geom;
    //GAS gas;
    //CALC calc;
    //OUTPUT output;
    //
    //
    int J,L,K,KK,KN,II,III,INC,NSET,NSC;
    long long N,M;
    double A,B,AA,BB,BBB,SN,XMIN,XMAX,WFMIN,DENG,ELTI,EA,XPREV;
    double DMOM[4];
    double VB[4][3];
    double ROTE[3];
    //
    //NSET the alternative set numbers in the setting of exact initial state
    //DMOM(N) N=1,2,3 for x,y and z momentum sums of initial molecules
    //DENG the energy sum of the initial molecules
    //VB alternative sets of velocity components
    //ROTE alternative sets of rotational energy
    //EA entry area
    //INC counting increment
    //ELTI  initial electronic temperature
    //XPREV the pevious x coordinate
    //
    //memget(DMOM,0.e00,sizeof(DMOM));
    for(int i=0;i<4;i++)
        DMOM[i]=0.e00;
    DENG=0.e00;
    //set the number of molecules, divisions etc. based on stream 1
    //
    NMI=10000*IMEG+2;    //small changes in number for statistically independent runs
    NDIV=NMI/MOLSC; //MOLSC molecules per division
    //WRITE (9,*) 'The number of divisions is',NDIV
    file_9<< "The number of divisions is "<<NDIV<<endl;
    //
    MDIV=NDIV;
    ILEVEL=0;
    //
    i_allocate(ILEVEL+1,MDIV,JDIV);
    // ALLOCATE (JDIV(0:ILEVEL,MDIV),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR JDIV ARRAY',ERROR
    // ENDIF
    //
    DDIV=(XB[2]-XB[1])/double(NDIV);
    NCELLS=NDIV;
    
    //WRITE (9,*) 'The number of sampling cells is',NCELLS
    file_9<<"The number of sampling cells is "<< NCELLS<<endl;
    NCIS=MOLSC/NMCC;
    NCCELLS=NCIS*NDIV;
    //WRITE (9,*) 'The number of collision cells is',NCCELLS
    file_9<< "The number of collision cells is "<<NCCELLS<<endl;
    //
    if(IFX == 0) XS=0.e00;
    //
    if(ISECS == 0){
        if(IFX == 0) FNUM=((XB[2]-XB[1])*FND[1])/double(NMI);
        if(IFX == 1) FNUM=PI*(pow(XB[2],2)-pow(XB[1],2))*FND[1]/double(NMI);
        if(IFX == 2) FNUM=1.3333333333333333333333e00*PI*(pow(XB[2],3)-pow(XB[1],3))*FND[1]/double(NMI);
    }
    else{
        if(IFX == 0) FNUM=((XS-XB[1])*FND[1]+(XB[2]-XS)*FND[2])/double(NMI);
        if(IFX == 1) FNUM=PI*((pow(XS,2)-pow(XB[1],2))*FND[1]+(pow(XB[2],2)-pow(XS,2))*FND[2])/double(NMI);
        if(IFX == 2) FNUM=1.3333333333333333333333e00*PI*((pow(XS,3)-pow(XB[1],3))*FND[1]+(pow(XB[2],3)-pow(XS,3))*FND[2])/double(NMI);
    }
    //
    FNUM=FNUM*FNUMF;
    if(FNUM < 1.e00) FNUM=1.e00;
    //
    FTIME=0.e00;
    //
    TOTMOV=0.e00;
    TOTCOL=0.e00;
    
    NDISSOC=0;
    //memget(TCOL,0.e00,sizeof(*TCOL));//TCOL=0.e00;
    for(int i=0;i<MSP+1;i++){
        for(int j=0;j<MSP+1;j++){
            get (TCOL , i,j)=0.e00;
        }
    }
    
    //memget(TDISS,0.e00,sizeof(*TDISS));//TDISS=0.e00;
    //memget(TRECOMB,0.e00,sizeof(*TRECOMB));//TRECOMB=0.e00;
    for(int i=0;i<MSP+1;i++)
        TDISS[i]=0.e00;
    for(int i=0;i<MSP+1;i++)
        TRECOMB[i]=0.e00;
    //TREACG=0;
    //TREACL=0;
    for(int i=0;i<5;i++){
        for(int j=0;j<MSP+1;j++){
            get (TREACG , i,j)=0;
        }
    }
    for(int i=0;i<5;i++){
        for(int j=0;j<MSP+1;j++){
            get (TREACL , i,j)=0;
        }
    }
    //memget(TNEX,0.e00,sizeof(*TNEX));//TNEX=0.e00;
    for(int i=0;i<MEX+1;i++)
        TNEX[i]= 0.e00;
    for(N=1;N<=NDIV;N++){
        get(JDIV ,1 , N)=-N;
    }
    
    //
    d_allocate(4,NCELLS,CELL);
    i_allocate(NCELLS,ICELL);
    d_allocate(5,NCCELLS,CCELL);
    i_allocate(3,NCCELLS,ICCELL);
    // ALLOCATE (CELL(4,NCELLS),ICELL(NCELLS),CCELL(5,NCCELLS),ICCELL(3,NCCELLS),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR CELL ARRAYS',ERROR
    // ENDIF
    //
    d_allocate(NCELLS,COLLS);
    d_allocate(NCELLS,WCOLLS);
    d_allocate(NCELLS,CLSEP);
    d_allocate(MNSR,SREAC);
    d_allocate(23,NCELLS,VAR);
    d_allocate(13,NCELLS,MSP,VARSP);
    d_allocate(36+MSP,2,VARS);
    d_allocate(10+MSP,NCELLS,MSP,CS);
    d_allocate(9,2,MSP,2,CSS);
    d_allocate(6,2,CSSS);
    
    // ALLOCATE (COLLS(NCELLS),WCOLLS(NCELLS),CLSEP(NCELLS),SREAC(MNSR),VAR(23,NCELLS),VARSP(0:12,NCELLS,MSP),    &
    //           VARS(0:35+MSP,2),CS(0:9+MSP,NCELLS,MSP),CSS(0:8,2,MSP,2),CSSS(6,2),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR SAMPLING ARRAYS',ERROR
    // ENDIF
    //
    if(MMVM >= 0){
        
        d_allocate(MSP,MMVM,151,VIBFRAC);
        d_allocate(MSP,MMVM,SUMVIB);
        // ALLOCATE (VIBFRAC(MSP,MMVM,0:150),SUMVIB(MSP,MMVM),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR RECOMBINATION ARRAYS',ERROR
        // ENDIF
    }
    //
    INITIALISE_SAMPLES();
    //
    //Set the initial cells
    
    for(N=1;N<=NCELLS;N++){
        get (CELL , 2,N)=XB[1]+double(N-1)*DDIV;
        get (CELL , 3,N)=get (CELL , 2,N)+DDIV;
        get (CELL , 1,N)=get (CELL , 2,N)+0.5e00*DDIV;
        if(IFX == 0) get (CELL , 4,N)=get (CELL , 3,N)-get (CELL , 2,N);    //calculation assumes unit cross-section
        if(IFX == 1) get (CELL , 4,N)=PI*(pow(get (CELL , 3,N),2)-pow(get (CELL , 2,N),2));  //assumes unit length of full cylinder
        if(IFX == 2) get (CELL , 4,N)=1.33333333333333333333e00*PI*(pow(get (CELL , 3,N),3)-pow(get (CELL , 2,N),3));    //flow is in the full sphere
        get (ICELL , N)=NCIS*(N-1);
        for(M=1;M<=NCIS;M++){
            L=get (ICELL , N)+M;
            XMIN=get (CELL , 2,N)+(M-1)*DDIV/double(NCIS);
            XMAX=XMIN+DDIV/double(NCIS);
            if(IFX == 0) get (CCELL , 1,L)=XMAX-XMIN;
            if(IFX == 1) get (CCELL , 1,L)=PI*(pow(XMAX,2)-pow(XMIN,2));  //assumes unit length of full cylinder
            if(IFX == 2) get (CCELL , 1,L)=1.33333333333333333333e00*PI*(pow(XMAX,3)-pow(XMIN,3));    //flow is in the full sphere
            get(CCELL , 2,L)=0.e00;
            get(ICCELL , 3,L)=N;
        }
        get(VAR , 11,N)=FTMP[1];
        get(VAR , 8,N)=FTMP[1];
    }
    //
    if(IWF == 0) AWF=1.e00;
    if(IWF == 1){
        //FNUM must be reduced to allow for the weighting factors
        A=0.e00;
        B=0.e00;
        for(N=1;N<=NCELLS;N++){
            A=A+get (CELL , 4,N);
            B=B+get (CELL , 4,N)/(1.0+WFM*pow(get (CELL , 1,N),IFX));
        }
        AWF=A/B;
        FNUM=FNUM*B/A;
    }
    //
    //WRITE (9,*) 'FNUM is',FNUM
    file_9<<"FNUM is "<<FNUM<<endl;
    //
    //set the information on the molecular species
    //
    A=0.e00;
    B=0.e00;
    for(L=1;L<=MSP;L++){
        A=A+get(SP ,5,L)*get(FSP ,L,1);
        B=B+(3.0+get(ISPR ,1,L))*get(FSP ,L,1);
        get (VMP , L,1)=sqrt(2.e00*BOLTZ*FTMP[1]/get(SP ,5,L));
        if((ITYPE[2]== 0) || (ISECS == 1)) get (VMP , L,2)=sqrt(2.e00*BOLTZ*FTMP[2]/get(SP ,5,L));
        VNMAX[L]=3.0*get (VMP , L,1);
        if(L == 1)
            VMPM=get (VMP , L,1);
        else
            if(get (VMP , L,1) > VMPM) VMPM=get (VMP , L,1);
    }
    //WRITE (9,*) 'VMPM =',VMPM
    file_9<< "VMPM = "<<VMPM<<endl;
    FDEN=A*FND[1];
    FPR=FND[1]*BOLTZ*FTMP[1];
    FMA=VFX[1]/sqrt((B/(B+2.e00))*BOLTZ*FTMP[1]/A);
    //set the molecular properties for collisions between unlike molecles
    //to the average of the molecules
    for(L=1;L<=MSP;L++){
        for(M=1;M<=MSP;M++){
            get (SPM , 4,L,M)=0.5e00*(get(SP ,1,L)+get(SP ,1,M));
            get (SPM , 3,L,M)=0.5e00*(get(SP ,3,L)+get(SP ,3,M));
            get (SPM , 5,L,M)=0.5e00*(get(SP ,2,L)+get(SP ,2,M));
            get (SPM , 1,L,M)=get(SP ,5,L)*(get(SP ,5,M)/(get(SP ,5,L)+get(SP ,5,M)));
            get (SPM , 2,L,M)=0.25e00*PI*pow((get(SP ,1,L)+get(SP ,1,M)),2);
            AA=2.5e00-get (SPM , 3,L,M);
            A=tgamma(AA);
            get (SPM , 6,L,M)=1.e00/A;
            get (SPM , 8,L,M)=0.5e00*(get(SP ,4,L)+get(SP ,4,M));
            if((get(ISPR ,1,L) > 0) && (get(ISPR ,1,M) > 0))
                get (SPM , 7,L,M)=(get(SPR ,1,L)+get(SPR ,1,M))*0.5e00;
            if((get(ISPR ,1,L) > 0) && (get(ISPR ,1,M) == 0))
                get (SPM , 7,L,M)=get(SPR ,1,L);
            if((get(ISPR ,1,M) > 0) && (get(ISPR ,1,L) == 0))
                get (SPM , 7,L,M)=get(SPR ,1,M);
        }
    }
    if(MSP == 1){   //set unscripted variables for the simple gas case
        RMAS=get (SPM , 1,1,1);
        CXSS=get (SPM , 2,1,1);
        RGFS=get (SPM , 6,1,1);
    }
    //
    for(L=1;L<=MSP;L++){
        CR[L]=0.e00;
        for(M=1;M<=MSP;M++){   //set the equilibrium collision rates
            CR[L]=CR[L]+2.e00*SPI*pow(get (SPM , 4,L,M),2)*FND[1]*get(FSP ,M,1)*pow((FTMP[1]/get (SPM , 5,L,M)),(1.0-get (SPM , 3,L,M)))*sqrt(2.0*BOLTZ*get (SPM , 5,L,M)/get (SPM , 1,L,M));
        }
    }
    A=0.e00;
    for(L=1;L<=MSP;L++)
        A=A+get(FSP ,L,1)*CR[L];
    CTM=1.e00/A;
    //WRITE (9,*) 'Collision time in the stream is',CTM
    file_9<< "Collision time in the stream is "<<CTM << endl ;
    //
    for(L=1;L<=MSP;L++){
        FP[L]=0.e00;
        for(M=1;M<=MSP;M++){
            FP[L]=FP[L]+PI*pow(get (SPM , 4,L,M),2)*FND[1]*get(FSP ,M,1)*pow((FTMP[1]/get (SPM , 5,L,M)),(1.0-get (SPM , 3,L,M)))*sqrt(1.e00+get(SP ,5,L)/get(SP ,5,M));
        }
        FP[L]=1.e00/FP[L];
    }
    FPM=0.e00;
    for(L=1;L<=MSP;L++)
        FPM=FPM+get(FSP ,L,1)*FP[L];
    //WRITE (9,*) 'Mean free path in the stream is',FPM
    file_9<<"Mean free path in the stream is "<<FPM<<endl;
    //
    TNORM=CTM;
    if(ICLASS == 1) TNORM= (XB[2]-XB[1])/VMPM;     //there may be alternative definitions
    //
    //set the initial time step
    DTM=CTM*CPDTM;
    //
    if(fabs(VFX[1]) > 1.e-6)
        A=(0.5e00*DDIV/VFX[1])*TPDTM;
    else
        A=0.5e00*DDIV/VMPM;
    
    if(IVB == 1){
        B=0.25e00*DDIV/(fabs(VELOB)+VMPM);
        if(B < A) A=B;
    }
    if(DTM > A) DTM=A;
    //
    DTM=0.1e00*DTM;   //OPTIONAL MANUAL ADJUSTMENT that is generally used with a fixed time step (e.g for making x-t diagram)
    //
    DTSAMP=SAMPRAT*DTM;
    DTOUT=OUTRAT*DTSAMP;
    TSAMP=DTSAMP;
    TOUT=DTOUT;
    ENTMASS=0.0;
    //
    //WRITE (9,*) 'The initial value of the overall time step is',DTM
    file_9<< "The initial value of the overall time step is "<<DTM<<endl;
    //
    //initialise cell quantities associated with collisions
    //
    for(N=1;N<=NCCELLS;N++){
        get (CCELL , 3,N)=DTM/2.e00;
        get (CCELL , 4,N)=2.e00*VMPM*get (SPM , 2,1,1);
        RANF=(double) (rand()%100000)/100001;
        // RANDOM_NUMBER(RANF)
        get (CCELL , 2,N)=RANF;
        get (CCELL , 5,N)=0.e00;
    }
    //
    //set the entry quantities
    //
    for(K=1;K<=2;K++){
        if((ITYPE[K] == 0) || ((K == 2) && (ITYPE[K] == 4))){
            if(IFX == 0) EA=1.e00;
            if(IFX == 1) EA=2.e00*PI*XB[K];
            if(IFX == 2) EA=4.e00*PI*pow(XB[K],2);
            for(L=1;L<=MSP;L++){
                if(K == 1) SN=VFX[1]/get (VMP , L,1);
                if(K == 2) SN=-VFX[2]/get (VMP , L,2);
                AA=SN;
                A=1.e00+erf(AA);
                BB=exp(-pow(SN,2));
                get (ENTR , 3,L,K)=SN;
                get (ENTR , 4,L,K)=SN+sqrt(pow(SN,2)+2.e00);
                get (ENTR , 5,L,K)=0.5e00*(1.e00+SN*(2.e00*SN-get (ENTR , 4,L,K)));
                get (ENTR , 6,L,K)=3.e00*get (VMP , L,K);
                B=BB+SPI*SN*A;
                get (ENTR , 1,L,K)=EA*FND[K]*get(FSP ,L,K)*get (VMP , L,K)*B/(FNUM*2.e00*SPI);
                get (ENTR , 2,L,K)=0.e00;
            }
        }
    }
    //
    //Set the uniform stream
    //
    MNM=1.1e00*NMI;
    //
    if(MMVM > 0){
        d_allocate(NCLASS,MNM,PX);
        d_allocate(MNM,PTIM);
        d_allocate(MNM,PROT);
        i_allocate(MNM,IPCELL);
        i_allocate(MNM,IPSP);
        i_allocate(MNM,ICREF);
        i_allocate(MNM,IPCP);
        d_allocate(3,MNM,PV);
        i_allocate(MMVM,MNM,IPVIB);
        d_allocate(MNM,PELE);
        // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),     &
        //      get(IPVIB  ,MMVM,MNM),PELE(MNM),STAT=ERROR)
    }
    
    else{
        if(MMRM > 0){
            d_allocate(NCLASS,MNM,PX);
            d_allocate(MNM,PTIM);
            d_allocate(MNM,PROT);
            i_allocate(MNM,IPCELL);
            i_allocate(MNM,IPSP);
            i_allocate(MNM,ICREF);
            i_allocate(MNM,IPCP);
            d_allocate(3,MNM,PV);
            d_allocate(MNM,PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
        else{
            d_allocate(NCLASS,MNM,PX);
            d_allocate(MNM,PTIM);
            i_allocate(MNM,IPCELL);
            i_allocate(MNM,IPSP);
            i_allocate(MNM,ICREF);
            i_allocate(MNM,IPCP);
            d_allocate(3,MNM,PV);
            d_allocate(MNM,PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR MOLECULE ARRAYS',ERROR
    // ENDIF
    //
    NM=0;
    if(IGS == 1){
        cout<<"Setting the initial gas"<<endl;
        for(L=1;L<=MSP;L++){
            //memget(ROTE,0.0,sizeof(ROTE));
            for(int i=0;i<3;i++)
                ROTE[i]=0.0;
            for(K=1;K<=ISECS+1;K++){
                if(ISECS == 0){         //no secondary stream
                    M=(double(NMI)*get(FSP ,L,1)*AWF);
                    XMIN=XB[1];
                    XMAX=XB[2];
                }
                else{
                    A=(pow(XS,JFX)-pow(XB[1],JFX))*FND[1]+(pow(XB[2],JFX)-pow(XS,JFX))*FND[2];
                    if(K == 1){
                        M=int(double(NMI)*((pow(XS,JFX)-pow(XB[1],JFX))*FND[1]/A)*get(FSP ,L,1));
                        XMIN=XB[1];
                        XMAX=XS;
                    }
                    else{
                        M=int(double(NMI)*((pow(XB[2],JFX)-pow(XS,JFX))*FND[2]/A)*get(FSP ,L,2));
                        XMIN=XS;
                        XMAX=XB[2];
                    }
                }
                if((K == 1) || (ISECS == 1)){
                    III=0;
                    WFMIN=1.e00+WFM*pow(XB[1],IFX);
                    N=1;
                    INC=1;
                    if((K== 2) && (JFX > 1)){
                        BBB=(pow(XMAX,JFX)-pow(XMIN,JFX))/double(M);
                        XPREV=XMIN;
                    }
                    while(N < M){
                        if((JFX == 1) || (K == 1))
                            A=pow((pow(XMIN,JFX)+(((N)-0.5e00)/(M))*pow((XMAX-XMIN),JFX)),(1.e00/double(JFX)));
                        else{
                            A=pow((pow(XPREV,JFX)+BBB),(1.e00/double(JFX)));
                            XPREV=A;
                        }
                        if(IWF == 0)
                            B=1.e00;
                        else{
                            B=WFMIN/(1.e00+WFM*pow(A,IFX));
                            if((B < 0.1e00) && (INC == 1)) INC=10;
                            if((B < 0.01e00) && (INC == 10)) INC=100;
                            if((B < 0.001e00) && (INC == 100)) INC=1000;
                            if((B < 0.0001e00) && (INC == 1000)) INC=10000;
                        }
                        RANF=((double)rand()/(double)RAND_MAX);
                        // CALL RANDOM_NUMBER(RANF)
                        if(B*double(INC) > RANF){
                            NM=NM+1;
                            get (PX , 1,NM)=A;
                            get(IPSP ,NM)=L;
                            PTIM[NM]=0.0;
                            if(IVB == 0) FIND_CELL_1D(get (PX , 1,NM),get(IPCELL  ,NM),KK);
                            if(IVB == 1) FIND_CELL_MB_1D(get (PX , 1,NM),get(IPCELL  ,NM),KK,PTIM[NM]);
                            //
                            for(NSET=1;NSET<=2;NSET++){
                                for(KK=1;KK<=3;KK++){
                                    RVELC(A,B,get (VMP , L,K));
                                    if(A < B){
                                        if(DMOM[KK] < 0.e00)
                                            BB=B;
                                        else
                                            BB=A;
                                    }           
                                    else{
                                        if(DMOM[KK] < 0.e00)
                                            BB=A;
                                        else
                                            BB=B;
                                    }
                                    VB[KK][NSET]=BB;
                                }
                                if(get(ISPR ,1,L) > 0) SROT(L,FTMP[K],ROTE[NSET]);
                            }
                            A=(0.5e00*get(SP ,5,L)*(pow(VB[1][1],2)+pow(VB[2][1],2)+pow(VB[3][1],2))+ROTE[1])/(0.5e00*BOLTZ*FTMP[K])-3.e00-double(get(ISPR ,1,L));
                            B=(0.5e00*get(SP ,5,L)*(pow(VB[1][2],2)+pow(VB[2][2],2)+pow(VB[3][2],2))+ROTE[2])/(0.5e00*BOLTZ*FTMP[K])-3.e00-double(get(ISPR ,1,L));
                            if(A < B){
                                if(DENG < 0.e00)
                                    KN=2;
                                else
                                    KN=1;
                            }
                            else{
                                if(DENG < 0.e00)
                                    KN=1;
                                else
                                    KN=2;
                            }
                            
                            for(KK=1;KK<=3;KK++){
                                get(PV  ,KK,NM)=VB[KK][KN];
                                DMOM[KK]=DMOM[KK]+VB[KK][KN];
                            }
                            get(PV  ,1,NM)=get(PV  ,1,NM)+VFX[K];
                            get(PV  ,2,NM)=get(PV  ,2,NM)+VFY[K];
                            if(get(ISPR ,1,L) > 0) PROT[NM]=ROTE[KN];
                            //           PROT(NM)=0.d00       //uncomment for zero initial rotational temperature (Figs. 6.1 and 6.2)
                            if(KN == 1) DENG=DENG+A;
                            if(KN == 2) DENG=DENG+B;
                            if(MMVM > 0){
                                if(get(ISPV  ,L) > 0){
                                    for(J=1;J<=get(ISPV  ,L);J++)
                                        SVIB(L,FVTMP[K],get (IPVIB , J,NM),J);
                                }
                                ELTI=FVTMP[K];
                                if(MELE > 1) SELE(L,ELTI,PELE[NM]);
                            }
                        }
                        N=N+INC;
                    }
                }
            }
        }
        //
        //WRITE (9,*) 'DMOM',DMOM
        //WRITE (9,*) 'DENG',DENG
        file_9<<"DMOM "<<DMOM[1] << "\t" << DMOM[2] << "\t" << DMOM[3] << endl;
        file_9<<"DENG "<< DENG <<endl;
    }
    //
    NMI=NM;
    //
    
    //SPECIAL CODING FOR INITIATION OF COMBUSION IN H2-02 MIXTURE (FORCED IGNITION CASES in section 6.7)
    //set the vibrational levels of A% random molecules to 5
    //  A=0.05D00
    //  M=0.01D00*A*NM
    //  DO N=1,M
    //    CALL RANDOM_NUMBER(RANF)
    //    K=INT(RANF*DFLOAT(NM))+1
    //    get(IPVIB  ,1,K)=5
    //  END DO
    //
    SAMPLE_FLOW();
    //OUTPUT_RESULTS();
    TOUT=TOUT-DTOUT;
    return;
}

void RVELC(double &U,double &V,double &VMP)
{
    //CALC calc;
    //generates two random velocity components U and V in an equilibrium
    //gas with most probable speed VMP
    //based on equations (4.4) and (4.5)
    double A,B;
    //
    // CALL RANDOM_NUMBER(RANF)
    RANF=((double)rand()/(double)RAND_MAX) ;
    A=sqrt(-log(RANF)) ;
    // CALL RANDOM_NUMBER(RANF)
    RANF=((double)rand()/(double)RAND_MAX) ;
    B=DPI*RANF ;
    U=A*sin(B)*VMP ;
    V=A*cos(B)*VMP ;
    return ;
}

void SROT(int &L,double &TEMP,double &ROTE)
{
    //sets a typical rotational energy ROTE of species L
    //CALC calc;
    //GAS gas;
    //
    // IMPLICIT NONE
    //
    int I;
    double A,B,ERM;
    //
    if(get(ISPR ,1,L) == 2){
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        ROTE=-log(RANF)*BOLTZ*TEMP;   //equation (4.8)
    }
    else{
        A=0.5e00*get(ISPR ,1,L)-1.e00;
        I=0;
        while(I == 0){
            // CALL RANDOM_NUMBER(RANF)
            RANF=((double)rand()/(double)RAND_MAX);
            ERM=RANF*10.e00;
            //there is an energy cut-off at 10 kT
            B=(pow((ERM/A),A))*exp(A-ERM);      //equation (4.9)
            // CALL RANDOM_NUMBER(RANF)
            RANF=((double)rand()/(double)RAND_MAX);
            if(B > RANF) I=1;
        }
        ROTE=ERM*BOLTZ*TEMP;
    }
    return;
}

void SVIB(int &L,double &TEMP,int &IVIB, int &K)
{
    //sets a typical vibrational state at temp. TEMP of mode K of species L
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int N;
    //    double TEMP;
    //    int IVIB;
    //
    // CALL RANDOM_NUMBER(RANF)
    RANF=((double)rand()/(double)RAND_MAX);
    N=-log(RANF)*TEMP/get(SPVM ,1,K,L);                 //eqn(4.10)
    //the state is truncated to an integer
    IVIB=N;
}

void SELE(int &L,double &TEMP, double &ELE)
{
    //sets a typical electronic energy at temp. TEMP of species L
    //employs direct sampling from the Boltzmann distribution
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,N;
    double EPF,A,B;
    double CTP[20];
    //
    //ELE electronic energy of a molecule
    //EPF electronic partition function
    //CTP(N) contribution of electronic level N to the electronic partition function
    //
    if(TEMP > 0.1){
        EPF=0.e00;
        for(N=1;N<=get(NELL  ,L);N++)
            EPF=EPF+get(QELC ,1,N,L)*exp(-get(QELC ,2,N,L)/TEMP) ;
        //
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        //
        A=0.0;
        K=0; //becomes 1 when the energy is set
        N=0;  //level
        while(K == 0){
            N=N+1;
            A=A+get(QELC ,1,N,L)*exp(-get(QELC ,2,N,L)/TEMP);
            B=A/EPF;
            if(RANF < B){
                K=1;
                ELE=BOLTZ*get(QELC ,2,N,L);
            }
        }
    }
    else
        ELE=0.e00;
    
    //
}

void CQAX(double &A,double &X,double &GAX)
{
    //calculates the function Q(a,x)=Gamma(a,x)/Gamma(a)
    //
    // IMPLICIT NONE
    double G,DT,T,PV,V;
    int NSTEP,N;
    //
    G=tgamma(A);
    //
    if(X < 10.e00){       //direct integration
        NSTEP=100000;
        DT=X/double(NSTEP);
        GAX=0.e00;
        PV=0.e00;
        for(N=1;N<=NSTEP;N++){
            T=double(N)*DT;
            V=exp(-T)*pow(T,(A-1));
            GAX=GAX+(PV+V)*DT/2.e00;
            PV=V;
        }
        GAX=1.e00-GAX/G;
    }
    else{      //asymptotic formula
        GAX=pow(X,(A-1.e00))*exp(-X)*(1.0+(A-1.e00)/X+(A-1.e00)*(A-2.e00)/pow(X,2)+(A-1.e00)*(A-2.e00)*(A-3.e00)/pow(X,3)+(A-1.e00)*(A-2.e00)*(A-3.e00)*(A-4.e00)/pow(X,4));
        GAX=GAX/G;
    }
    //
    return;
}
//****
//
void LBS(double XMA,double XMB,double &ERM)
{
    //selects a Larsen-Borgnakke energy ratio using eqn (11.9)
    //
    double PROB,RANF;
    int I,N;
    //
    //I is an indicator
    //PROB is a probability
    //ERM ratio of rotational to collision energy
    //XMA degrees of freedom under selection-1
    //XMB remaining degrees of freedom-1
    //
    I=0;
    while(I == 0){
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        ERM=RANF;
        if((XMA < 1.e-6) || (XMB < 1.e-6)){
            //    IF (XMA < 1.E-6.AND.XMB < 1.E-6) RETURN
            //above can never occur if one mode is translational
            if(XMA < 1.e-6) PROB=pow((1.e00-ERM),XMB);
            if(XMB < 1.e-6) PROB=pow((1.e00-ERM),XMA);
        }
        else
            PROB=pow(((XMA+XMB)*ERM/XMA),XMA)*pow(((XMA+XMB)*(1.e00-ERM)/XMB),XMB);
        
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        if(PROB > RANF) I=1;
    }
    //
    return;
}

void FIND_CELL_1D(double &X,int &NCC,int &NSC)
{
    //find the collision and sampling cells at a givem location in a 0D or 1D case
    //MOLECS molecs;
    //GEOM_1D geom;
    //CALC calc;
    
    int N,L,M,ND;
    double FRAC,DSC;
    //
    //NCC collision cell number
    //NSC sampling cell number
    //X location
    //ND division number
    //DSC the ratio of the sub-division width to the division width
    //
    ND=(X-XB[1])/DDIV+0.99999999999999e00 ;
    //
    if(get(JDIV ,1,ND) < 0){    //the division is a level 0 (no sub-division) sampling cell
        NSC=-get(JDIV ,1,ND);
        //  IF (IFX == 0)
        NCC=NCIS*(X-get (CELL , 2,NSC))/(get (CELL , 3,NSC)-get (CELL , 2,NSC))+0.9999999999999999e00;
        NCC=NCC+get (ICELL , NSC);
        //  IF (NCC == 0) NCC=1
        return;
    }
    else{  //the molecule is in a subdivided division
        FRAC=(X-XB[1])/DDIV-double(ND-1);
        M=ND;
        for(N=1;N<=ILEVEL;N++){
            DSC=1.e00/double(N+1);
            for(L=1;L<=2;L++){  //over the two level 1 subdivisions
                if(((L == 1) && (FRAC < DSC)) || ((L == 2) || (FRAC >= DSC))){
                    M=get(JDIV ,N,M)+L;  //the address in JDIV
                    if(get(JDIV ,N+1,M) < 0){
                        NSC=-get(JDIV ,N+1,M);
                        NCC=NCIS*(X-get (CELL , 2,NSC))/(get (CELL , 3,NSC)-get (CELL , 2,NSC))+0.999999999999999e00;
                        if(NCC == 0) NCC=1;
                        NCC=NCC+get (ICELL , NSC);
                        return;
                    }
                }
            }
            FRAC=FRAC-DSC;
        }
    }
   // file_9<<"No cell for molecule at x= "<<X<<endl; // dsuedit
    return ;
}

void FIND_CELL_MB_1D(double &X,int &NCC,int &NSC,double &TIM)
{
    //find the collision and sampling cells at a givem location in a 0D or 1D case
    //when there is a moving boundary
    //MOLECS molecs;
    //GEOM_1D geom;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int N,L,M,ND;
    double FRAC,DSC,A,B,C;
    //
    //NCC collision cell number
    //NSC sampling cell number
    //X location
    //ND division number
    //DSC the ratio of the sub-division width to the division width
    //TIM the time
    //
    A=(XB[2]+VELOB*TIM-XB[1])/double(NDIV);      //new DDIV
    ND=(X-XB[1])/A+0.99999999999999e00;
    B=XB[1]+double(ND-1)*A;
    //
    //the division is a level 0 sampling cell
    NSC=-get(JDIV ,1,ND);
    NCC=NCIS*(X-B)/A+0.99999999999999e00;
    NCC=NCC+get (ICELL , NSC);
    
    //WRITE (9,*) 'No cell for molecule at x=',X
    file_9<< "No cell for molecule at x= "<<X<<endl;
    return;
    //return ;
    //
}

void REFLECT_1D(int &N,int J,double &X)
{
    //reflects molecule N and samples the surface J properties
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int L,K,M;
    double A,B,VMPS,DTR,XI,DX,DY,DZ,WF;
    //
    //VMPS most probable velocity at the surface temperature
    //DTR time remaining after molecule hits a surface
    //
    L=get(IPSP ,N);
    WF=1.e00;
    if(IWF == 1) WF=1.e00+WFM*pow(X,IFX);
    get (CSS , 1+ 0,J,L,1)=get (CSS , 1+ 0,J,L,1)+1.e00;
    get (CSS , 1+ 1,J,L,1)=get (CSS , 1+ 1,J,L,1)+WF;
    get (CSS , 1+ 2,J,L,1)=get (CSS , 1+ 2,J,L,1)+WF*get(PV  ,1,N)*get(SP ,5,L);
    get (CSS , 1+ 3,J,L,1)=get (CSS , 1+ 3,J,L,1)+WF*(get(PV  ,2,N)-VSURF[J])*get(SP ,5,L);
    get (CSS , 1+ 4,J,L,1)=get (CSS , 1+ 4,J,L,1)+WF*get(PV  ,3,N)*get(SP ,5,L);
    A=pow(get(PV  ,1,N),2)+pow((get(PV  ,2,N)-VSURF[J]),2)+pow(get(PV  ,3,N),2);
    get (CSS , 1+ 5,J,L,1)=get (CSS , 1+ 5,J,L,1)+WF*0.5e00*get(SP ,5,L)*A;
    if(get(ISPR ,1,L) > 0) get (CSS , 1+ 6,J,L,1)=get (CSS , 1+ 6,J,L,1)+WF*PROT[N];
    if(MELE > 1) get (CSS , 1+ 8,J,L,1)=get (CSS , 1+ 8,J,L,1)+WF*PELE[N];
    if(MMVM > 0){
        if(get(ISPV  ,L) > 0){
            for(K=1;K<=get(ISPV  ,L);K++)
                get (CSS , 1+ 7,J,L,1)=get (CSS , 1+ 7,J,L,1)+WF*double(get (IPVIB , K,N))*BOLTZ*get(SPVM ,1,K,L);
        }
    }
    A=pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2)+pow(get(PV  ,3,N),2);
    B=fabs(get(PV  ,1,N));
    get (CSSS , 1,J)=get (CSSS , 1,J)+WF/B;
    get (CSSS , 2,J)=get (CSSS , 2,J)+WF*get(SP ,5,L)/B;
    get (CSSS , 3,J)=get (CSSS , 3,J)+WF*get(SP ,5,L)*get(PV  ,2,N)/B;
    //this assumes that any flow normal to the x direction is in the y direction
    get (CSSS , 4,J)=get (CSSS , 4,J)+WF*get(SP ,5,L)*A/B;
    if(get(ISPR ,1,L) > 0){
        get (CSSS , 5,J)=get (CSSS , 5,J)+WF*PROT[N]/B;
        get (CSSS , 6,J)=get (CSSS , 6,J)+WF*get(ISPR ,1,L)/B;
    }
    //
    // CALL RANDOM_NUMBER(RANF)
    RANF=((double)rand()/(double)RAND_MAX);
    if(FSPEC[J] > RANF){      //specular reflection
        X=2.e00*XB[J]-X;
        get(PV  ,1,N)=-get(PV  ,1,N);
        DTR=(X-XB[J])/get(PV  ,1,N);
    }
    else{                         //diffuse reflection
        VMPS=sqrt(2.e00*BOLTZ*TSURF[J]/get(SP ,5,L));
        DTR=(XB[J]-get (PX , 1,N))/get(PV  ,1,N);
        // CALL RANDOM_NUMBER(RANF)
        
        RANF=((double)rand()/(double)RAND_MAX);
        get(PV  ,1,N)=sqrt(-log(RANF))*VMPS;
        
        if(J == 2) get(PV  ,1,N)=-get(PV  ,1,N);
        RVELC(get(PV  ,2,N),get(PV  ,3,N),VMPS);
        get(PV  ,2,N)=get(PV  ,2,N)+VSURF[J];
        if(get(ISPR ,1,L) > 0) SROT(L,TSURF[J],PROT[N]);
        if(MMVM > 0){
            for(K=1;K<=get(ISPV  ,L);K++)
                SVIB(L,TSURF[J],get (IPVIB , K,N),K);
        }
        if(MELE > 1) SELE(L,TSURF[J],PELE[N]);
    }
    //
    get (CSS , 1+ 2,J,L,2)=get (CSS , 1+ 2,J,L,2)-WF*get(PV  ,1,N)*get(SP ,5,L);
    get (CSS , 1+ 3,J,L,2)=get (CSS , 1+ 3,J,L,2)-WF*(get(PV  ,2,N)-VSURF[J])*get(SP ,5,L);
    get (CSS , 1+ 4,J,L,2)=get (CSS , 1+ 4,J,L,2)-WF*get(PV  ,3,N)*get(SP ,5,L);
    A=pow(get(PV  ,1,N),2)+pow((get(PV  ,2,N)-VSURF[J]),2)+pow(get(PV  ,3,N),2);
    get (CSS , 1+ 5,J,L,2)=get (CSS , 1+ 5,J,L,2)-WF*0.5e00*get(SP ,5,L)*A;
    if(get(ISPR ,1,L) > 0) get (CSS , 1+ 6,J,L,2)=get (CSS , 1+ 6,J,L,2)-WF*PROT[N];
    if(MELE > 1) get (CSS , 1+ 8,J,L,2)=get (CSS , 1+ 8,J,L,2)-WF*PELE[N];
    if(MMVM > 0){
        if(get(ISPV  ,L) > 0){
            for(K=1;K<=get(ISPV  ,L);K++)
                get (CSS , 1+ 7,J,L,2)=get (CSS , 1+ 7,J,L,2)-WF*double(get (IPVIB , K,N))*BOLTZ*get(SPVM ,1,K,L);
        }
    }
    A=pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2)+pow(get(PV  ,3,N),2);
    B=fabs(get(PV  ,1,N));
    get (CSSS , 1,J)=get (CSSS , 1,J)+WF/B;
    get (CSSS , 2,J)=get (CSSS , 2,J)+WF*get(SP ,5,L)/B;
    get (CSSS , 3,J)=get (CSSS , 3,J)+WF*get(SP ,5,L)*get(PV  ,2,N)/B;
    //this assumes that any flow normal to the x direction is in the y direction
    get (CSSS , 4,J)=get (CSSS , 4,J)+WF*get(SP ,5,L)*A/B;
    if(get(ISPR ,1,L) > 0){
        get (CSSS , 5,J)=WF*get (CSSS , 5,J)+PROT[N]/B;
        get (CSSS , 6,J)=get (CSSS , 6,J)+WF*get(ISPR ,1,L)/B;
    }
    //
    XI=XB[J];
    DX=DTR*get(PV  ,1,N);
    DZ=0.e00;
    if(IFX > 0) DY=DTR*get(PV  ,2,N);
    if(IFX == 2) DZ=DTR*get(PV  ,3,N);
    if(IFX == 0) X=XI+DX;
    if(IFX > 0) AIFX(XI,DX,DY,DZ,X,get(PV  ,1,N),get(PV  ,2,N),get(PV  ,3,N));
    //
    return;
}

void DERIVED_GAS_DATA()
{
    //
    //GAS gas;
    //CALC calc;
    int I,II,J,JJ,K,L,M,MM,N,JMAX,MOLSP,MOLOF,NSTEP,IMAX;
    double A,B,BB,C,X,T,CUR,EAD,TVD,ZVT,ERD,PETD,DETD,PINT,ETD,SUMD,VAL;
    double *BFRAC,*TOT;
    double *VRRD;
    double *VRREX;
    //
    //VRRD(1,L,M,K) dissociation rate coefficient to species L,M for vibrational level K at 5,000 K
    //VRRD(2,L,M,K) similar for 15,000 K
    //VRREX(1,J,L,M,K)  Jth exchange rate coefficient to species L,M for vibrational level K at 1,000 K
    //VRREX(2,J,L,M,K) similar for 3,000 K
    //BFRAC(2,J) Boltzmann fraction
    //JMAX imax-1
    //T temperature
    //CUR sum of level resolved rates
    //
    
    d_allocate(2,MSP,MSP, MVIBL+1 , VRRD ) ;
    d_allocate(MVIBL+1,2 ,BFRAC ) ;
    d_allocate(2 ,MMEX , MSP,MSP, MVIBL+1 , VRREX ) ;
    d_allocate(MVIBL+1 , 2 , TOT ) ;
    
    // ALLOCATE (VRRD(2,MSP,MSP,0:MVIBL),BFRAC(0:MVIBL,2),VRREX(2,MMEX,MSP,MSP,0:MVIBL),TOT(0:MVIBL,2),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE VIB. RES. DISS. RATES',ERROR
    // END IF
    //
    cout<<"Setting derived gas data"<<endl;
    //copy the L,M data that has been specified for L < M so that it applies also for M>L
    for(L=1;L<=MSP;L++){
        for(M=1;M<=MSP;M++){
            if(L > M){
                get(NSPEX  ,L,M)=get(NSPEX  ,M,L);
                get(ISPRC ,L,M)=get(ISPRC ,M,L);
                get(ISPRK ,L,M)=get(ISPRK ,M,L);
                for(K=1;K<=MSP;K++){
                    get( SPRT ,1,L,M)=get( SPRT ,1,M,L);
                    get( SPRT ,2,L,M)=get( SPRT ,2,M,L);
                    get( SPRC ,1,L,M,K)=get( SPRC ,1,M,L,K);
                    get( SPRC ,2,L,M,K)=get( SPRC ,2,M,L,K);
                }
                for(K=1;K<=MMEX;K++){
                    get(NEX  ,K,L,M)=get(NEX  ,K,M,L);
                    for(J=1;J<=6;J++){
                        get(SPEX  ,J,K,L,M)=get(SPEX  ,J,K,M,L);
                    }
                    for(J=1;J<=7;J++){
                        get(ISPEX  ,K,J,L,M)=get(ISPEX  ,K,J,M,L);
                    }
                }
            }
        }
    }
    //
    if(MMVM > 0){
        //set the characteristic dissociation temperatures
        for(L=1;L<=MSP;L++){
            if(get(ISPV  ,L) > 0){
                for(K=1;K<=get(ISPV  ,L);K++)
                {
                    I=get( ISPVM ,1,K,L);
                    J=get( ISPVM ,2,K,L);
                    get(SPVM ,4,K,L)=(get(SP ,6,I)+get(SP ,6,J)-get(SP ,6,L))/BOLTZ;
                    //WRITE (9,*) 'Char. Diss temp of species',L,' is',SPVM(4,K,L)
                    file_9<<"Char. Diss temp of species "<<L<<" is "<<get(SPVM ,4,K,L)<<endl;
                }
            }
        }
    }
    //
    if(MMEX > 0){
        //set the heats of reaction of the exchange and chain reactions
        for(L=1;L<=MSP;L++){
            for(M=1;M<=MSP;M++){
                for(J=1;J<=MMEX;J++){
                    if((get(ISPEX  ,J,3,L,M)> 0) && (get(ISPEX  ,J,4,L,M)>0) && (get(ISPEX  ,J,1,L,M)>0) && (get(ISPEX  ,J,2,L,M)>0)){
                        get(SPEX  ,3,J,L,M)=get(SP ,6,get(ISPEX  ,J,1,L,M))+get(SP ,6,get(ISPEX  ,J,2,L,M))-get(SP ,6,get(ISPEX  ,J,3,L,M))-get(SP ,6,get(ISPEX  ,J,4,L,M));
                        // WRITE (9,*) 'Reaction',NEX(J,L,M),' heat of reaction',SPEX(3,J,L,M)
                        file_9<<"Reaction "<<get(NEX  ,J,L,M)<<" heat of reaction"<<get(SPEX  ,3,J,L,M)<<endl;
                    }
                }
            }
        }
    }
    //
    if(MELE > 1){
        //set the electronic cross-section ratios to a mean electronic relaxation collision number
        //(equipartition is not achieved unless there is a single number)
        for(L=1;L<=MSP;L++){
            A=0.e00;
            for(K=1;K<=get(NELL  ,L);K++){
                A=A+get(QELC ,3,K,L);
            }
            get(QELC ,3,1,L)=A/double(get(NELL  ,L));
        }
    }
    //
    //set the cumulative distributions of the post-recombination vibrational distributions for establishment of detailed balance
    for(L=1;L<=MSP;L++){
        for(M=1;M<=MSP;M++){
            if(get(ISPRC ,L,M) > 0){
                N=get(ISPRC ,L,M);   //recombined species
                K=get(ISPRK ,L,M);   //relevant vibrational mode
                //WRITE (9,*) 'SPECIES',L,M,' RECOMBINE TO',N
                file_9<<"SPECIES "<<L<<" "<<M<<" RECOMBINE TO"<<N<<endl;
                JMAX=get(SPVM ,4,K,N)/get(SPVM ,1,K,N);
                if(JMAX > MVIBL){
                    cout<<" The variable MVIBL="<<MVIBL<<" in the gas database must be increased to"<<JMAX<<endl;
                    cout<<"Enter 0 ENTER to stop";
                    cin>> A;
                    return ;
                }
                A=2.5e00-get(SP ,3,N);
                for(I=1;I<=2;I++){
                    if(I == 1) T=get( SPRT ,1,L,M);
                    if(I == 2) T=get( SPRT ,2,L,M);
                    //WRITE (9,*) 'TEMPERATURE',T
                    file_9<<"TEMPERATURE "<<T<<endl;
                    CUR=0.e00;
                    for(J=0;J<=JMAX;J++){
                        X=double(JMAX+1-J)*get(SPVM ,1,K,N)/T;
                        CQAX(A,X,B);
                        get (VRRD , I,L,M,J+1)=B*exp(-double(J)*get(SPVM ,1,K,N)/T);
                        CUR=CUR+get (VRRD , I,L,M,J+1);
                    }
                    B=0.e00;
                    for(J=0;J<=JMAX;J++){
                        B=B+get (VRRD , I,L,M,J+1)/CUR;
                        get (SPRP , I,L,M,J+1)=B;
                        //WRITE (9,*) 'CDF level dissoc',J,SPRP(I,L,M,J)
                        file_9<< "CDF level dissoc "<<J<<" "<<get (SPRP , I,L,M,J+1);
                    }
                }
            }
        }
    }
    //
    //READ (*,*)  //optionally pause program to check cumulative distributions for exchange and chain reactions
    //
    //set the cumulative distributions of the post-reverse vibrational distributions for establishment of detailed balance
    for(L=1;L<=MSP;L++){
        for(M=1;M<=MSP;M++){
            if(get(NSPEX  ,L,M) > 0){
                for(K=1;K<=get(NSPEX  ,L,M);K++){
                    if(get(SPEX  ,3,K,L,M) > 0.e00){         //exothermic (reverse) exchange reaction
                        //L,M are the species in the reverse reaction, E_a of forward reaction is SPEX(3,K,L,M)
                        //WRITE (9,*) 'SPECIES',L,M,' REVERSE REACTION'
                        file_9<<"SPECIES "<<L<<" "<<M<<" REVERSE REACTION"<<endl;
                        MOLSP=get(ISPEX  ,K,3,L,M);  //molecuke that splits in the forward reaction
                        MOLOF=get(ISPEX  ,K,4,L,M);
                        JMAX=(get(SPEX  ,3,K,L,M)+get(SPEX  ,6,K,MOLSP,MOLOF))/(BOLTZ*get(SPVM ,1,get(ISPEX  ,K,5,L,M),MOLSP))+15;   //should always be less than the JMAX set by dissociation reactions
                        for(I=1;I<=2;I++){
                            if(I == 1) T=get(SPEX  ,4,K,L,M);
                            if(I == 2) T=get(SPEX  ,5,K,L,M);
                            for(J=0;J<=JMAX;J++){
                                EAD=(get(SPEX  ,3,K,L,M)+get(SPEX  ,6,K,MOLSP,MOLOF))/(BOLTZ*T);
                                TVD=get(SPVM ,1,get(ISPEX  ,K,5,L,M),MOLSP)/T;
                                ZVT=1.e00/(1.e00-exp(-TVD));
                                C=ZVT/(tgamma(2.5e00-get(SP ,3,MOLSP))*exp(-EAD));  //coefficient of integral
                                ERD=EAD-double(J)*TVD;
                                if(ERD < 0.e00) ERD=0.e00;
                                PETD=ERD;
                                DETD=0.01e00;
                                PINT=0.e00;  //progressive value of integral
                                NSTEP=0;
                                A=1.e00;
                                while(A > 1.e-10){
                                    NSTEP=NSTEP+1;
                                    ETD=PETD+0.5e00*DETD;
                                    SUMD=0.e00;  //normalizing sum in the denominator
                                    IMAX=ETD/TVD+J;
                                    for(II=0;II<=IMAX;II++){
                                        SUMD=SUMD+pow((1.e00-double(II)*TVD/(ETD+double(J)*TVD)),(1.5e00-get(SP ,3,MOLSP)));
                                    }
                                    VAL=(pow((ETD*(1.e00-EAD/(ETD+double(J)*TVD))),(1.5e00-get(SP ,3,MOLSP)))/SUMD)*exp(-ETD);
                                    PINT=PINT+VAL*DETD;
                                    A=VAL/PINT;
                                    PETD=ETD+0.5e00*DETD;
                                }
                                get (VRREX , I,K,L,M,J+1)=C*PINT;
                                //              WRITE (*,*) 'Level ratio exch',I,J,VRREX(I,K,L,M,J)
                            }
                        }
                        //
                        //memget(TOT,0.e00,sizeof(*TOT));//TOT=0.e00;
                        for(int i=0;i<MVIBL+1;i++){
                            for(int j=0;j<MVIBL+1;j++){
                                get (TOT,1+ i,j)=0;
                            }
                        }
                        for(I=1;I<=2;I++){
                            if(I == 1) T=get(SPEX  ,4,K,L,M);
                            if(I == 2) T=get(SPEX  ,5,K,L,M);
                            for(J=0;J<=JMAX;J++){
                                TVD=get(SPVM ,1,get(ISPEX  ,K,5,L,M),MOLSP)/T;
                                ZVT=1.e00/(1.e00-exp(-TVD));
                                get (BFRAC , J+1,I)=exp(-J*get(SPVM ,1,get(ISPEX  ,K,5,L,M),MOLSP)/T)/ZVT;    //Boltzmann fraction
                                get (VRREX , I,K,L,M,J+1)=get (VRREX , I,K,L,M,J+1)*get (BFRAC , 1+J,I);
                                //              WRITE (*,*) 'Contribution',I,J,VRREX(I,K,L,M,J)
                                for(MM=0;MM<=J;MM++)
                                    get (TOT,1+ J,I)=get (TOT,1+ J,I)+get (VRREX , I,K,L,M,MM+1);
                            }
                        }
                        //
                        for(I=1;I<=2;I++){
                            for(J=0;J<=JMAX;J++){
                                get (SPREX , I,K,L,M,J+1 )=get (TOT,1+ J,I);
                                if(J == JMAX) get (SPREX , I,K,L,M,J+1)=1.e00;
                                //WRITE (9,*) 'Cumulative',I,J,SPREX(I,K,L,M,J)
                                file_9<<"Cumulative "<<I<<" "<<J<<" "<<get (SPREX , I,K,L,M,J+1);
                            }
                        }
                    }
                }
                NSLEV=0;
                //memget(SLER,0.e00,sizeof(*SLER));//SLER=0.e00;
                for(int i=0;i<MSP+1;i++)
                    SLER[i]=0.e00;
            }
        }
    }
    //
    //READ (*,*)  //optionally pause program to check cumulative distributions for exchange abd chain reactions
    return;
}


void MOLECULES_MOVE_1D()
{//
    //molecule moves appropriate to the time step
    //for homogeneous and one-dimensional flows
    //(homogeneous flows are calculated as one-dimensional)
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int N,L,M,K,NCI,J,II,JJ;
    double A,B,X,XI,XC,DX,DY,DZ,DTIM,S1,XM,R,TI,DTC,POB,UR,WFI,WFR,WFRI;
    //
    //N working integer
    //NCI initial cell time
    //DTIM time interval for the move
    //POB position of the outer boundary
    //TI initial time
    //DTC time interval to collision with surface
    //UR radial velocity component
    //WFI initial weighting factor
    //WFR weighting factor radius
    //WFRI initial weighting factor radius
    //

    if((ITYPE[2] == 4) && (ICN == 1)){
        //memget(ALOSS,0.e00,sizeof(*ALOSS));//ALOSS=0.e00;
        for(int i=0;i<MSP+1;i++)
            ALOSS[i]=0.e00;
        
        NMP=NM;
    }
    //
    N=1;
   
    while(N <= NM){
        //
        NCI=get(IPCELL  ,N);
        if((IMTS == 0) || (IMTS == 2)) DTIM=DTM;
        if(IMTS == 1) DTIM=2.e00*get (CCELL , 3,NCI);
        if(FTIME-PTIM[N] > 0.5*DTIM){
            WFI=1.e00;
            if(IWF == 1) WFI=1.e00+WFM*pow(get (PX , 1,N),IFX);
            II=0; //becomes 1 if a molecule is removed
            TI=PTIM[N];
            PTIM[N]=TI+DTIM;
            TOTMOV=TOTMOV+1;
            //
            XI=get (PX , 1,N);
            DX=DTIM*get(PV  ,1,N);
            X=XI+DX;
            //
            if(IFX > 0){
                DY=0.e00;
                DZ=DTIM*get(PV  ,3,N);
                if(IFX == 2) DY=DTIM*get(PV  ,2,N);
                R=sqrt(X*X+DY*DY+DZ*DZ);
            }
            //
            if(IFX == 0){
                for(J=1;J<=2;J++){    // 1 for minimum x boundary, 2 for maximum x boundary
                    if(II == 0){
                        if(((J == 1) && (X < XB[1])) || ((J == 2) && (X > (XB[2]+VELOB*PTIM[N])))){  //molecule crosses a boundary
                            if((ITYPE[J] == 0) || (ITYPE[J] == 3) || (ITYPE[J] == 4)){
                                if(XREM > XB[1]){
                                    L=get(IPSP ,N);
                                    ENTMASS=ENTMASS-get(SP ,5,L);
                                }
                                if((ITYPE[2] == 4) && (ICN == 1)){
                                    L=get(IPSP ,N);
                                    ALOSS[L]=ALOSS[L]+1.e00;
                                }
                                REMOVE_MOL(N);
                                N=N-1;
                                II=1;
                            }
                            //
                            if(ITYPE[J] == 1){
                                if((IVB == 0) || (J == 1)){
                                    X=2.e00*XB[J]-X;
                                    get(PV  ,1,N)=-get(PV  ,1,N);
                                }
                                else if((J == 2) && (IVB == 1)){
                                    DTC=(XB[2]+TI*VELOB-XI)/(get(PV  ,1,N)-VELOB);
                                    XC=XI+get(PV  ,1,N)*DTC;
                                    get(PV  ,1,N)=-get(PV  ,1,N)+2*VELOB;
                                    X=XC+get(PV  ,1,N)*(DTIM-DTC);
                                }
                            }
                            //
                            if(ITYPE[J] == 2)
                                REFLECT_1D(N,J,X);
                            // END IF
                        }
                    }
                }
            }
            else{         //cylindrical or spherical flow
                //check boundaries
                if((X < XB[1]) && (XB[1] > 0.e00)){
                    RBC(XI,DX,DY,DZ,XB[1],S1);
                    if(S1 < 1.e00){     //intersection with inner boundary
                        if(ITYPE[1] == 2){//solid surface
                            DX=S1*DX;
                            DY=S1*DY;
                            DZ=S1*DZ;
                            AIFX(XI,DX,DY,DZ,X,get(PV  ,1,N),get(PV  ,2,N),get(PV  ,3,N));
                            REFLECT_1D(N,1,X);
                        }
                        else{
                            REMOVE_MOL(N);
                            N=N-1;
                            II=1;
                        }
                    }
                }
                else if((IVB == 0) && (R > XB[2])){
                    RBC(XI,DX,DY,DZ,XB[2],S1);
                    if(S1 < 1.e00){     //intersection with outer boundary
                        if(ITYPE[2] == 2){ //solid surface
                            DX=S1*DX;
                            DY=S1*DY;
                            DZ=S1*DZ;
                            AIFX(XI,DX,DY,DZ,X,get(PV  ,1,N),get(PV  ,2,N),get(PV  ,3,N));
                            X=1.001e00*XB[2];
                            while(X > XB[2])
                                REFLECT_1D(N,2,X);
                            // END DO
                        }
                        else{
                            REMOVE_MOL(N);
                            N=N-1;
                            II=1;
                        }
                    }
                }
                else if((IVB == 1) && (R > (XB[2]+PTIM[N]*VELOB))){
                    if(IFX == 1) UR=sqrt(pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2));
                    if(IFX == 2) UR=sqrt(pow(get(PV  ,1,N),2)+pow(get(PV  ,2,N),2)+pow(get(PV  ,3,N),2));
                    DTC=(XB[2]+TI*VELOB-XI)/(UR-VELOB);
                    S1=DTC/DTIM;
                    DX=S1*DX;
                    DY=S1*DY;
                    DZ=S1*DZ;
                    AIFX(XI,DX,DY,DZ,X,get(PV  ,1,N),get(PV  ,2,N),get(PV  ,3,N));
                    get(PV  ,1,N)=-get(PV  ,1,N)+2.0*VELOB;
                    X=X+get(PV  ,1,N)*(DTIM-DTC);
                }
                else
                    AIFX(XI,DX,DY,DZ,X,get(PV  ,1,N),get(PV  ,2,N),get(PV  ,3,N));
                
                
                //DIAGNOSTIC
                if(II == 0){
                    if(X > XB[2]+PTIM[N]*VELOB){
                        //WRITE (*,*) N,FTIME,X,XB[2]+PTIM[N]*VELOB;
                        cout<<N<<" "<<FTIME<<" "<<X<<" "<<(XB[2]+PTIM[N]*VELOB)<<endl;
                    }
                }
                
                //Take action on weighting factors
                if((IWF == 1) && (II == 0)){
                    WFR=WFI/(1.e00+WFM*pow(X,IFX));
                    L=0;
                    WFRI=WFR;
                    if(WFR >= 1.e00){
                        while(WFR >= 1.e00){
                            L=L+1;
                            WFR=WFR-1.e00;
                        }
                    }
                    // CALL RANDOM_NUMBER(RANF)
                    RANF=((double)rand()/(double)RAND_MAX);
                    if(RANF <= WFR) L=L+1;
                    if(L == 0){
                        REMOVE_MOL(N);
                        N=N-1;
                        II=1;
                    }
                    L=L-1;
                    if(L > 0){
                        for(K=1;K<=L;K++){
                            if(NM >= MNM) EXTEND_MNM(1.1);
                            NM=NM+1;
                            get (PX , 1,NM)=X;
                            for(M=1;M<=3;M++)
                                get(PV  ,M,NM)=get(PV  ,M,N);
                            
                            if(MMRM > 0) PROT[NM]=PROT[N];
                            get(IPCELL  ,NM)=fabs(get(IPCELL  ,N));
                            get(IPSP ,NM)=get(IPSP ,N);
                            IPCP[NM]=IPCP[N];
                            if(MMVM > 0){
                                for(M=1;M<=MMVM;M++)
                                    get (IPVIB , M,NM)=get (IPVIB , M,N);
                                
                            }
                            PTIM[NM]=PTIM[N];    //+5.D00*DFLOAT(K)*DTM
                            //note the possibility of a variable time advance that may take the place of the duplication buffer in earlier programs
                            
                            if(get (PX , 1,NM) > XB[2]+PTIM[NM]*VELOB)
                                //WRITE (*,*) 'DUP',NM,FTIME,PX(1,NM),XB(2)+PTIM(NM)*VELOB
                                cout<<"DUP "<<NM<<" "<<FTIME<<" "<<get (PX , 1,NM)<<" "<<(XB[2]+PTIM[NM]*VELOB)<<endl;
                            
                        }
                    }
                }
            }
            //
            if(II == 0) {
                get (PX , 1,N)=X;
                
                if(get (PX , 1,N) > XB[1] && (get (PX , 1,N) < XB[2]))
                    continue;
                else{
                    //cout<< N<<" OUTSIDE FLOWFIELD AT "<<get (PX , 1,N]<<" VEL "<<get(PV  ,1,N]<<endl;
                    REMOVE_MOL(N);
                    N=N-1;
                    II=1;
                }
            }
            //
            if(II == 0){
                if(IVB == 0) FIND_CELL_1D(get (PX , 1,N),get(IPCELL  ,N),JJ);
                if(IVB == 1) FIND_CELL_MB_1D(get (PX , 1,N),get(IPCELL  ,N),JJ,PTIM[N]);
            }
            //
        }
        //
        N=N+1;
    }
    //
    return;
}


void MOLECULES_ENTER_1D()
{
    //molecules enter boundary at XB(1) and XB(2) and may be removed behind a wave
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //GEOM_1D geom;

    //
    int K,L,M,N,NENT,II,J,JJ,KK,NTRY;
    double A,B,AA,BB,U,VN,XI,X,DX,DY,DZ;
    //
    //NENT number to enter in the time step
    //
    ENTMASS=0.e00;
    //
    for(J=1;J<=2;J++){     //J is the end
        if((ITYPE[J] == 0) || (ITYPE[J] == 4)){
            KK=1;//the entry surface will normally use the reference gas (main stream) properties
            if((J == 2) && (ISECS == 1) && (XB[2] > 0.e00)) KK=2;    //KK is 1 for reference gas 2 for the secondary stream
            for(L=1;L<=MSP;L++){
                A=get (ENTR , 1,L,J)*DTM+get (ENTR , 2,L,J);
                if((ITYPE[2] == 4) && (ICN == 1)){
                    NENT=A;
                    if(J == 1) EME[L]=NENT;
                    if(J == 2) {
                        A=ALOSS[L]-EME[L]-AJM[L];
                        AJM[L]=0.e00;
                        if(A < 0.e00){
                            AJM[L]=-A;
                            A=0.e00;
                        }
                    }
                }
                NENT=A;
                get (ENTR , 2,L,J)=A-NENT;
                if((ITYPE[2] == 4) && (J == 2) && (ICN == 1)) get (ENTR , 2,L,J)=0.e00;
                if(NENT > 0){
                    for(M=1;M<=NENT;M++){
                        if(NM >= MNM){
                          
                            EXTEND_MNM(1.1);
                        }
                        NM=NM+1;
                        AA=max(0.e00,get (ENTR , 3,L,J)-3.e00);
                        BB=max(3.e00,get (ENTR , 3,L,J)+3.e00);
                        II=0;
                        while(II == 0){
                            RANF=((double)rand()/(double)RAND_MAX);
                            // CALL RANDOM_NUMBER(RANF)
                            B=AA+(BB-AA)*RANF;
                            U=B-get (ENTR , 3,L,J);
                            A=(2.e00*B/get (ENTR , 4,L,J))*exp(get (ENTR , 5,L,J)-U*U);
                            RANF=((double)rand()/(double)RAND_MAX);
                            // CALL RANDOM_NUMBER(RANF)
                            if(A > RANF) II=1;
                        }
                        get(PV  ,1,NM)=B*get (VMP , L,KK);
                        if(J == 2) get(PV  ,1,NM)=-get(PV  ,1,NM);
                        //
                        RVELC(get(PV  ,2,NM),get(PV  ,3,NM),get (VMP , L,KK));
                        get(PV  ,2,NM)=get(PV  ,2,NM)+VFY[J];
                        //
                        if(get(ISPR ,1,L) > 0) SROT(L,FTMP[KK],PROT[NM]);
                        //
                        if(MMVM > 0){
                            for(K=1;K<=get(ISPV  ,L);K++)
                                SVIB(L,FVTMP[KK],get (IPVIB , K,NM),K);
                        }
                        if(MELE > 1) SELE(L,FTMP[KK],PELE[NM]);
                        //
                        if(PELE[NM] > 0.e00)
                            continue;                     //DEBUG
                        //
                        get(IPSP ,NM)=L;
                        //advance the molecule into the flow
                        RANF=((double)rand()/(double)RAND_MAX);
                        // CALL RANDOM_NUMBER(RANF)
                        XI=XB[J];
                        DX=DTM*RANF*get(PV  ,1,NM);
                        if((IFX == 0) || (J == 2)) X=XI+DX;
                        if(J == 1){   //1-D move at outer boundary so molecule remains in flow
                            if(IFX > 0) DY=DTM*RANF*get(PV  ,2,NM);
                            DZ=0.e00;
                            if(IFX == 2) DZ=DTM*RANF*get(PV  ,3,NM);
                            if(IFX > 0) AIFX(XI,DX,DY,DZ,X,get(PV  ,1,NM),get(PV  ,2,NM),get(PV  ,3,NM));
                        }
                        get (PX , NCLASS,NM)=X;
                        PTIM[NM]=FTIME;
                        if(IVB == 0) FIND_CELL_1D(get (PX , NCLASS,NM),get(IPCELL  ,NM),JJ);
                        if(IVB == 1) FIND_CELL_MB_1D(get (PX , NCLASS,NM),get(IPCELL  ,NM),JJ,PTIM[NM]);
                        IPCP[NM]=0;
                        if(XREM > XB[1]) ENTMASS=ENTMASS+get(SP ,5,L);
                    }
                }
            }
            if((ITYPE[2] == 4) && (J==2) && (NM != NMP) && (ICN == 1))
                continue;
        }
    }
    //
    //stagnation streamline molecule removal
    if(XREM > XB[1]){
        ENTMASS=FREM*ENTMASS;
        NTRY=0;
        ENTMASS=ENTMASS+ENTREM;
        while((ENTMASS > 0.e00) && (NTRY < 10000)){
            NTRY=NTRY+1;
            if(NTRY == 10000){
                cout<<"Unable to find molecule for removal"<<endl;
                ENTMASS=0.e00;
                //memget(VNMAX,0.e00,sizeof(*VNMAX));//VNMAX=0.e00;
                for(int i=0;i<MSP+1;i++)
                    VNMAX[i]=0.e00;
            }
            RANF=((double)rand()/(double)RAND_MAX) ;
            // CALL RANDOM_NUMBER(RANF)
            N=NM*RANF+0.9999999e00;
            if(get (PX , NCLASS,N) > XREM){
                // CALL RANDOM_NUMBER(RANF)
                RANF=((double)rand()/(double)RAND_MAX) ;
                //IF (RANF < ((PX(N)-XREM)/(XB(2)-XREM))*2) THEN
                if(fabs(VFY[1]) < 1.e-3)
                    VN=sqrt(get(PV  ,2,N)*get(PV  ,2,N)+get(PV  ,3,N)*get(PV  ,3,N)) ;   //AXIALLY SYMMETRIC STREAMLINE
                else
                    VN=fabs(get(PV  ,3,N)) ;   //TWO-DIMENSIONAL STREAMLINE
                 
                L=get(IPSP ,N);
                if(VN > VNMAX[L]) VNMAX[L]=VN;
                // CALL RANDOM_NUMBER(RANF)
                RANF=((double)rand()/(double)RAND_MAX);
                if(RANF < VN/VNMAX[L]){
                    REMOVE_MOL(N);
                    ENTMASS=ENTMASS-get(SP ,5,L);
                    NTRY=0;
                }
                //END IF
            }
        }
        ENTREM=ENTMASS;
    }
}


void INDEX_MOLS()
{
    //index the molecules to the collision cells
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    // IMPLICIT NONE
    //
    int N,M,K;
    //
    //N,M,K working integer
    //
    for(N=0 ;N<=NCCELLS;N++)
        get (ICCELL , 2,N)=0;

    
    //
    if(NM != 0){
        for(N=1;N<=NM;N++){
            M=get(IPCELL  ,N);
            get (ICCELL , 2,M)=get (ICCELL , 2,M)+1;
        }
        //

        M=0;
        for(N=1;N<=NCCELLS;N++){
            get (ICCELL , 1,N)=M;
            M=M+get (ICCELL , 2,N);
            get (ICCELL , 2,N)=0;
        }
        //

        for(N=1;N<=NM;N++){
            M=get(IPCELL  ,N);
            get (ICCELL , 2,M)=get (ICCELL , 2,M)+1;
            K=get (ICCELL , 1,M)+get (ICCELL , 2,M);
            ICREF[K]=N;
        }
        //cin.get();
        //
    }
    return;
}


void RBC(double &XI, double &DX, double &DY,double &DZ, double &R,double &S)
{
    //calculates the trajectory fraction S from a point at radius XI with
    //note that the axis is in the y direction
    //--displacements DX, DY, and DZ to a possible intersection with a
    //--surface of radius R, IFX=1, 2 for cylindrical, spherical geometry
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    double A,B,C,DD,S1,S2;
    //
    DD=DX*DX+DZ*DZ;
    if(IFX == 2) DD=DD+DY*DY;
    B=XI*DX/DD;
    C=(XI*XI-R*R)/DD;
    A=B*B-C;
    if(A >= 0.e00){
        //find the least positive solution to the quadratic
        A=sqrt(A);
        S1=-B+A;
        S2=-B-A;
        if(S2 < 0.e00){
            if(S1 > 0.e00)
                S=S1;
            else
                S=2.e00;
        }
        else if(S1 < S2)
            S=S1;
        else
            S=S2;
    }
    else
        S=2.e00;
    //setting S to 2 indicates that there is no intersection
    return;
    //
}

void AIFX(double &XI,double &DX, double &DY, double &DZ, double &X, double &U, double &V, double &W)
{
    //
    //calculates the new radius and realigns the velocity components in
    //--cylindrical and spherical flows
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    //INTEGER ::
    double A,B,C,DR,VR,S;
    //
    if(IFX == 1){
        DR=DZ;
        VR=W;
    }
    else if(IFX == 2){
        DR=sqrt(DY*DY+DZ*DZ);
        VR=sqrt(V*V+W*W);
    }
    A=XI+DX;
    X=sqrt(A*A+DR*DR);
    S=DR/X;
    C=A/X;
    B=U;
    U=B*C+VR*S;
    W=-B*S+VR*C;
    if(IFX == 2){
        VR=W;
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        A=DPI*RANF;
        V=VR*sin(A);
        W=VR*cos(A);
    }
    //
    return;
    //
}


void REMOVE_MOL(int &N)
{
    //remove molecule N and replaces it by NM
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    // IMPLICIT NONE
    //
    int NC,M,K;
    
    //N the molecule number
    //M,K working integer
    //
    if(N != NM){
        for(M=1;M<=NCLASS;M++)
            get (PX , M,N)=get (PX , M,NM);
        for(M=1;M<=3;M++)
            get(PV  ,M,N)=get(PV  ,M,NM);
        
        if(MMRM > 0) PROT[N]=PROT[NM];
        get(IPCELL  ,N)=fabs(get(IPCELL  ,NM));
        get(IPSP ,N)=get(IPSP ,NM);
        IPCP[N]=IPCP[NM];
        if(MMVM > 0){
            for(M=1;M<=MMVM;M++)
                get (IPVIB , M,N)=get (IPVIB , M,NM);
        }
        if(MELE > 1) PELE[N]=PELE[NM];
        PTIM[N]=PTIM[NM];
    }
    NM=NM-1;
    //
    return;
    //
}

void DISSOCIATION()
{
    //dissociate diatomic molecules that have been marked for dissociation by -ve level or -99999 for ground state
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,KK,L,N,M,LS,MS,KV,IDISS;
    double A,B,C,EA,VRR,VR,RMM,RML;
    double VRC[4],VCM[4],VRCP[4];
    //
    N=0;
    while(N < NM){
        N=N+1;
        IDISS=0;
        L=get(IPSP ,N);
        if(get(ISPV  ,L) > 0){
            for(K=1;K<=get(ISPV  ,L);K++){
                M=get (IPVIB , K,N);
                if(M < 0){
                    //dissociation
                    TDISS[L]=TDISS[L]+1.e00;
                    IDISS=1;
                }
            }
            if(IDISS == 1){
                EA=PROT[N];    //EA is energy available for relative translational motion of atoms
                if(MELE > 1) EA=EA+PELE[N];
                if(NM >= MNM) EXTEND_MNM(1.1);
                NM=NM+1;
                //set center of mass velocity as that of molecule
                VCM[1]=get(PV  ,1,N);
                VCM[2]=get(PV  ,2,N);
                VCM[3]=get(PV  ,3,N);
                get (PX , NCLASS,NM)=get (PX , NCLASS,N);
                get(IPCELL  ,NM)=get(IPCELL  ,N);
                LS=get(IPSP ,N);
                get (TREACL , 1,LS)=get (TREACL , 1,LS)-1;
                get(IPSP ,NM)=get( ISPVM ,1,1,L);
                MS=get(IPSP ,NM);
                get(IPSP ,N)=get( ISPVM ,2,1,L);
                LS=get(IPSP ,N);
                get (TREACG , 1,LS)=get (TREACG , 1,LS)+1;
                get (TREACG , 1,MS)=get (TREACG , 1,MS)+1;
                PTIM[NM]=PTIM[N];
                VRR=2.e00*EA/get (SPM , 1,LS,MS);
                VR=sqrt(VRR);
                RML=get (SPM , 1,LS,MS)/get(SP ,5,MS);
                RMM=get (SPM , 1,LS,MS)/get(SP ,5,LS);
                // CALL RANDOM_NUMBER(RANF)
                RANF=((double)rand()/(double)RAND_MAX);
                B=2.e00*RANF-1.e00;
                A=sqrt(1.e00-B*B);
                VRCP[1]=B*VR;
                // CALL RANDOM_NUMBER(RANF)
                RANF=((double)rand()/(double)RAND_MAX);
                C=2.e00*PI*RANF;
                VRCP[2]=A*cos(C)*VR;
                VRCP[3]=A*sin(C)*VR;
                for(KK=1;KK<=3;KK++){
                    get(PV  ,KK,N)=VCM[KK]+RMM*VRCP[KK];
                    get(PV  ,KK,NM)=VCM[KK]-RML*VRCP[KK];
                }
                
                if((fabs(get(PV  ,1,N)) > 100000.e00) || (fabs(get(PV  ,1,NM)) > 100000.e00)) {
                    cout<< "EXCESSIVE SPEED, DISS "<< N<< " "<<get(PV  ,1,N)<<" "<<NM<<" "<<get(PV  ,1,NM)<<endl;
                   
                }
                
                
                
                //set any internal modes to the ground state
                if(get(ISPV  ,LS) > 0){
                    for(KV=1;KV<=get(ISPV  ,LS);KV++)
                        get (IPVIB , KV,N)=0;
                }
                if(get(ISPR ,1,LS) > 0) PROT[N]=0.e00;
                if(MELE > 1) PELE[N]=0.e00;
                if(get(ISPV  ,MS) > 0){
                    for(KV=1;KV<=get(ISPV  ,MS);KV++)
                        get (IPVIB , KV,NM)=0;
                }
                if(get(ISPR ,1,MS) > 0) PROT[NM]=0.0;
                if(MELE > 1) PELE[NM]=0.e00;
            }
        }
    }
    return;
}

void EXTEND_MNM(double FAC)
{  //
    //the maximum number of molecules is increased by a specified factor
    //the existing molecules are copied TO disk storage
    //MOLECS molecs;
    //CALC calc;
    //GAS gas;
    //
    // IMPLICIT NONE
    //
    int M,N,MNMN;
    fstream file_7;
    // REAL :: FAC
    //
    //M,N working integers
    //MNMN extended value of MNM
    //FAC the factor for the extension
    MNMN=FAC*MNM;
    cout<< "Maximum number of molecules is to be extended from "<<MNM<<" to "<<MNMN<<endl;
    cout<< "( if the additional memory is available //// )"<<endl;
    
    file_7.open("EXTMOLS.TXT", ios::binary | ios::out);
    if(file_7.is_open()){
        cout<<"EXTMOLS.TXT is opened"<<endl;
    }
    else{
        cout<<"EXTMOLS.TXT not opened"<<endl;
    }
    cout<<"Start write to disk storage"<<endl;
    //OPEN (7,FILE='EXTMOLS.SCR',FORM='BINARY')
    //WRITE (*,*) 'Start write to disk storage'
    
    for(N=1;N<=MNM;N++){
        if(MMVM > 0){
            file_7<<get (PX , NCLASS,N)<<endl<<PTIM[N]<<endl<<PROT[N]<<endl;
            for(M=1;M<=3;M++)
                file_7<<get(PV  ,M,N)<<endl;
            file_7<<get(IPSP ,N)<<endl<<get(IPCELL  ,N)<<endl<<ICREF[N]<<endl<<IPCP[N]<<endl;
            for(M=1;M<=MMVM;M++)
                file_7<<get (IPVIB , M,N)<<endl;
            file_7<<PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),(get(IPVIB  ,M,N),M=1,MMVM),PELE(N)
        }
        else{
            if(MMRM > 0){
                file_7<<get (PX , NCLASS,N)<<endl<<PTIM[N]<<endl<<PROT[N]<<endl;
                for(M=1;M<=3;M++)
                    file_7<<get(PV  ,M,N)<<endl;
                file_7<<get(IPSP ,N)<<endl<<get(IPCELL  ,N)<<endl<<ICREF[N]<<endl<<IPCP[N]<<endl<<PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            else{
                file_7<<get (PX , NCLASS,N)<<endl<<PTIM[N]<<endl;
                for(M=1;M<=3;M++)
                    file_7<<get(PV  ,M,N)<<endl;
                file_7<<get(IPSP ,N)<<endl<<get(IPCELL  ,N)<<endl<<ICREF[N]<<endl<<IPCP[N]<<endl<<PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            
        }
    }
    cout<<"Disk write completed"<<endl;
    // WRITE (*,*) 'Disk write completed'
    // CLOSE (7)
    file_7.close();
    if(MMVM > 0){
        
        free(PX); //delete [) PX;

        free(PTIM); //delete [) PTIM;

        free(PROT);

        
        free(PV); //delete [) PV;

        free(IPSP);
        free(IPCELL);
        free(ICREF);
        free(IPCP);
        free(PELE);
        
        free(IPVIB); //delete IPVIB;
        // for(int i=0;i<NCLASS+1;i++){
        //     delete [) get (PX , i];
        // }
        // delete [] PX;
        // delete [] PTIM;
        // delete [] PROT;
        // for(int i=0;i<4;i++){
        //     delete [] get(PV  ,i];
        // }
        // delete [] PV;
        // delete [] IPSP;
        // delete [] IPCELL;
        // delete [] ICREF;
        // delete [] IPCP;
        // delete [] PELE;
        // for(int i=0;i<MMVM;i++){
        //     delete [] get (IPVIB , i];
        // }
        // delete IPVIB;
        //DEALLOCATE (PX,PTIM,PROT,PV,IPSP,IPCELL,ICREF,IPCP,IPVIB,PELE,STAT=ERROR)
    }
    else{
        if(MMRM > 0){
            
            free(PX); //delete [) PX;

            free(PTIM); //delete [) PTIM;

            free(PROT);

           
            free(PV); //delete [) PV;

            free(IPSP);
            free(IPCELL);
            free(ICREF);
            free(IPCP);
            free(PELE);
            // delete [) IPSP;
            // delete [) IPCELL;
            // delete [) ICREF;
            // delete [) IPCP;
            // delete [) PELE;//DEALLOCATE (PX,PTIM,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
            // for(int i=0;i<NCLASS+1;i++){
            //     delete [) get (PX , i);
            // }
            // delete [) PX;
            // delete [) PTIM;
            // delete [) PROT;
            // for(int i=0;i<4;i++){
            //     delete [) get(PV  ,i);
            // }
            // delete [) PV;
            // delete [) IPSP;
            // delete [) IPCELL;
            // delete [) ICREF;
            // delete [) IPCP;
            // delete [) PELE;
            //DEALLOCATE (PX,PTIM,PROT,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
        }
        else{
            
            free(PX); //delete [) PX;

            free(PTIM); //delete [) PTIM;

          
            free(PV); //delete [) PV;

            free(IPSP);
            free(IPCELL);
            free(ICREF);
            free(IPCP);
            free(PELE);
            // delete [) IPSP;
            // delete [) IPCELL;
            // delete [) ICREF;
            // delete [) IPCP;
            // delete [) PELE;//DEALLOCATE (PX,PTIM,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT DEALLOCATE MOLECULES',ERROR
    // !  STOP
    // END IF
    // !
    
    if(MMVM > 0){
        d_allocate(NCLASS,MNMN,PX);
        d_allocate(MNMN,PTIM);
        d_allocate(MNMN,PROT);
        d_allocate(3,MNMN,PV);
        i_allocate(MNMN,IPSP);
        i_allocate(MNMN,IPCELL);
        i_allocate(MNMN,ICREF);
        i_allocate(MNMN,IPCP);
        i_allocate(MMVM,MNMN,IPVIB);
        d_allocate(MNMN,PELE);
        // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PROT(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),get(IPVIB  ,MMVM,MNMN),PELE(MNMN),STAT=ERROR)
    }
    else{
        if(MMRM > 0){
            d_allocate(NCLASS,MNMN,PX);
            d_allocate(MNMN,PTIM);
            d_allocate(MNMN,PROT);
            d_allocate(3,MNMN,PV);
            i_allocate(MNMN,IPSP);
            i_allocate(MNMN,IPCELL);
            i_allocate(MNMN,ICREF);
            i_allocate(MNMN,IPCP);
            d_allocate(MNMN,PELE);
            // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PROT(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),PELE(MNMN),STAT=ERROR)
        }
        else{
            d_allocate(NCLASS,MNMN,PX);
            d_allocate(MNMN,PTIM);
            d_allocate(3,MNMN,PV);
            i_allocate(MNMN,IPSP);
            i_allocate(MNMN,IPCELL);
            i_allocate(MNMN,ICREF);
            i_allocate(MNMN,IPCP);
            d_allocate(MNMN,PELE);
            // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),PELE(MNMN),STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SPACE FOR EXTEND_MNM',ERROR
    // !  STOP
    // END IF
    // !
    //memget(PX,0.0,sizeof(*PX)); memget(PTIM,0.0,sizeof(*PTIM)); memget(PV,0.0,sizeof(*PV)); memget(IPSP,0,sizeof(*IPSP)); memget(IPCELL,0,sizeof(*IPCELL)); memget(ICREF,0,sizeof(*ICREF)); memget(IPCP,0,sizeof(*IPCP)); memget(PELE,0,sizeof(*PELE));
    
    for(int i=0;i<NCLASS+1;i++){
        for(int j=0;j<MNMN+1;j++)
            get (PX , i,j)=0.0;
    }
    
    for(int i=0;i<4;i++){
        for(int j=0;j<MNMN+1;j++)
            get(PV  ,i,j)=0.0;
    }
    for(int i=0;i<MNMN+1;i++){
        PTIM[i]=0.0;
        get(IPSP ,i)=0;
        get(IPCELL  ,i)=0;
        ICREF[i]=0;
        IPCP[i]=0;
        PELE[i]=0;
    }
        
    
    if(MMRM > 0) {
        for(int i=0;i<MNMN+1;i++)
            PROT[i]=0.0;
        //memget(PROT,0.0,sizeof(*PROT));
    }
    if(MMVM > 0) {
        for(int i=0;i<MMVM+1;i++){
            for(int j=0;j<MNMN+1;j++)
                get (IPVIB , i,j)=0;
        }
        //memget(IPVIB,0,sizeof(*IPVIB));
    }
    //restore the original molecules
    // OPEN (7,FILE='EXTMOLS.SCR',FORM='BINARY')
    // WRITE (*,*) 'Start read back from disk storage'
    file_7.open("EXTMOLS.TXT", ios::binary | ios::in);
    if(file_7.is_open()){
        cout<<"EXTMOLS.TXT is opened"<<endl;
    }
    else{
        cout<<"EXTMOLS.TXT not opened"<<endl;
    }
    for(N=1;N<=MNM;N++){
        if(MMVM > 0){
            file_7>>get (PX , NCLASS,N)>>PTIM[N]>>PROT[N];
            for(M=1;M<=3;M++)
                file_7>>get(PV  ,M,N);
            file_7>>get(IPSP ,N)>>get(IPCELL  ,N)>>ICREF[N]>>IPCP[N];
            for(M=1;M<=MMVM;M++)
                file_7>>get (IPVIB , M,N);
            file_7>>PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),(get(IPVIB  ,M,N),M=1,MMVM),PELE(N)
        }
        else{
            if(MMRM > 0){
                file_7>>get (PX , NCLASS,N)>>PTIM[N]>>PROT[N];
                for(M=1;M<=3;M++)
                    file_7>>get(PV  ,M,N);
                file_7>>get(IPSP ,N)>>get(IPCELL  ,N)>>ICREF[N]>>IPCP[N]>>PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            else{
                file_7>>get (PX , NCLASS,N)>>PTIM[N];
                for(M=1;M<=3;M++)
                    file_7>>get(PV  ,M,N);
                file_7>>get(IPSP ,N)>>get(IPCELL  ,N)>>ICREF[N]>>IPCP[N]>>PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
        }
    }
    cout<<"Disk read completed"<<endl;
    // WRITE (*,*) 'Disk read completed'
    // CLOSE (7,STATUS='DELETE')
    file_7.close();
    //
    MNM=MNMN;
    //
    return;
}

void SAMPLE_FLOW()
{
    //sample the flow properties
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    file_3 << "SAMPLE_FLOW IS Running \n" ;
    file_9 << "sample flow is running\n" ;
    //GAS gas;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int NC,NCC,LS,N,M,K,L,I,KV;
    double A,TE,TT,WF;
    //
    //NC the sampling cell number
    //NCC the collision cell number
    //LS the species code
    //N,M,K working integers
    //TE total translational energy
    //
    NSAMP=NSAMP+1;
    cout<<"Sample \t"<<NSAMP<<endl<<endl;
    //WRITE (9,*) NM,'Mols. at sample',NSAMP
    file_9<<NM<<"\t  Mols. at sample \t "<<NSAMP<<endl;
    //
    cout << "get(PV  ,1,4856) = "<< get(PV  ,1,4856) << endl ;
    for(N=1;N<=NM;N++){
        
        //if(N == 4856)   continue ;  // in cpp you will see that this number is equal to infinity.
        NCC=get(IPCELL  ,N);
        NC=get (ICCELL , 3,NCC);
        WF=1.e00;
        if(IWF == 1) WF=1.e00+WFM*pow(get (PX , 1,N),IFX);
        if((NC > 0) && (NC <= NCELLS)){
            if(MSP > 1)
                LS=fabs(get(IPSP ,N));
            else
                LS=1;
            
            get (CS,1+ 0,NC,LS)=get (CS,1+ 0,NC,LS)+1.e00;
            get (CS,1+ 1,NC,LS)=get (CS,1+ 1,NC,LS)+WF;
            if(N == 1)
            
            for(M=1;M<=3;M++){
                get (CS,1+ M+1,NC,LS) = get (CS,1+ M+1,NC,LS) + WF*get(PV  ,M,N) ;
                get (CS,1+ M+4,NC,LS)=get (CS,1+ M+4,NC,LS)+ WF*pow(get(PV  ,M,N),2) ;
            }
            if(MMRM > 0) get (CS,1+ 8,NC,LS)=get (CS,1+ 8,NC,LS)+WF*PROT[N];
            if(MELE > 1) get (CS,1+ 9,NC,LS)=get (CS,1+ 9,NC,LS)+WF*PELE[N];
            if(MMVM > 0){
                if(get(ISPV  ,LS) > 0){
                    for(K=1;K<=get(ISPV  ,LS);K++)
                        get (CS,1+ K+9,NC,LS)=get (CS,1+ K+9,NC,LS)+WF*(double)(get (IPVIB , K,N));
                }
            }
        }
        else{
            cout<<"Illegal sampling cell  "<<NC<<"  "<<NCC<<"  for MOL  "<<N<<"  at  "<<get (PX , 1,N)<<endl;
            return;
        }


        
    }

    for(int II = 0 ; II <11 ; II++)
        {
            cout << "get (CS,1+ "<<II<<",1,1] =" <<get (CS, 1+ II , 1, 1 ) <<endl ;
        }
    //
    
    if(FTIME > 0.5e00*DTM) TSAMP=TSAMP+DTSAMP;
    //
    return;
}

string itos(int c)
{
    stringstream ss ;
    ss << c  ;
    string b= ss.str() ;
    return b ;
}

void OUTPUT_RESULTS()
{
    //--calculate the surface and flowfield properties
    //--generate TECPLOT files for displaying these properties
    //--calculate collisiion rates and flow transit times and reset time intervals
    //--add molecules to any flow plane molecule output files
    //CALC calc;
    //MOLECS molecs;
    //GAS gas;
    //OUTPUT output;
    //GEOM_1D geom;
    cout << 111 << endl ;
    file_9 << "output results is running \n" ;
    fstream file_3;
    fstream file_10;
    fstream file_7;
    
    int IJ,J,JJ,K,L,LL,M,N,NN,NMCR,CTIME,II;
    long long NNN;
    double AS,AT,C1,C2,C3,C4,C5,C6,C7,C8,C9;
    double A,B,C,SDTM,SMCR,DOF,AVW,UU,VDOFM,TVIBM,VEL,DTMI,TT;
    //dout
    double SUM[14];
    //d_allocate(14 , SUM) ;
    double *SUMS;
    d_allocate( 10 , 2 , SUMS ) ;
    double *TVIB,*VDOF,*PPA,*TEL,*ELDOF,*SDOF,*CDTM;
    double *TV,*THCOL;
    double *DF;
    int *NMS;
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:) :: TVIB,VDOF,PPA,TEL,ELDOF,SDOF,CDTM
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:) :: TV,THCOL
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:,:) :: DF
    //    INTEGER, ALLOCATABLE, DIMENSION(:) :: NMS
    //INTEGER, ALLOCATABLE, DIMENSION(:,:) ::
    string F,E;
    //--CTIME  computer time (microseconds)
    //--SUMS(N,L) sum over species of CSS(N,J,L,M) for surface properties
    //
    //--For flowfield properties,where <> indicates sampled sum
    //--SUM(0) the molecular number sum over all species
    //--SUM(1) the weighted number sum over all species
    //--SUM(2) the weighted sum of molecular masses
    //--SUM(3),(4),(5) the weighted sum over species of m*<u>,<v>,<w>
    //--SUM(6) the weighted sum over species of m*(<u*2>+<v*2>+<w*2>)
    //--SUM(7) the weighted sum over species of <u*2>+<v*2>+<w*2>
    //--SUM(8) the weighted sum of rotational energy
    //--SUM(9) the weighted sum of rotational degrees of freedom
    //--SUM(10) the weighted sum over species of m*<u*2>
    //--SUM(11) the weighted sum over species of m*<v*2>
    //--SUM(12) sum over species of m*<w*2>
    //--SUM(13) the weighted sum of electronic energy
    //--UU velocity squared
    //--DOF degrees of freedom
    //--AVW the average value of the viscosity-temperature exponent
    //--DVEL velocity difference
    //--TVEL thermal speed
    //--SMCR sum of mcs/mfp over cells
    //--NMCR number in the sum
    //--VDOFM effective vibrational degrees of freedom of mixture
    //--TVIB(L)
    //--VDOF(L)
    //--TV(K,L) the temperature of vibrational mode K of species L
    //--PPA particles per atom
    //--NMS number per species
    //--SDOF(L) total degrees of freedom for species L
    //
    //
    //--calculate the flowfield properties in the cells
    //dout
    
    d_allocate(MMVM , MSP, TV ) ;
    
    d_allocate(MSP, TVIB) ;
    
    
    d_allocate(NCELLS , MMVM , MSP  , DF) ;
    
    //VDOF= new double[MSP];
    d_allocate(MSP, VDOF );
    //TEL = new double[MSP];
    d_allocate(MSP, TEL) ;
    //ELDOF = new double[MSP];
    d_allocate(MSP, ELDOF) ;
    //PPA = new double[MSP];
    d_allocate(MSP, PPA) ;
    //NMS = new int[MSP];
    i_allocate(MSP, NMS) ;
    
    d_allocate(MSP, MSP, THCOL) ;
    
    d_allocate(MSP, SDOF) ;
    
    d_allocate(NCELLS , CDTM) ;
    
    
    //    ALLOCATE (TV(MMVM,MSP),TVIB(MSP),DF(NCELLS,MMVM,MSP),VDOF(MSP),TEL(MSP),ELDOF(MSP),PPA(MSP),NMS(MSP),THCOL(MSP,MSP)    &
    //              ,SDOF(MSP),CDTM(NCELLS),STAT=ERROR)
    //    if(ERROR!=0)
    //    {
    //        cout<<"ROGRAM COULD NOT ALLOCATE OUTPUT VARIABLES"<<ERROR<<endl;
    //    }
    if(FTIME>0.5e00*DTM)
    {
        NOUT+=1;
        if(NOUT>9999)
            NOUT=NOUT-9999;
        cout << "ISF = "  << ISF << endl ;  // dsuedit
        cout<<"Generating files for output interval"<<NOUT<<endl;
        if(ISF==0)
        {
            //dout
            //OPEN (3,FILE='DS1OUT.DAT')
            file_3.open("DS1OUT.DAT" , ios::out);
            if(file_3.is_open()){
                cout<<"DS1OUT.DAT is opened"<<endl;
            }
            else{
                cout<<"DS1OUT.DAT not opened"<<endl;
            }
            //F='DS';//E//'.OUT'
        }
        else
        {
            //--the files are DS1n.DAT, where n is a four digit integer equal to NOUT
            //dout
            //500 FORMAT(I5)
            //ENCODE(5,500,E) 10000+NOUT

            cout << "ISF = "<< ISF << endl ;

            int a=NOUT+10000;
            E= itos(a) ;
            F="DS" + E + "OUT.DAT";

    // copying the contents of the  
    // string to char array 
            //strcpy(char_array, F.c_str()); 
            //dout
            file_3.open(F.c_str() , ios::out);
            if(file_3.is_open()){
                cout<<F<<" is opened"<<endl;
            }
            else{
                cout<<F<<" not opened"<<endl;
            }
            //OPEN (3,FILE=F)
        }
    }
    //dout
    //memget(VAR,0.e00,sizeof(*VAR));
    for(int i=0;i<24;i++){
        for(int j=0;j<NCELLS+1;j++)
            get (VAR , i,j)=0.e00;
    }
    if(IFX==0)
        A=(double)FNUM/(FTIME-TISAMP);
    for(JJ=1;JJ<=2;JJ++)
    {
        if(IFX==1)
            A=FNUM/(2.e00*PI*XB[JJ]*(FTIME-TISAMP));
        if(IFX==2)
            A=FNUM/(4.e00*PI*XB[JJ]*XB[JJ]*(FTIME-TISAMP));
        //--JJ=1 for surface at XB(1), JJ=2 for surface at XB(2)
        if(ITYPE[JJ]==2)
        {
            //dout
            //memget(SUMS,0.e00,sizeof(SUMS));
            for(int i=0;i<10;i++){
                for(int j=0;j<3;j++)
                    get (SUMS,1+ i,j)=0.e00;
            }
            for( L=1;L<=MSP;L++)
            {
                for(J=0;J<=8;J++)
                {
                    for(IJ=1;IJ<=2;IJ++)
                    {
                        get (SUMS,1+ J,IJ)=get (SUMS,1+ J,IJ)+get (CSS , 1+ J,JJ,L,IJ);
                    }
                }
            }
            get (VARS , 1,JJ)=get (SUMS,1+ 0,1);
            get (VARS , 2,JJ)=get (SUMS,1+ 1,1);
            get (VARS , 3,JJ)=get (SUMS,1+ 1,2);
            get (VARS , 4,JJ)=get (SUMS,1+ 1,1)*A;
            get (VARS , 5,JJ)=get (SUMS,1+ 1,2)*A;
            get (VARS , 6,JJ)=get (SUMS,1+ 2,1)*A;
            get (VARS , 7,JJ)=get (SUMS,1+ 2,2)*A;
            get (VARS , 8,JJ)=get (SUMS,1+ 3,1)*A;
            get (VARS , 9,JJ)=get (SUMS,1+ 3,2)*A;
            get (VARS , 10,JJ)=get (SUMS,1+ 4,1)*A;
            get (VARS , 11,JJ)=get (SUMS,1+ 4,2)*A;
            get (VARS , 12,JJ)=get (SUMS,1+ 5,1)*A;
            get (VARS , 13,JJ)=get (SUMS,1+ 5,2)*A;
            get (VARS , 14,JJ)=get (SUMS,1+ 6,1)*A;
            get (VARS , 15,JJ)=get (SUMS,1+ 6,2)*A;
            get (VARS , 16,JJ)=get (SUMS,1+ 7,1)*A;
            get (VARS , 17,JJ)=get (SUMS,1+ 7,2)*A;
            get (VARS , 34,JJ)=get (SUMS,1+ 8,1)*A;
            get (VARS , 35,JJ)=get (SUMS,1+ 8,2)*A;
            //   VARS(17,JJ)=SUMS(9,1)*A        //--SURFACE REACTIONS NOT YET IMPLEMENTED
            //   VARS(18,JJ)=SUMS(9,2)*A
            if(get (CSSS , 1,JJ)>1.e-6)
            {
                get (VARS , 20,JJ)=get (CSSS , 3,JJ)/get (CSSS , 2,JJ); ////--n.b. must be modified to include second component in 3D
                get (VARS , 21,JJ)=(get (CSSS , 4,JJ)-get (CSSS , 2,JJ)*get (VARS , 20,JJ)*get (VARS , 20,JJ))/(get (CSSS , 1,JJ)*3.e00*BOLTZ)-TSURF[JJ];
                get (VARS , 20,JJ)=get (VARS , 20,JJ)-VSURF[JJ];
                if(get (CSSS , 6,JJ)>0e00)
                {
                    get (VARS , 22,JJ)=(2.e000/BOLTZ)*(get (CSSS , 5,JJ)/get (CSSS , 6,JJ))-TSURF[JJ];
                }
                else
                {
                    get (VARS , 22,JJ)=0.e00;
                }
            }
            else
            {
                get (VARS ,1+ 19,JJ)=0.e00;
                get (VARS ,1+ 20,JJ)=0.e00;
                get (VARS ,1+ 21,JJ)=0.e00;
            }
            get (VARS ,1+ 22,JJ)=(get (SUMS,1+ 2,1)+get (SUMS,1+ 2,2))*A;
            get (VARS ,1+ 23,JJ)=(get (SUMS,1+ 3,1)+get (SUMS,1+ 3,2))*A;
            get (VARS ,1+ 24,JJ)=(get (SUMS,1+ 4,1)+get (SUMS,1+ 4,2))*A;
            get (VARS ,1+ 25,JJ)=(get (SUMS,1+ 5,1)+get (SUMS,1+ 5,2))*A;
            get (VARS ,1+ 26,JJ)=(get (SUMS,1+ 6,1)+get (SUMS,1+ 6,2))*A;
            get (VARS ,1+ 27,JJ)=(get (SUMS,1+ 7,1)+get (SUMS,1+ 7,2))*A;
            get (VARS ,1+ 28,JJ)=(get (SUMS,1+ 9,1)+get (SUMS,1+ 9,2))*A;
            get (VARS ,1+ 29,JJ)=get (VARS ,1+ 11,JJ)+get (VARS ,1+ 13,JJ)+get (VARS ,1+ 15,JJ)+get (VARS ,1+ 33,JJ);
            get (VARS ,1+ 30,JJ)=get (VARS ,1+ 12,JJ)+get (VARS ,1+ 14,JJ)+get (VARS ,1+ 16,JJ)+get (VARS ,1+ 34,JJ);
            get (VARS ,1+ 31,JJ)=get (VARS ,1+ 29,JJ)+get (VARS ,1+ 30,JJ);
            get (VARS ,1+ 35,JJ)=get (VARS ,1+ 33,JJ)+get (VARS ,1+ 34,JJ);
            for(L=1;MSP;L++)
            {
                if(get (SUMS,1+ 1,1)>0)
                {
                    get (VARS ,1+ 35+L,JJ)=100*get (CSS , 1+ 1,JJ,L,1)/get (SUMS,1+ 1,1);
                }
                else
                {
                    get (VARS ,1+ 35+L,JJ)=0.0;
                }
            }
        }
    }
    //VARSP=0;
    for(int i=0;i<13;i++){
        for(int j=0;j<NCELLS+1;j++){
            for(int k=0;k<MSP+1;k++)
                get (VARSP , i+1,j,k)=0;
        }
    }
    SMCR=0;
    NMCR=0;
    for(N=1;N<=NCELLS;N++)
    {
        if(N==120)
        {
            continue;
        }
        A=FNUM/(get (CELL , 4,N)*NSAMP);
        if(IVB==1)
            A=A*pow((XB[2]-XB[1])/(XB[2]+VELOB*0.5e00*(FTIME+TISAMP)-XB[1]) , IFX+1);
        //--check the above for non-zero XB(1)
        //dout
        //memget(SUM,0,sizeof(SUM));
        for(int i=0;i<14;i++)
            SUM[i]=0;


        
        NMCR+=1;
        //dsuedit
        
        
        //dsuedit
        for(L=1;L<=MSP;L++)
        {
           // dsuedit

            SUM[0]=SUM[0]+get (CS,1+ 0,N,L);
            SUM[1]=SUM[1]+get (CS,1+ 1,N,L);
            SUM[2]=SUM[2]+get (SP , 5,L)*get (CS,1+ 1,N,L);

            
            for(K=1;K<=3;K++)
            {
                SUM[K+2] = SUM[K+2]+get (SP , 5,L)*get (CS,1+ K+1,N,L);
                if(get (CS,1+ 1,N,L)>0.1e00)
                {
                    get (VARSP , K+2,N,L)=get (CS,1+ K+4,N,L)/get (CS,1+ 1,N,L);
                    //--VARSP(2,3,4 are temporarily the mean of the squares of the velocities
                    get (VARSP , K+8+1,N,L)=get (CS,1+ K+1,N,L)/get (CS,1+ 1,N,L);
                }
            }
            SUM[6]=SUM[6]+get (SP , 5,L)*(get (CS,1+ 5,N,L)+get (CS,1+ 6,N,L)+get (CS,1+ 7,N,L));
            SUM[10]=SUM[10]+get (SP , 5,L)*get (CS,1+ 5,N,L);
            SUM[12]=SUM[11]+get (SP , 5,L)*get (CS,1+ 6,N,L);
            SUM[12]=SUM[12]+get (SP , 5,L)*get (CS,1+ 7,N,L);
            SUM[13]=SUM[13]+get (CS,1+ 9,N,L);
            if(get (CS,1+ 1,N,L)>0.5e00)
                SUM[7]=SUM[7]+get (CS,1+ 5,N,L)+get (CS,1+ 6,N,L)+get (CS,1+ 7,N,L);
            if(get(ISPR ,1,L)>0)
            {
                SUM[8]=SUM[8]+get (CS,1+ 8,N,L);
                SUM[9]=SUM[9]+get (CS,1+ 1,N,L)*get(ISPR ,1,L);
            }
        }
        AVW=0;
        for(L=1;L<=MSP;L++)
        {
            get (VARSP , 0+1,N,L)=get (CS,1+ 1,N,L) ;
            get (VARSP , 1+1,N,L)=0.e00;
            get (VARSP , 6+1,N,L)=0.0;
            get (VARSP , 7+1,N,L)=0.0;
            get (VARSP , 8+1,N,L)=0.0;
            if(SUM[1]>0.1)
            {
                get (VARSP , 1+1,N,L)=get (CS,1+ 1,N,L)/SUM[1];
                AVW=AVW+get (SP , 3,L)*get (CS,1+ 1,N,L)/SUM[1];
                if(get(ISPR ,1,L)>0 && get (CS,1+ 1,N,L)>0.5)
                    get (VARSP , 6+1,N,L)=(2.e00/BOLTZ)*get (CS,1+ 8,N,L)/((double)(get(ISPR ,1,L))*get (CS,1+ 1,N,L));
            }
            get (VARSP , 5+1,N,L)=0;
            for(K=1;K<=3;K++)
            {
                get (VARSP , K+1+1,N,L)=(get (SP , 5,L)/BOLTZ)*(get (VARSP , K+1+1,N,L)-pow(get (VARSP , K+8+1,N,L),2));
                get (VARSP , 5+1,N,L)=get (VARSP , 5+1,N,L)+get (VARSP , K+1+1,N,L);
            }
            get (VARSP , 5+1,N,L)=get (VARSP , 5+1,N,L)/3.e00;
            get (VARSP , 8+1,N,L)=(3.e00*get (VARSP , 5+1,N,L)+(double)get(ISPR ,1,L)*get (VARSP , 6+1,N,L))/(3.e00+(double)(get(ISPR ,1,L)));
        }
        if(IVB==0)
            get (VAR , 1,N)=get (CELL , 1,N);
        if(IVB==1)
        {
            C=(XB[2]+VELOB*FTIME-XB[1])/(double)(NDIV); //new DDIV
            get (VAR , 1,N)=XB[1]+((double)(N-1)+0.5)*C;
        }
        get (VAR , 2,N)=SUM[0];

        for(int II = 0 ; II <11 ; II++)
        {
            cout << "SUM["<<II<<"]=" <<SUM[II] <<endl ;
        }

        if(SUM[1]>0.5)
        {
            get (VAR , 3,N)=SUM[1]*A; //--number density Eqn. (4.28)
            get (VAR , 4,N)=get (VAR , 3,N)*SUM[2]/SUM[1]; //--density  Eqn. (4.29)
            get (VAR , 5,N)=SUM[3]/SUM[2];//--u velocity component  Eqn. (4.30)
            get (VAR , 6,N)=SUM[4]/SUM[2]; //--v velocity component  Eqn. (4.30)
            get (VAR , 7,N)=SUM[5]/SUM[2]; //--w velocity component  Eqn. (4.30)
            UU= pow(get (VAR , 5,N),2)+pow(get (VAR , 6,N),2)+pow(get (VAR , 7,N),2);
            if(SUM[1]>1)
            {   
                get (VAR , 8,N)=(fabs(SUM[6]-SUM[2]*UU))/(3.e00*BOLTZ*SUM[1]); //Eqn. (4.39)
                //--translational temperature
                get (VAR , 19,N)=fabs(SUM[10]-SUM[2]*pow(get (VAR , 5,N),2))/(BOLTZ*SUM[1]);
                get (VAR , 20,N)=fabs(SUM[11]-SUM[2]*pow(get (VAR , 6,N),2))/(BOLTZ*SUM[1]);
                get (VAR , 21,N)=fabs(SUM[12]-SUM[2]*pow(get (VAR , 7,N),2))/(BOLTZ*SUM[1]);
            }
            else
            {
                get (VAR , 8,N)=1.0;
                get (VAR , 19,N)=1.0;
                get (VAR , 20,N)=1.0;
                get (VAR , 21,N)=1.0;
            }
            if(SUM[9]>0.1e00)
            {
                get (VAR , 9,N)=(2.e00/BOLTZ)*SUM[8]/SUM[9]; ////--rotational temperature Eqn. (4.36)
            }
            else
                get (VAR , 9,N)=0.0;
            
            get (VAR , 10,N)=FTMP[1]; ////vibration default
            DOF=(3.e00+SUM[9]/SUM[1]);
            get (VAR , 11,N)=(3.0*get (VAR , 8,N)+(SUM[9]/SUM[1])*get (VAR , 9,N))/DOF;
            //--overall temperature based on translation and rotation
            get (VAR , 18,N)=get (VAR , 3,N)*BOLTZ*get (VAR , 8,N);
            //--scalar pressure (now (from V3) based on the translational temperature)
            if(MMVM>0)
            {
                for(L=1;L<=MSP;L++)
                {
                    VDOF[L]=0.0;
                    //dout
                    if(get(ISPV  ,L) > 0)
                    {
                        for(K=1;K<=get(ISPV  ,L);K++)
                        {
                            if(get (CS,1+ K+9,N,L)<BOLTZ)
                            {
                                get(TV , K,L)=0.0;
                                get(DF , N,K,L)=0.0;
                            }
                            else
                            {
                                get(TV , K,L)=get(SPVM ,1,K,L)/log(1.0+get (CS,1+ 1,N,L)/get (CS,1+ K+9,N,L)) ;//--Eqn.(4.45)
                                get(DF , N,K,L)=2.0*(get (CS,1+ K+9,N,L)/get (CS,1+ 1,N,L))*log(1.0+get (CS,1+ 1,N,L)/get (CS,1+ K+9,N,L)); //--Eqn. (4.46)
                            }
                            VDOF[L]=VDOF[L]+get(DF , N,K,L);
                        }
                        //memget(TVIB,0.0,sizeof(*TVIB));
                        for(int i=0;i<MSP+1;i++)
                            TVIB[i]=0.0;
                        
                        for(K=1;K<=get(ISPV  ,L);K++)
                        {
                            if(VDOF[L]>1.e-6)
                            {
                                TVIB[L]=TVIB[L]+get(TV , K,L)*get(DF , N,K,L)/VDOF[L] ;
                            }
                            else
                                TVIB[L]=FVTMP[1] ;
                        }
                    }
                    else
                    {
                        TVIB[L]=TREF;
                        VDOF[L]=0.0;
                    }
                    get (VARSP , 7+1,N,L)=TVIB[L];
                }
                VDOFM=0.0;
                TVIBM=0.0;
                A=0.e00;
                for(L=1;L<=MSP;L++)
                {
                    //doubt
                    if(get(ISPV  ,L) > 0)
                    {
                        A=A+get (CS,1+ 1,N,L);
                    }
                }
                for(L=1;L<=MSP;L++)
                {
                    //dout
                    if(get(ISPV  ,L) > 0)
                    {
                        VDOFM=VDOFM+VDOF[L]*get (CS,1+ 1,N,L)/A;
                        TVIBM=TVIBM+TVIB[L]*get (CS,1+ 1,N,L)/A;
                    }
                }
                get (VAR , 10,N)=TVIBM;
            }
            for(L=1;L<=MSP;L++)
            {
                if(get (VARSP , 0+1,N,L)>0.5)
                {
                    //--convert the species velocity components to diffusion velocities
                    for(K=1;K<=3;K++)
                    {
                        get (VARSP , K+8+1,N,L)=get (VARSP , K+8+1,N,L)-get (VAR , K+4,N);
                    }
                    if(MELE>1)
                    {
                        //--calculate the electronic temperatures for the species
                        //memget(ELDOF,0.e00,sizeof(*ELDOF));
                        for(int i=0;i<MSP+1;i++)
                            ELDOF[i] = 0.e00;
                        //dout
                        //memget(TEL,0.e00,sizeof(*TEL));
                        for(int i=0;i<MSP+1;i++)
                            TEL[i] = 0.e00;
                        if(MELE>1)
                        {
                            A=0.e00;
                            B=0.e00;
                            for(M=1;M<=get(NELL  ,L);M++)
                            {
                                if(get (VARSP , 5+1,N,L)>1.e00)
                                {
                                    C=get(QELC ,2,M,L)/get (VARSP , 5+1,N,L);
                                    A=A+get(QELC ,1,M,L)*exp(-C);
                                    B=B+get(QELC ,1,M,L)*C*exp(-C);
                                }
                            }
                            if(B>1.e-10)
                            {
                                TEL[L]=(get (CS,1+ 9,N,L)/get (CS,1+ 1,N,L))/(BOLTZ*B/A);
                            }
                            else
                                TEL[L]=get (VAR , 11,N);
                            get (VARSP , 12+1,N,L)=TEL[L];
                            ELDOF[L]=0.e00;
                            if(get (VARSP , 5+1,N,L)>1.e00)
                                ELDOF[L]=2.e00*(get (CS,1+ 9,N,L)/get (CS,1+ 1,N,L))/(BOLTZ*get (VARSP , 5+1,N,L) ) ;
                            if(ELDOF[L]<0.01)
                            {
                                get (VARSP , 12+1,N,L)=get (VAR , 11,N);
                            }
                        }
                        else
                        {
                            ELDOF[L]=0.0;
                        }
                    }
                }
                else
                {
                    for(K=8;K<=12;K++)
                    {
                        get (VARSP , K+1,N,L)=0.e00;
                    }
                }
            }
            //--set the overall electronic temperature
            if(MELE>1)
            {
                C=0.e00;
                for(L=1;L<=MSP;L++)
                {
                    if(ELDOF[L]>1.e-5)
                        C=C+get (CS,1+ 1,N,L);
                }
                if(C>0.e00)
                {
                    A=0.e00;
                    B=0.e00;
                    for(L=1;L<=MSP;L++)
                    {
                        if(ELDOF[L]>1.e-5)
                        {
                            A=A+get (VARSP , 12+1,N,L)*get (CS,1+ 1,N,L);
                            B=B+get (CS,1+ 1,N,L);
                        }
                    }
                    get (VAR , 22,N)=A/B;
                }
                else{
                    get (VAR , 22,N)=get (VAR , 11,N);
                }
            }
            else{
                get (VAR , 22,N)=FTMP[1] ;
            }
            if(MMVM>0)
            {
                //--set the overall temperature and degrees of freedom for the individual species
                for(L=1;L<=MSP;L++)
                {
                    if(MELE>1){
                        SDOF[L]=3.e00+get(ISPR ,1,L)+VDOF[L]+ELDOF[L];
                        get (VARSP , 8+1,N,L)=(3.0*get (VARSP , 5+1,N,L)+get(ISPR ,1,L)*get (VARSP , 6+1,N,L)+VDOF[L]*get (VARSP , 7+1,N,L)+ELDOF[L]*get (VARSP , 12+1,N,L))/SDOF[L];
                    }
                    else{
                        SDOF[L]=3.e00+get(ISPR ,1,L)+VDOF[L];
                        get (VARSP , 8+1,N,L)=(3.0*get (VARSP , 5+1,N,L)+get(ISPR ,1,L)*get (VARSP , 6+1,N,L)+VDOF[L]*get (VARSP , 7+1,N,L))/SDOF[L];
                    }
                }
                //--the overall species temperature now includes vibrational and electronic excitation
                //--the overall gas temperature can now be set
                A=0.e00;
                B=0.e00;
                for(L=1;L<=MSP;L++)
                {
                    A=A+SDOF[L]*get (VARSP , 8+1,N,L)*get (CS,1+ 1,N,L);
                    B=B+SDOF[L]*get (CS,1+ 1,N,L);
                }
                get (VAR , 11,N)=A/B ;
            }
            VEL=sqrt(pow(get (VAR , 5,N),2)+pow(get (VAR , 6,N),2)+pow(get (VAR , 7,N),2));
            get (VAR , 12,N)=VEL/sqrt((DOF+2.e00)*get (VAR , 11,N)*(SUM[1]*BOLTZ/SUM[2])/DOF);
            //--Mach number
            get (VAR , 13,N)=SUM[0]/NSAMP; ////--average number of molecules in cell
            //dout
            if(COLLS[N] > 2.0)
            {
                get (VAR , 14,N)=0.5e00*(FTIME-TISAMP)*(SUM[1]/NSAMP)/WCOLLS[N];
                //--mean collision time
                get (VAR , 15,N)=0.92132e00*sqrt(fabs(SUM[7]/SUM[1]-UU))*get (VAR , 14,N);
                //--mean free path (based on r.m.s speed with correction factor based on equilibrium)
                get (VAR , 16,N)=CLSEP[N]/(COLLS[N]*get (VAR , 15,N));
            }
            else{
                get (VAR , 14,N)=1.e10;
                get (VAR , 15,N)=1.e10/get (VAR , 3,N);
                //--m.f.p set by nominal values
            }
        }
        else
        {
            for(L=3;L<=22;L++)
            {
                get (VAR , L,N)=0.0;
            }
        }
        get (VAR , 17,N)=VEL;
    }
    if(FTIME>0.5e00*DTM)
    {
        if(ICLASS==1){
            if(IFX==0)
                file_3<<"DSMC program for a one-dimensional plane flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
            if(IFX==1)
                file_3<<"DSMC program for a cylindrical flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
            if(IFX==2)
                file_3<<"DSMC program for a spherical flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
        }
        file_3<<endl;//WRITE (3,*)
        file_3<<"Interval "<<NOUT<<" Time "<<FTIME<< " with "<<NSAMP<<" samples from "<<TISAMP<<endl;
        //WRITE (3,*) 'Interval',NOUT,'Time ',FTIME, ' with',NSAMP,' samples from',TISAMP
        //990 FORMAT(I7,G13.5,I7,G13.5)
        //Dout
        NNN=TOTMOV;
        cout<<"TOTAL MOLECULES = "<< NM<<endl;
        //dout
        //NMS=0;
        for(int i=0;i<MSP+1;i++)
            NMS[i]=0;

        for(N=1;N<=NM;N++)
        {
            M=get(IPSP ,N);
            NMS[M]+=1;
        }
        file_3<<"Total simulated molecules = "<<NM<<endl;
        for(N=1;N<=MSP;N++)
        {
            cout<< " SPECIES "<<N<<" TOTAL = "<<NMS[N]<<endl;
            file_3<<"Species "<<N<<" total = "<<NMS[N]<<endl;
        }
        if(MEX>0)
        {
            ENERGY(0,A);
            for(N=1;N<=MSP;N++)
            {
                if(get(ISPV  ,N)>0){
                    file_9<< "SP "<<N<<" DISSOCS "<<TDISS[N]<<" RECOMBS "<<TRECOMB[N]<<endl;
                    cout<<"SP"<<N<<"DISSOCS"<<TDISS[N]<<" RECOMBS "<<TRECOMB[N]<<endl;
                    file_3<<"SP "<<N<<" DISSOCS "<<TDISS[N]<<" RECOMBS "<<TRECOMB[N]<<endl;
                }
            }
            for(N=1;N<=MEX;N++)
            {
                cout<<"EX,C reaction\t"<<N<<" number"<<TNEX[N]<<endl;
                file_9<<"EX,C reaction\t "<<N<<" number "<<TNEX[N]<<endl;
                file_3<<"EX,C reaction \t"<<N<<" number "<<TNEX[N]<<endl;
                
            }
        }
        
        file_3<<"Total molecule moves   = "<<NNN<<endl;
        //dout
        NNN=TOTCOL;
        file_3<<"Total collision events = "<<NNN<<endl;
        //
        file_3<<"Species dependent collision numbers in current sample"<<endl;
        for(N=1;N<=MSP;N++)
        {
            if(IGAS!=8){
                for(M=1;M<=MSP;M++)
                    file_3<<get (TCOL , N,M)<<"\t";
                file_3<<endl;
                //WRITE(3,901) (get (TCOL , N,M),M=1,MSP);
            }
            if(IGAS==8){
                for(M=1;M<=MSP;M++)
                    file_3<<get (TCOL , N,M)<<"\t";
                file_3<<endl;
                // WRITE(3,902) (get (TCOL , N,M),M=1,MSP);
            }
        }
        //Dout
        //901 FORMAT(5G13.5)
        //902 FORMAT(8G13.5)
        //dout
        CTIME=clock();
        file_3<<"Computation time "<<(double)CTIME/1000.0<< "seconds"<<endl;
        file_3<<"Collision events per second "<<(TOTCOL-TOTCOLI)*1000.e00/(double)CTIME<<endl;
        file_3<<"Molecule moves per secon "<<(TOTMOV-TOTMOVI)*1000.e00/(double)CTIME<<endl;
        if(ICLASS==0 && MMVM==0 && ISF==0){
            //--a homogeneous gas with no vibratioal modes - assume that it is a collision test run
            //*PRODUCES DATA FOR TABLES 6.1 AND 6.2 IN SECTION 6.2*
            //
            A=0.e00;
            B=0.e00;
            C=0.e00;
            for(N=1;N<=NCCELLS;N++)
            {
                A+=get (CCELL , 5,N);
                B+=get (CCELL , 4,N);
                C+=get (CCELL , 3,N);
            }
            file_3<<"Overall time step "<<DTM<<endl;
            file_3<<"Molecules per collision cell "<<(double)(NM)/(double)(NCCELLS)<<endl;
            file_3<<"Mean cell time ratio "<< A/((double)(NCCELLS)*FTIME)<<endl;
            file_3<<"Mean value of cross-section and relative speed "<<B/(double)(NCCELLS)<<endl;
            file_3<<"Mean half collision cell time step "<<C/(double)(NCCELLS)<<endl;
            if(MSP==1){
                A=2.e00*SPI*get (VAR , 3,1)  *(pow(get (SP , 1,1),2))*sqrt(4.e00*BOLTZ*get (SP , 2,1)/get (SP , 5,1))*pow((get (VAR , 11,1))/get (SP , 2,1),(1.e00-get (SP , 3,1)));
                //--Eqn. (2.33) for equilibhrium collision rate
                file_3<<"Coll. rate ratio to equilib "<<get (TCOL , 1,1)/((double)(NM)*(FTIME-TISAMP))/A<<endl;
            }
            else{
                file_3<<"Species collision rate ratios to equilibrium"<<endl;
                for(N=1;N<=MSP;N++){
                    file_3<<"Collision rate for species "<<N<<endl;
                    for(M=1;M<=MSP;M++)
                    {
                        THCOL[N,M]=2.e00*(1.e00/SPI)*get (VAR , 3,1)*get (VARSP , 1+1,1,M)*get (SPM , 2,N,M)*sqrt(2.e00*BOLTZ*get (SPM , 5,N,M)/get (SPM , 1,N,M))*pow(get (VAR , 11,1)/get (SPM , 5,N,M),1.e00-get (SPM , 3,N,M));
                        //--Eqn. (2.36) for equilibhrium collision rate of species N with species M
                        file_3<<" with species "<<M<<" is "<<get (TCOL , N,M)/((double)(NM)*get(FSP ,N,1)*(FTIME-TISAMP))/THCOL[N,M]<<endl;
                    }
                }
                file_3<<endl;
                for(N=1;N<=MSP;N++){
                    file_3<<"Collision numbers for species "<<N<<endl;
                    for(M=1;M<=MSP;M++){
                        file_3<<"with species "<<M<<" "<<get (TCOL , N,M)<<endl;
                    }
                }
            }
        }
        file_3<<endl;
        if(ITYPE[1]==2|| ITYPE[2]==2)
            file_3<<"Surface quantities"<<endl;
        for(JJ=1;JJ<=2;JJ++)
        {
            if(ITYPE[JJ]==2){
                file_3<<endl;
                file_3<<"Surface at "<<XB[JJ]<<endl;
                file_3<<"Incident sample "<<get (VARS ,1+ 0,JJ)<<endl;
                file_3<<"Number flux "<<get (VARS ,1+ 3,JJ)<<" /sq m/s"<<endl;
                file_3<<"Inc pressure "<<get (VARS ,1+ 5,JJ)<<" Refl pressure "<<get (VARS ,1+ 6,JJ)<<endl;
                file_3<<"Pressure "<< get (VARS ,1+ 5,JJ)+get (VARS ,1+ 6,JJ)<<" N/sq m"<<endl;
                file_3<<"Inc y shear "<<get (VARS ,1+ 7,JJ)<<" Refl y shear "<<get (VARS ,1+ 8,JJ)<<endl;
                file_3<<"Net y shear "<<get (VARS ,1+ 7,JJ)-get (VARS ,1+ 8,JJ)<<" N/sq m"<<endl;
                file_3<<"Net z shear "<<get (VARS ,1+ 9,JJ)-get (VARS ,1+ 10,JJ)<<" N/sq m"<<endl;
                file_3<<"Incident translational heat flux "<<get (VARS ,1+ 11,JJ)<<" W/sq m"<<endl;
                if(MMRM>0)
                    file_3<<"Incident rotational heat flux "<<get (VARS ,1+ 13,JJ)<<" W/sq m"<<endl;
                if(MMVM>0)
                    file_3<<"Incident vibrational heat flux "<<get (VARS ,1+ 15,JJ)<<" W/sq m"<<endl;
                if(MELE>1)
                    file_3<<"Incident electronic heat flux "<<get (VARS ,1+ 33,JJ)<<" W/sq m"<<endl;
                file_3<<"Total incident heat flux "<<get (VARS ,1+ 29,JJ)<<" W/sq m"<<endl;
                file_3<<"Reflected translational heat flux "<<get (VARS ,1+ 12,JJ)<<" W/sq m"<<endl;
                if(MMRM>0)
                    file_3<<"Reflected rotational heat flux "<<get (VARS ,1+ 14,JJ)<<" W/sq m"<<endl;
                if(MMVM>0)
                    file_3<<"Reflected vibrational heat flux "<<get (VARS ,1+ 16,JJ)<<" W/sq m"<<endl;
                if(MELE>1)
                    file_3<<"Reflected electronic heat flux "<<get (VARS ,1+ 34,JJ)<<" W/sq m"<<endl;
                file_3<<"Total reflected heat flux "<<get (VARS ,1+ 30,JJ)<<" W/sq m"<<endl;
                file_3<<"Net heat flux "<<get (VARS ,1+ 31,JJ)<<" W/sq m"<<endl;
                file_3<<"Slip velocity (y direction) "<<get (VARS ,1+ 19,JJ)<<" m/s"<<endl;
                file_3<<"Translational temperature slip"<<get (VARS ,1+ 20,JJ)<<" K"<<endl;
                if(MMRM>0)
                    file_3<<"Rotational temperature slip "<<get (VARS ,1+ 21,JJ)<<" K"<<endl;
                if(MSP>1)
                {
                    for(L=1;L<=MSP;L++)
                    {
                        file_3<<"Species "<<L<<" percentage "<<get (VARS ,1+ L+35,JJ)<<endl;
                    }
                }
            }
        }

        file_3<<endl;
        //PPA=0;
        for(int i=0;i<MSP+1;i++)
            PPA[i]=0;

        for(N=1;N<=NCELLS;N++)
        {
            for(M=1;M<=MSP;M++){
                PPA[M]=PPA[M]+get (VARSP , 1,N,M);
            }
        }
        // WRITE (*,*)
        //cin.get();
        if(MSP>1)
        {
            file_3<<"GAINS FROM REACTIONS"<<endl;
            file_3<<"                          Dissoc.     Recomb. Endo. Exch.  Exo. Exch."<<endl;
            for(M=1;M<=MSP;M++){
                file_3<<"                          SPECIES "<<M<<" "<<get (TREACG , 1,M)<<" "<<get (TREACG , 2,M)<<" "<<get (TREACG , 3,M)<<" "<<get (TREACG , 4,M)<<endl;
            }
            file_3<<endl;
            file_3<<"LOSSES FROM REACTIONS"<<endl;
            file_3<<"                          Dissoc.     Recomb. Endo. Exch.  Exo. Exch."<<endl;
            for(M=1;M<=MSP;M++){
                file_3<<"                          SPECIES "<<M<<" "<<get (TREACL , 1,M)<<" "<<get (TREACL , 2,M)<<" "<<get (TREACL , 3,M)<<" "<<get (TREACL , 4,M)<<endl;
            }
            file_3<<endl;
            file_3<<"TOTALS"<<endl;
            for(M=1;M<=MSP;M++){
                file_3<<"                        SPECIES "<<M<<" GAINS "<<get (TREACG , 1,M)+get (TREACG , 2,M)+get (TREACG , 3,M)+get (TREACG , 4,M)<<" LOSSES "<<get (TREACL , 1,M)+get (TREACL , 2,M)+get (TREACL , 3,M)+get (TREACL , 4,M)<<endl;
            }
        }
        file_3<<endl;
        file_3<<"Flowfield properties "<<endl;
        file_3<< NSAMP<<" Samples"<<endl;
        file_3<<"Overall gas"<<endl;
        cout << "NCELLS = " << NCELLS << endl ;
        file_3<<"Cell\tx coord.\tSample\tNumber Dens.\t Density\tu velocity\tv velocity\tw velocity\tTrans. Temp.\tRot. Temp.\tVib. Temp. \tEl. Temp. \tTemperature \tMach no. \tMols/cell\tm.c.t   \tm.f.p\tmcs/mfp\tspeed \tPressure \tTTX \tTTY\tTTZ\tSpecies Fractions "<<endl;
        for(N=1;N<=NCELLS;N++)
        {
            file_3<< N<<" \t";
            for(M=1;M<=10;M++){
                file_3<<get (VAR , M,N)<<"\t";
            }
            file_3<<get (VAR , 22,N)<<"\t ";
            for(M=11;M<=21;M++){
                file_3<<get (VAR , M,N)<<" \t";
            }
            for(L=1;M<=MSP;M++){
                file_3<<get (VARSP , 2,N,L)<<"\t ";
            }
            file_3<<endl;
        }

        cout <<" sum[2] = "<< SUM[2] << endl ; // dsuedit
        cout << " sum[3] = " << SUM[3] << endl ; // dsuedit
        file_3<<"Individual molecular species"<<endl;
        for(L=1;L<=MSP;L++){
            file_3<<"Species "<<L<<endl;
            file_3<<"Cell\t x coord.  \t    Sample  \t     Percentage \t  Species TTx  \t Species TTy \t Species TTz \t Trans. Temp.\t  Rot. Temp.\t  Vib. Temp. \t  Spec. Temp  \tu Diff. Vel.\t v Diff. Vel.\t w. Diff. Vel.\t Elec. Temp."<<endl;
            for(N=1;N<=NCELLS;N++){
                file_3<< N<<" "<<get (VAR , 1,N)<<" \t";
                for(M=0;M<=12;M++)
                    file_3<<get (VARSP , M+1,N,L)<<"\t ";
                file_3<<endl;
            }
        }
        //dout
        //999 FORMAT (I5,30G13.5)
        //998 FORMAT (G280.0)
        // 997 FORMAT (G188.0)
        // CLOSE (3)
        file_3.close();
    }
    if(ICLASS==0 && ISF==1){
        //--a homogeneous gas and the "unsteady sampling" option has been chosen-ASSUME THAT IT IS A RELAXATION TEST CASE FOR SECTION 6.2
        INITIALISE_SAMPLES();
        //write a special output file for internal temperatures and temperature versus collision number
        //dout
        file_10.open("RELAX.DAT", ios::app | ios::out);
        if(file_10.is_open()){
            cout<<"RELAX.DAT is opened"<<endl;
        }
        else{
            cout<<"RELAX.DAT not opened"<<endl;
        }
        // OPEN (10,FILE='RELAX.DAT',ACCESS='APPEND')
        A=2.0*TOTCOL/NM; //--mean collisions
        //--VAR(11,N)   //--overall
        //--VAR(8,N)    //--translational
        //--VAR(9,N)    //--rotational
        //--VAR(10,N)   //--vibrational
        //--VAR(22,N)   //--electronic
        //file_10<<std::right<<setw(15)<<A<<setw(15)<<get (VAR , 8,1)<<setw(15)<<get (VAR , 9,1)<<setw(15)<<get (VAR , 8,1)-get (VAR , 9,1)<<endl;
        file_10<<std::right<<setw(15)<<A<<setw(15)<<get (VAR , 11,1)<<setw(15)<<get (VAR , 8,1)<<setw(15)<<get (VAR , 9,1)<<setw(15)<<get (VAR , 10,1)<<setw(15)<<get (VAR , 22,1)<<endl;
        //file_10<<std::right<<setw(15)<<A<<setw(15)<<get (VAR , 8,1]<<setw(15)<<get (VAR , 9,1]<<setw(15)<<get (VAR , 8,1]-get (VAR , 9,1]<<endl;
        //  WRITE (10,950) A,VAR(8,1),VAR(9,1),VAR(8,1)-VAR(9,1)   //--Generates output for Figs. 6.1 and 6.2
        //  WRITE (10,950) A,VAR(11,1),VAR(8,1),VAR(9,1),VAR(10,1),VAR(22,1)   //--Generates output for modal temperatures in Figs. 6.3, 6.5 +
        //  WRITE (10,950) A,0.5D00*(VAR(8,1)+VAR(9,1)),VAR(10,1),0.5D00*(VAR(8,1)+VAR(9,1))-VAR(10,1)  //--Generates output for Figs. 6.4
        //
        //--VARSP(8,N,L) //--overall temperature of species L
        //  WRITE (10,950) A,VARSP(8,1,3),VARSP(8,1,2),VARSP(8,1,5),VARSP(8,1,4),A  //--output for Fig 6.17
        // CLOSE (10)
        file_10.close();
    }
    //dout
    // 950 FORMAT (6G13.5)
    if(IGAS==8||IGAS==6||IGAS==4)
    {
        //--Write a special output file for the composition of a reacting gas as a function of time
        //dout
        //OPEN (10,FILE='COMPOSITION.DAT',ACCESS='APPEND')
        file_10.open("COMPOSITION.DAT", ios::app | ios::out);
        if(file_10.is_open()){
            cout<<"COMPOSITION.DAT is opened"<<endl;
        }
        else{
            cout<<"COMPOSITION.DAT not opened"<<endl;
        }
        AS=NM;
        //dout
        AT=FTIME*1.e6;
        if (IGAS == 4)
            file_10<< AT <<" "<<(double)(NMS[1])/1000000<<" "<<A<<" "<<get (VAR , 11,1)<<endl;    //--Data for fig
        if (IGAS == 8)
            file_10<<AT<<" "<<NMS[1]/AS<<" "<<NMS[2]/AS<<" "<<NMS[3]/AS<<" "<<NMS[4]/AS<<" "<<NMS[5]/AS<<" "<<NMS[6]/AS<<" "<<NMS[7]/AS<<" "<<NMS[8]/AS<<" "<<get (VAR , 11,1)<<endl;
        if (IGAS == 6)
            file_10<<AT<<" "<<NMS[1]/AS<<" "<<NMS[2]/AS<<" "<<NMS[3]/AS<<" "<<NMS[4]/AS<<" "<<NMS[5]/AS<<" "<<get (VAR , 11,1)<<endl;
        //dout
        // 888 FORMAT(10G13.5)
        file_10.close();
    }
    if(FTIME>0.5e00*DTM){
        //
        //--reset collision and transit times etc.
        //
        cout<<"Output files written "<<endl;
        DTMI=DTM;
        if(IMTS<2){
            if(ICLASS>0)
                DTM*=2;
            //--this makes it possible for DTM to increase, it will be reduced as necessary
            for(NN=1;NN<=NCELLS;NN++)
            {
                CDTM[NN]=DTM;
                B=get (CELL , 3,NN)-get (CELL , 2,NN) ;//--sampling cell width
                if(get (VAR , 13,NN)>20.e00){
                    //consider the local collision rate
                    CDTM[NN]=get (VAR , 14,NN)*CPDTM;
                    //look also at sampling cell transit time based on the local flow speed
                    A=(B/(fabs(get (VAR , 5,NN))))*TPDTM;
                    if(A<CDTM[NN])
                        CDTM[NN]=A;
                }
                else{
                    //-- base the time step on a sampling cell transit time at the refence vmp
                    A=TPDTM*B/VMPM;
                    if(A<CDTM[NN])
                        CDTM[NN]=A;
                }
                if(CDTM[NN]<DTM)
                    DTM=CDTM[NN];
            }
        }
        else
        {
            //dout
            //memget(CDTM, DTM, sizeof(*CDTM));
            for(int i=0;i<NCELLS+1;i++)
                CDTM[i]= DTM;
            //CDTM=DTM;
        }
        for(N=1;N<=NCELLS;N++){
            NN=get (ICCELL , 3,N);
            get (CCELL , 3,N)=0.5*CDTM[NN];
        }
        file_9<<"DTM changes  from "<<DTMI<<" to "<<DTM<<endl;
        DTSAMP=DTSAMP*DTM/DTMI;
        DTOUT=DTOUT*DTM/DTMI;
    }
    else
    {
        INITIALISE_SAMPLES();
    }
    if(ICLASS==1&& ISF==1)
    {
        //****
        //--write TECPLOT data files for x-t diagram (unsteady calculation only)
        //--comment out if not needed
        //dout
        file_18.open("DS1xt.DAT", ios::app | ios::out);
        if(file_18.is_open()){
            cout<<"DS1xt.DAT is opened"<<endl;
        }
        else
            cout<<"DS1xt.DAT not opened"<<endl;
        // OPEN (18,FILE='DS1xt.DAT',ACCESS='APPEND')
        //--make sure that it is empty at the stary of the run
        //SETXT();
        // CLOSE (18)
        file_18.close();
        //****
    }
    //WRITE (19,*) FTIME,-get (VARS ,1+ 5,1],-get (VARS ,1+ 5,1]-get (VARS , 6,1]
    
    file_7.open("PROFILE.DAT" , ios::out);
    if(file_7.is_open()){
        cout<<"PROFILE.DAT is opened"<<endl;
    }
    else
        cout<<"PROFILE.DAT not opened"<<endl;
    // OPEN (7,FILE='PROFILE.DAT',FORM='FORMATTED')
    //
    //OPEN (8,FILE='ENERGYPROF.DAT',FORM='FORMATTED')
    //
    // 995 FORMAT (22G13.5)
    // 996 FORMAT (12G14.6)
    for(N=1;N<=NCELLS;N++)
    {
        //
        //--the following line is the default output
        //  WRITE (7,995) VAR(1,N),VAR(4,N),VAR(3,N),VAR(11,N),VAR(18,N),VAR(5,N),VAR(12,N),VAR(8,N),VAR(9,N),VAR(10,N),VAR(22,N),     &
        //        (VARSP(8,N,M),M=1,MSP),(VARSP(1,N,M),M=1,MSP)
        //
        //--calculate energies per unit mass (employed for re-entry shock wave in Section 7.5)
        C1=0.5e00*pow(get (VAR , 5,N),2);    //--Kinetic
        C2=0.e00;                 //--Thermal
        C3=0.e00;                //--Rotational
        C4=0.e00;               //--Vibrational
        C5=0.e00;              //--Electronic
        C6=0.e00;             //--Formation
        for(L=1;L<=MSP;L++)
        {
            //    C2=C2+3.D00*BOLTZ*VARSP(5,N,L)*VARSP(1,N,L)/SP(5,L)
            A=(get (CS,1+ 1,N,L)/get (VARSP , 2,N,L))*get (SP , 5,L);
            if(get (CS,1+ 1,N,L)>0.5e00){
                C2=C2+0.5e00*(get (CS,1+ 5,N,L)+get (CS,1+ 6,N,L)+get (CS,1+ 7,N,L))*get (SP , 5,L)/A;
                if(get(ISPR ,1,L)>0)
                    C3=C3+get (CS,1+ 8,N,L)/A;
                if(get(ISPV  ,L)>0)
                    C4=C4+get (CS,1+ 10,N,L)*BOLTZ*get(SPVM ,1,1,L)/A;
                if(get(NELL  ,L)>1)
                    C5=C5+get (CS,1+ 9,N,L)/A;
                C6=C6+get (SP , 6,L)*get (CS,1+ 1,N,L)/A;
            }
        }
        C2=C2-C1;
        //  A=0.5D00*VFX(1)*2+2.5D00*BOLTZ*FTMP(1)/(0.75*SP(5,2)+0.25*SP(5,1))
        C7=C1+C2+C3+C4+C5+C6;
        //
        //  WRITE (8,995) VAR(1,N),C1/A,C2/A,C3/A,C4/A,C5/A,C6/A,C7/A
        //
        //--the following lines are for normalised shock wave output in a simple gas (Sec 7.3)
        C1=FND[2]-FND[1];
        C2=FTMP[2]-FTMP[1];
        
        file_7<<get (VAR , 1,N)<<" "<<get (VAR , 2,N)<<" "<<(0.5*(get (VAR , 20,N)+get (VAR , 21,N))-FTMP[1])/C2<<" "<<(get (VAR , 19,N)-FTMP[1])/C2<<" "<<(get (VAR , 11,N)-FTMP[1])/C2<<" "<<(get (VAR , 3,N)-FND[1])/C1<<endl;
        //--the following replaces sample size with density
        //C3=0.D00
        //DO L=1,MSP
        //  C3=C3+FND(1)*FSP(L,1)*SP(5,L)  //--upstream density
        //END DO
        //C4=0.D00
        //DO L=1,MSP
        //  C4=C4+FND(2)*FSP(L,2)*SP(5,L)  //--upstream density
        //END DO
        //
        //  WRITE (7,996) VAR(1,N),(VAR(4,N)-C3)/(C4-C3),(0.5*(VAR(20,N)+VAR(21,N))-FTMP(1))/C2,(VAR(19,N)-FTMP(1))/C2,(VAR(11,N)-FTMP(1))/C2,    &
        //        (VAR(3,N)-FND(1))/C1
        //--the following lines is for a single species in a gas mixture
        //  C1=C1*FSP(3,1)
        //  WRITE (7,996) VAR(1,N),VARSP(1,N,3),(0.5*(VARSP(3,N,3)+VARSP(4,N,3))-FTMP(1))/C2,(VARSP(2,N,3)-FTMP(1))/C2,(VARSP(5,N,3)-FTMP(1))/C2,(VAR(3,N)*VARSP(1,N,3)-FND(1)*FSP(3,1))/C1
        //
        //--the following line is for Couette flow (Sec 7.4)
        //  WRITE (7,996) VAR(1,N),VAR(2,N),VAR(5,N),VAR(6,N),VAR(7,N),VAR(11,N)
        //--the following line is for the breakdown of equilibrium in expansions (Sec 7.10)
        //  WRITE (7,996) VAR(1,N),VAR(2,N),VAR(12,N),VAR(4,N),VAR(5,N),VAR(8,N),VAR(9,N),VAR(10,N),VAR(11,N),VAR(19,N),VAR(20,N),VAR(21,N)
        //
    }
    if(ISF==1)
        INITIALISE_SAMPLES();
    // CLOSE(7)
    file_7.close();
    //
    //--deallocate local variables
    //
    //dout
    
    // DEALLOCATE (TV,TVIB,VDOF,THCOL,STAT=ERROR)
    // if(ERROR)
    //     cout<<"PROGRAM COULD NOT DEALLOCATE OUTPUT VARIABLES"<<ERROR;
    TOUT=TOUT+DTOUT;
   
    return;

}

void cuda_collisions(int N)
{
    //CALC calc;
    //MOLECS molecs;
    //GAS gas;
    //OUTPUT output;
    //GEOM_1D geom;
    int NN,M,MM,L,LL,K,KK,KT,J,I,II,III,NSP,MAXLEV,IV,NSEL,KV,LS,MS,KS,JS,IIII,LZ,KL,IS,IREC,NLOOP,IA,IDISS,IEX,NEL,NAS,NPS,
    JJ,LIMLEV,KVV,KW,INIL,INIM,JI,LV,IVM,NMC,NVM,LSI,JX,MOLA,KR,JKV,NSC,KKV,IAX,NSTEP,NTRY,NLEVEL,NSTATE,IK,NK,MSI ;
    double A,AA,AAA,AB,B,BB,BBB,ABA,ASEL,DTC,SEP,VR,VRR,ECT,EVIB,ECC,ZV,ERM,C,OC,SD,D,CVR,PROB,RML,RMM,ECTOT,ETI,EREC,ET2,
    XMIN,XMAX,WFC,CENI,CENF,VRRT,EA,DEN,E1,E2,VRI,VRA ;
    double VRC[4],VCM[4],VRCP[4],VRCT[4];
    //   //N,M,K working integer
    // //LS,MS,KS,JS molecular species
    // //VRC components of the relative velocity
    // //RML,RMM molecule mass parameters
    // //VCM components of the center of mass velocity
    // //VRCP post-collision components of the relative velocity
    // //SEP the collision partner separation
    // //VRR the square of the relative speed
    // //VR the relative speed
    // //ECT relative translational energy
    // //EVIB vibrational energy
    // //ECC collision energy (rel trans +vib)
    // //MAXLEV maximum vibrational level
    // //ZV vibration collision number
    // //SDF the number of degrees of freedom associated with the collision
    // //ERM rotational energy
    // //NSEL integer number of selections
    // //NTRY number of attempts to find a second molecule
    // //CVR product of collision cross-section and relative speed
    // //PROB a probability
    // //KT third body molecule code
    // //ECTOT energy added at recmbination
    // //IREC initially 0, becomes 1 of a recombination occurs
    // //WFC weighting factor in the cell
    // //IEX is the reaction that occurs (1 if only one is possible)
    // //EA activation energy
    // //NPS the number of possible electronic states
    // //NAS the number of available electronic states
    //cout<<"START COLLISIONS"<<endl;
    
       
        if((FTIME-get (CCELL , 5,N)) > (get (CCELL , 3,N)))
        {

            DTC=2.e00*get (CCELL , 3,N);
            //calculate collisions appropriate to  time DTC
            if(get (ICCELL , 2,N)>1)
            {
                //no collisions calculated if there are less than two molecules in collision cell
                NN=get (ICCELL , 3,N);
                WFC=1.e00;
                if(IWF==1 && IVB==0)
                {
                    //dout
                    WFC=1.e00+WFM*powf(get (CELL , 1,NN),IFX);
                }
                get (CCELL , 5,N)=get (CCELL , 5,N)+DTC ;
                if(IVB==0)
                {
                    AAA=get (CCELL , 1,N);
                }
                if(IVB==1)
                {
                    C=(XB[2]+VELOB*FTIME-XB[1])/(double)(NDIV*NCIS);
                    //dout
                    XMIN=XB[1]+(double)(N-1)*C;
                    XMAX=XMIN+C;
                    //dout
                    WFC=1.e00+WFM*powf((0.5e00*(XMIN+XMAX)),IFX);
                    if(IFX==0)
                    {
                        AAA=XMAX-XMIN;
                    }
                    if(IFX==1)
                    {
                        AAA=PI*(powf(XMAX,2)-powf(XMIN,2)); //assumes unit length of full cylinder
                    }
                    if(IFX==2)
                    {
                        AAA=1.33333333333333333333e00*PI*(powf(XMAX,3)-powf(XMIN,3));    //flow is in the full sphere
                    }
                }
                //these statements implement the N(N-1) scheme
                ASEL=0.5e00*get (ICCELL , 2,N)*(get (ICCELL , 2,N)-1)*WFC*FNUM*get (CCELL , 4,N)*DTC/AAA+get (CCELL , 2,N);
                NSEL=ASEL;
                //dout
                get (CCELL , 2,N)=ASEL-(double)(NSEL);
                if(NSEL>0)
                {
                    I=0; //counts the number of selections
                    KL=0; //becomes 1 if it is the last selection
                    IIII=0; //becomes 1 if there is a recombination
                    for(KL=1;KL<=NSEL;KL++)
                    {
                        I=I+1;
                        III=0; //becomes 1 if there is no valid collision partner
                        if(get (ICCELL , 2,N)==2)
                        {
                            K=1+get (ICCELL , 1,N);
                            //dout
                            L=ICREF[K];
                            K=2+get (ICCELL , 1,N);
                            //dout
                            M=ICREF[K];
                            if(M==IPCP[L])
                            {
                                III=1;
                                get (CCELL , 5,N)=get (CCELL , 5,N)-DTC;
                            }
                        }
                        else
                        {
                            //dout
                            //                            RANDOM_NUMBER(RANF);
                            RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                            K=(int)(RANF*(double)(get (ICCELL , 2,N)))+get (ICCELL , 1,N)+1;
                            //dout
                            L=ICREF[K];
                            //one molecule has been selected at random
                            if(NNC==0)
                            {
                                //select the collision partner at random
                                M=L;
                                NTRY=0;
                                while(M==L)
                                {
                                    //dout
                                    //                                    RANDOM_NUMBER(RANF);
                                    RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                    K=(int)(RANF*(double)(get (ICCELL , 2,N)))+get (ICCELL , 1,N)+1;
                                    M=ICREF[K];
                                    if(M==IPCP[L])
                                    {
                                        if(NTRY<5*get (ICCELL , 2,N))
                                        {
                                            M=L;
                                        }
                                        else
                                        {
                                            III = 1;
                                            get (CCELL , 5,N)=get (CCELL , 5,N)-DTC/ASEL;
                                            M=L+1;
                                        }
                                    }
                                }
                            }
                            else
                            {
                                //elect the nearest from the total number (< 30) or a random 30
                                if(get (ICCELL , 2,N)<30)
                                {
                                    LL=get (ICCELL , 2,N);
                                }
                                else
                                {
                                    LL=30;
                                }
                                SEP=1.0e10;
                                M=0;
                                for(J=1;J<=LL;J++)
                                {
                                    if(LL<30)
                                    {
                                        K=J+get (ICCELL , 1,N);
                                    }
                                    else
                                    {
                                        //                                        RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        K=(int)(RANF*(double)(get (ICCELL , 2,N)))+get (ICCELL , 1,N)+1;
                                    }
                                    MM=ICREF[K];
                                    if(MM != L)
                                    {
                                        //exclude the already selected molecule
                                        if(MM != IPCP[L])
                                        {
                                            //exclude the previous collision partner
                                            //dout
                                            A=fabsf(get (PX , 1,L)-get (PX , 1,MM));
                                            if(A<SEP&& A>1.e-8*DDIV)
                                            {
                                                M=MM;
                                                SEP=A;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                        if(III==0)
                        {
                            for(KK=1;KK<=3;KK++)
                            {
                                VRC[KK]=get(PV  ,KK,L)-get(PV  ,KK,M);
                            }
                            VRR=VRC[1]*VRC[1]+VRC[2]*VRC[2]+VRC[3]*VRC[3];
                            VR=sqrtf(VRR);
                            VRI=VR;
                            //Simple GAs
                            if(MSP==1)
                            {
                                //dout
                                CVR=VR*CXSS*powf(2.e00*BOLTZ*get (SP , 2,1)/(RMAS*VRR),(get (SP , 3,1)-0.5e00))*RGFS;
                                if(CVR>get (CCELL , 4,N))
                                {
                                    get (CCELL , 4,N)=CVR;
                                }
                                //dout
                                //      RANDOM_NUMBER(RANF);
                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                if(RANF<CVR/get (CCELL , 4,N))
                                {
                                    // the collision occurs
                                    if(M==IPCP[L]&& L==IPCP[M])
                                    {
                                        //file_9<<"Duplicate collision"<<endl;
                                    }
                                    //atomicAdd(&TOTCOL,1.e00);
                                    //TOTCOL=TOTCOL+1.e00;
                                    get (COLL_TOTCOL , N)=get (COLL_TOTCOL , N)+1.e00;    //problem
                                    get (TCOL , 1,1)=get (TCOL , 1,1)+2.e00;    //problem
                                    COLLS[NN]=COLLS[NN]+1.e000;     //problem
                                    WCOLLS[NN]=WCOLLS[NN]+WFC;
                                    //dout
                                    SEP=fabsf(get (PX , 1,L)-get (PX , 1,M));
                                    CLSEP[NN]=CLSEP[NN]+SEP;
                                    if(get(ISPR ,1,1)>0)
                                    {
                                        //Larsen-Borgnakke serial redistribution
                                        ECT=0.5e00*RMAS*VRR;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            //consider the molecules in turn
                                            if(NSP==1)
                                            {
                                                K=L;
                                            }
                                            else
                                            {
                                                K=M;
                                            }
                                            if(MMVM>0)
                                            {
                                                if(get(ISPV  ,1)>0)
                                                {
                                                    for(KV=1;KV<=get(ISPV  ,1);KV++)
                                                    {
                                                        EVIB=(double)(get (IPVIB , KV,K)*BOLTZ*get(SPVM ,1,KV,1));
                                                        ECC=ECT+EVIB;
                                                        if(get(SPVM ,3,KV,1)>0.0)
                                                        {
                                                            MAXLEV=ECC/(BOLTZ*get(SPVM ,1,KV,1));
                                                            B=get(SPVM ,4,KV,1)/get(SPVM ,3,KV,1); //Tdiss/Tref
                                                            A= get(SPVM ,4,KV,1)/get (VAR , 8,NN) ;//Tdiss/Ttrans
                                                            //ZV=(A*SPM(3,1,1))*(SPVM(3,KV,1)*(B*(-SPM(3,1,1))))*(((A*0.3333333D00)-1.D00)/((B*0.33333D00)-1.D00))
                                                            ZV=powf(A,get (SPM , 3,1,1))*powf(get(SPVM ,3,KV,1)*powf(B,-get (SPM , 3,1,1)),((powf(A,0.3333333e00)-1e00)/(powf(B,33333e00)-1.e00)));
                                                        }
                                                        else
                                                        {
                                                            ZV=get(SPVM ,2,KV,1);
                                                            MAXLEV=ECC/(BOLTZ*get(SPVM ,1,KV,1))+1;
                                                        }
                                                        //dout
                                                        //                                                        RANDOM_NUMBER(RANF);
                                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                        if(1.e00/ZV>RANF)
                                                        {
                                                            II=0;
                                                            while(II==0)
                                                            {
                                                                //dout
                                                                //                                                                RANDOM_NUMBER(RANF);
                                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                IV=RANF*(MAXLEV+0.99999999e00);
                                                                get (IPVIB , KV,K)=IV;
                                                                EVIB=(double)(IV)*BOLTZ;
                                                                if(EVIB<ECC)
                                                                {
                                                                    PROB=powf((1.e00-EVIB/ECC),(1.5e00-get (SPM , 3,KV,1)));
                                                                    //PROB is the probability ratio of eqn (3.28)
                                                                    //dout
                                                                    //                                                                    RANDOM_NUMBER(RANF);
                                                                    RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                    if(PROB>RANF)
                                                                        II=1;
                                                                }
                                                            }
                                                            ECT=ECC-EVIB;
                                                        }
                                                    }
                                                }
                                            }
                                            //now rotation of this molecule
                                            //dout
                                            if(get(ISPR ,1,1) > 0)
                                            {
                                                if(get(ISPR ,2,1)==0)
                                                {
                                                    B=1.e00/get(SPR ,1,1);
                                                }
                                                else //use molecule rather than mean value
                                                {
                                                    B=1.e00/(get(SPR ,1,1)+get(SPR ,2,1)*get (VAR , 8,NN)+get(SPR ,3,1)*powf(get (VAR , 8,NN),2));
                                                }
                                                //dout
                                                //                                                RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                if(B>RANF)
                                                {
                                                    ECC=ECT +PROT[K];
                                                    if(get(ISPR ,1,1)==2)
                                                    {
                                                        //dout
                                                        //                                                        RANDOM_NUMBER(RANF);
                                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                        ERM=1.e00-powf(RANF,1.e00/(2.5e00-get (SP , 3,1))); //eqn(5.46)
                                                    }
                                                    else
                                                    {
                                                        //dout
                                                        LBS(0.5e00*get(ISPR ,1,1)-1.e00,1.5e00-get (SP , 3,1),ERM);
                                                    }
                                                    PROT[K]=ERM*ECC;
                                                    ECT=ECC-PROT[K];
                                                }
                                            }
                                        }
                                        //adjust VR for the change in energy;
                                        VR=sqrtf(2.e00*ECT/get (SPM , 1,1,1));
                                    }
                                    //end of L-B redistribution
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        VCM[KK]=0.5e00*(get(PV  ,KK,L)+get(PV  ,KK,M));
                                    }
                                    //dout
                                    if(fabsf(get (SP , 4,1)-1.0) < 0.001)
                                    {
                                        //use the VHS logic //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*RANF-1.e00;
                                        //B is the cosine of a random elevation angle
                                        A=sqrtf(1.e00-B*B);
                                        VRCP[1]=B*VR;
                                        //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*RANF;
                                        //C is a random azimuth angle
                                        //dout
                                        VRCP[2]=A*cos(C)*VR;
                                        VRCP[3]=A*sin(C)*VR;
                                    }
                                    else
                                    {
                                        //use the VSS logic //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*(powf(RANF,get (SP , 4,1)))-1.e00;
                                        //B is the cosine of the deflection angle for the VSS model (Eqn. 11.8) of Bird(1994))
                                        A=sqrtf(1.e00-B*B);
                                        //dout
                                        //                                                 RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*RANF;
                                        //dout
                                        OC=(double)cos(C);
                                        SD=(double)sin(C);
                                        D=sqrtf(powf(VRC[2],2)+powf(VRC[3],2));
                                        VRA=VR/VRI;
                                        VRCP[1]=(B*VRC[1]+A*SD*D)*VRA;
                                        VRCP[2]=(B*VRC[2]+A*(VRI*VRC[3]*OC-VRC[1]*VRC[2]*SD)/D)*VRA;
                                        VRCP[3]=(B*VRC[2]+A*(VRI*VRC[2]*OC-VRC[1]*VRC[3]*SD)/D)*VRA;
                                        //the post-collision rel. velocity components are based on eqn (3.18)
                                    }
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        get(PV  ,KK,L)=VCM[KK]+0.5e00*VRCP[KK];
                                        get(PV  ,KK,M)=VCM[KK]-0.5e00*VRCP[KK];
                                    }
                                    IPCP[L]=M;
                                    IPCP[M]=L;
                                }
                            } //collision occurrence
                            
                            else
                            {
                                //Gas Mixture
                                LS=fabsf(get(IPSP ,L));
                                MS=fabsf(get(IPSP ,M));
                                CVR=VR*get (SPM , 2,LS,MS)*powf(((2.e00*BOLTZ*get (SPM , 5,LS,MS))/((get (SPM , 1,LS,MS))*VRR)),(get (SPM , 3,LS,MS)-0.5e00))*get (SPM , 6,LS,MS);
                                if(CVR>get (CCELL , 4,N))
                                {
                                    get (CCELL , 4,N)=CVR;
                                }
                                //dout
                                //                                    RANDOM_NUMBER(RANF);
                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                if(RANF<CVR/get (CCELL , 4,N) && get(IPCELL  ,L)>0 && get(IPCELL  ,M)>0)
                                {
                                    //the collision occurs (-ve IPCELL indicates recombined molecule marled for removal)
                                    if(M==IPCP[L] && L==IPCP[M])
                                    {
                                        //file_9<<"Duplicate collision";
                                    }
                                    //atomicAdd(&TOTCOL,1.e00);
                                    //TOTCOL=TOTCOL+1.e00;
                                    get (COLL_TOTCOL , N)=get (COLL_TOTCOL , N)+1.e00;
                                    get (TCOL , LS,MS)=get (TCOL , LS,MS)+1.e00;
                                    get (TCOL , MS,LS)=get (TCOL , MS,LS)+1.e00;
                                    COLLS[NN]=COLLS[NN]+1.e00;
                                    WCOLLS[NN]=WCOLLS[NN]+WFC;
                                    SEP=fabsf(get (PX , 1,L)-get (PX , 1,M));
                                    CLSEP[NN]=CLSEP[NN]+SEP;
                                    RML=get (SPM , 1,LS,MS)/get (SP , 5,MS);
                                    RMM=get (SPM , 1,LS,MS)/get (SP , 5,LS);
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        VCM[KK]=RML*get(PV  ,KK,L)+RMM*get(PV  ,KK,M);
                                    }
                                    IDISS=0;
                                    IREC=0;
                                    IEX=0;
                                    //check for dissociation
                                    if(get(ISPR ,1,LS)>0 || get(ISPR ,1,MS)>0)
                                    {
                                        ECT=0.5e00*get (SPM , 1,LS,MS)*VRR;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            if(NSP==1)
                                            {
                                                K=L; KS=LS; JS=MS;
                                            }
                                            else
                                            {
                                                K=M ; KS=MS ; JS=LS;
                                            }
                                            if(MMVM>0)
                                            {
                                                if(get(ISPV  ,KS)>0)
                                                {
                                                    for(KV=1;KV<=get(ISPV  ,KS);KV++)
                                                    {
                                                        if(get (IPVIB , KV,K)>=0 && IDISS==0)
                                                        {
                                                            //do not redistribute to a dissociating molecule marked for removal
                                                            EVIB=(double)(get (IPVIB , KV,K)*BOLTZ*get(SPVM ,1,KV,KS));
                                                            ECC=ECT+EVIB;
                                                            MAXLEV=ECC/(BOLTZ*get(SPVM ,1,KV,KS));
                                                            LIMLEV=get(SPVM ,4,KV,KS)/get(SPVM ,1,KV,KS);
                                                            if(MAXLEV > LIMLEV)
                                                            {
                                                                //dissociation occurs subject to reduction factor  -  reflects the infinity of levels past the dissociation limit
                                                                //dout
                                                                //                                                                    RANDOM_NUMBER(RANF)
                                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                if(RANF<get(SPVM ,5,KV,KS))
                                                                {
                                                                    IDISS=1;
                                                                    LZ=get (IPVIB , KV,K);
                                                                    //NDISSL[LZ]=NDISSL[LZ]+1;
                                                                    ECT=ECT-BOLTZ*get(SPVM ,4,KV,KS)+EVIB;
                                                                    //adjust VR for the change in energy
                                                                    VRR=2.e00*ECT/get (SPM , 1,LS,MS);
                                                                    VR=sqrtf(VRR);
                                                                    get (IPVIB , KV,K)=-1;
                                                                    //a negative IPVIB marks a molecule for dissociation
                                                                }
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                        }
                                    }
                                    IEX=0;    //becomes the reaction number if a reaction occurs
                                    IREC=0;   //becomes 1 if a recombination occurs
                                    if(IDISS==0)
                                    {
                                        //dissociation has not occurred
                                        //consider possible recombinations
                                        if(get(ISPRC ,LS,MS)>0 && get (ICCELL , 2,N)>2)
                                        {
                                            //possible recombination using model based on collision volume for equilibrium
                                            KT=L;
                                            //NTRY=0
                                            while(KT==L||KT==M)
                                            {
                                                NTRY+=1;
                                                // if(NTRY>100)
                                                // {
                                                //  cout>>"NTRY 3rd body"<<NTRY;
                                                // }
                                                //RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);\
                                                K=(int)(RANF*(double)(get (ICCELL , 2,N]))+get (ICCELL , 1,N]+1;
                                                KT=ICREF[K];
                                            }
                                            KS=get(IPSP ,KT);
                                            //the potential third body is KT OF species KS
                                            AA=(PI/6.e00)*powf((get (SP , 1,LS)+get (SP , 1,MS)+get (SP , 1,KS)),3); //reference volume
                                            BB=AA*get( SPRC ,1,LS,MS,KS)*powf(get (VAR , 8,NN)/get(SPVM ,1,get(ISPRK ,LS,MS),get(ISPRC ,LS,MS)),get( SPRC ,2,LS,MS,KS));//collision volume
                                            B=BB*get (ICCELL , 2,N)*FNUM/AAA;
                                            if(B>1.e00)
                                            {
                                                printf("THREE BODY PROBABILITY %f\n", B);
                                                //cout<<"THREE BODY PROBABILITY"<<B;
                                                //for low density flows in which three-body collisions are very rare, it is advisable to consider recombinations in only a small
                                                //fraction of collisions and to increase the pribability by the inverse of this fraction.  This message provides a warning if this
                                                //factor has been set to an excessively large value
                                            }
                                            //dout
                                            //                                                RANDOM_NUMBER(RANF);
                                            RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                            if(RANF<B)
                                            {
                                                IREC=1;
                                                TRECOMB[get(ISPRC ,LS,MS)]=TRECOMB[get(ISPRC ,LS,MS)]+1.e00;
                                                //the collision now becomes a collision between these with L having the center of mass velocity
                                                A=0.5e00*get (SPM , 1,LS,MS)*VRR ;//the relative energy of the recombining molecules
                                                if(get(ISPR ,1,LS)>0)
                                                    A=A+PROT[L];
                                                if(MELE>1)
                                                    A=A+PELE[L];
                                                if(get(ISPV  ,LS)>0)
                                                {
                                                    for(KVV=1;KVV<=get(ISPV  ,LS);KVV++)
                                                    {
                                                        JI=get (IPVIB , KVV,L);
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        A=A+(double)(JI)*BOLTZ*get(SPVM ,1,KVV,LS);
                                                    }
                                                }
                                                if(get(ISPR ,1,MS)>0)
                                                    A+=PROT[M];
                                                if(MELE>1)
                                                    A=A+PELE[M];
                                                if(get(ISPV  ,MS)>0)
                                                {
                                                    for(KVV=1;KVV<=get(ISPV  ,MS);KVV++)
                                                    {
                                                        JI=get (IPVIB , KVV,M);
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        A=A+(double)(JI)*BOLTZ*get(SPVM ,1,KVV,MS);
                                                    }
                                                }
                                                get (TREACL , 2,LS)=get (TREACL , 2,LS)-1;
                                                get (TREACL , 2,MS)=get (TREACL , 2,MS)-1;
                                                LSI=LS;
                                                MSI=MS;
                                                LS=get(ISPRC ,LS,MS);
                                                get(IPSP ,L)=LS;
                                                //any additional vibrational modes must be set to zero
                                                IVM=get(ISPV  ,LSI);
                                                NMC=get(IPSP ,L);
                                                NVM=get(ISPV  ,NMC);
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        get (IPVIB , KV,L)=0;
                                                    }
                                                }
                                                if(MELE>1)
                                                    PELE[KV]=0.e00;

                                                get(IPCELL  ,M) = -100; //recombining molecule M marked for removal
                                                M=KT; //third body molecule is set as molecule M
                                                MS=KS;
                                                get (TREACG , 2,LS)=get (TREACG , 2,LS)+1;
                                                if(get(ISPR ,1,LS)>0)
                                                {
                                                    PROT[L]=0.e00;
                                                }
                                                if(MELE>1)
                                                    PELE[L]=0.e00;
                                                if(get(ISPV  ,LS)>0)
                                                {
                                                    for(KVV=1;KVV<=get(ISPV  ,LS);KVV++)
                                                    {
                                                        if(get (IPVIB , KVV,L)<0)
                                                        {
                                                            get (IPVIB , KVV,L)=-99999;
                                                        }
                                                        else
                                                        {
                                                            get (IPVIB , KVV,L)=0;
                                                        }
                                                    }
                                                }
                                                if(get(ISPR ,1,MS)>0)
                                                {
                                                    PROT[M]=PROT[KT];
                                                }
                                                if(MELE>1)
                                                    PELE[M]=PELE[KT];
                                                if(get(ISPV  ,MS)>0)
                                                {
                                                    for(KVV=1;KVV<=get(ISPV  ,MS);KVV++)
                                                    {
                                                        get (IPVIB , KVV,M)=get (IPVIB , KVV,KT);
                                                    }
                                                }
                                                ECTOT=A+get(SPVM ,4,1,LS)*BOLTZ ; //the energy added to this collision
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    get(PV  ,KK,L)=VCM[KK];
                                                }
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    VRC[KK]=get(PV  ,KK,L)-get(PV  ,KK,M);
                                                }
                                                VRR=VRC[1]*VRC[1]+VRC[2]*VRC[2]+VRC[3]*VRC[3];
                                                ECT=0.5e00*get (SPM , 1,LS,MS)*VRR*ECTOT;
                                                //set the vibrational energy of the recombined molecule L to enforce detailed balance
                                                IK=-1;
                                                NK=-1;
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                //NTRY=0;
                                                while(IK<0)
                                                {
                                                    // NTRY+=1;
                                                    // if(NTRY>100)
                                                    //   cout<<"NTRY VibEn"<<NTRY;
                                                    NK=NK+1;
                                                    BB=(get (VAR , 8,NN)-get( SPRT ,1,LSI,MSI))*(get (SPRP , 2,LSI,MSI,NK+1)-get (SPRP , 1,LSI,MSI,NK+1))/(get( SPRT ,2,LSI,MSI)-get( SPRT ,1,LSI,MSI))-get (SPRP , 1,LSI,MSI,NK+1);
                                                    if(RANF<BB)
                                                        IK=NK;
                                                }
                                                get (IPVIB , 1,L)=IK;
                                                ECT=ECT-(double)(IK)*BOLTZ*get(SPVM ,1,get(ISPRK ,LSI,MSI),LS);
                                                VRR=2.e00*ECT/get (SPM , 1,LS,MS);
                                                VR=sqrtf(VRR);
                                                RML=get (SPM , 1,LS,MS)/get (SP , 5,MS);
                                                RMM=get (SPM , 1,LS,MS)/get (SP , 5,LS);
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    VCM[KK]=RML*get(PV  ,KK,L)+RMM*get(PV  ,KK,M);
                                                }
                                            }
                                        }
                                        //consider exchange and chain reactions
                                        if(get(NSPEX  ,LS,MS)>0 && IREC==0 && IDISS==0)
                                        {
                                            //possible exchange reaction
                                            //memget(PSF,0.e00,sizeof(*PSF));//PSF=0.e00; //PSF(MMEX) PSF is the probability that this reaction will occur in this collision
                                            for(int i=0;i<MMEX+1;i++)
                                                PSF[i]=0.e00;
                                            
                                            for(JJ=1;JJ<=get(NSPEX  ,LS,MS);JJ++)
                                            {
                                                if(LS==get(ISPEX  ,JJ,1,LS,MS))
                                                {
                                                    K=L; KS=LS;JS=MS;
                                                }
                                                else
                                                {
                                                    K=M; KS=MS; JS=LS;
                                                }
                                                //the pre-collision molecule that splits is K of species KS
                                                if(get(SPEX  ,3,JJ,LS,MS)<0.e00)
                                                    KV=get(ISPEX  ,JJ,5,LS,MS);
                                                if(get(SPEX  ,3,JJ,LS,MS)>0.e00)
                                                {
                                                    KV=get(ISPEX  ,JJ,7,LS,MS);
                                                }
                                                JI=get (IPVIB , KV,K);
                                                if(JI<0)
                                                    JI=-JI;
                                                if(JI==99999)
                                                    JI=0;
                                                ECC=0.5e00*get (SPM , 1,LS,MS)*VRR+(double)(JI)*BOLTZ*get(SPVM ,1,KV,KS);
                                                if(get(SPEX  ,3,JJ,KS,JS)>0.e00)
                                                {
                                                    //reverse exothermic reaction
                                                    PSF[JJ]=(get(SPEX  ,1,JJ,KS,JS)*powf(get (VAR , 8,NN)/273.e00,get(SPEX  ,2,JJ,KS,JS)))*expf(-get(SPEX  ,6,JJ,KS,JS)/(BOLTZ*get (VAR , 8,NN)));
                                                }
                                                else
                                                {
                                                    //forward endothermic reaction
                                                    MAXLEV=ECC/(BOLTZ*get(SPVM ,1,KV,KS));
                                                    EA=fabsf(get(SPEX  ,3,JJ,KS,JS)); //temporarily just the heat of reaction;
                                                    if(ECC>EA)
                                                    {
                                                        //the collision energy must exceed the heat of reaction
                                                        EA=EA+get(SPEX  ,6,JJ,KS,JS); //the activation energy now includes the energy barrier
                                                        DEN=0.e00;
                                                        for(IAX=0;IAX<=MAXLEV;IAX++)
                                                        {
                                                            DEN=DEN+powf((1.e00-(double)(IAX)*BOLTZ*get(SPVM ,1,KV,KS)/ECC),(1.5e00-get (SPM , 3,KS,JS)));
                                                        }
                                                        PSF[JJ]=(double)(get(ISPEX  ,JJ,6,LS,MS))*powf((1.e00-EA/ECC),(1.5e00-get (SPM , 3,KS,JS)))/DEN;
                                                    }
                                                }
                                            }
                                            if(get(NSPEX  ,LS,MS)>1)
                                            {
                                                BB=0.e00;
                                                for(JJ=1;JJ<=get(NSPEX  ,LS,MS);JJ++)
                                                {
                                                    BB=BB+PSF[JJ];
                                                }
                                                //BB is the sum of the probabilities
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                if(BB>RANF)
                                                {
                                                    BB=0.e00;
                                                    IEX=0;
                                                    JJ=0;
                                                    //NTRY=0;
                                                    while(JJ<get(NSPEX  ,LS,MS)&& IEX==0)
                                                    {
                                                        // NTRY=NTRY+1;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout<<"NTRY find IEX"<<NTRY;
                                                        // }
                                                        JJ+=1;
                                                        BB+=PSF[JJ];
                                                        if(BB>RANF)
                                                            IEX=JJ;
                                                    }
                                                }
                                            }
                                            else
                                            {
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                IEX=0;
                                                if(PSF[1]>RANF)
                                                    IEX=1;
                                            }
                                            if(IEX>0)
                                            {
                                                //exchange or chain reaction occurs
                                                JX=get(NEX  ,IEX,LS,MS);
                                                //cout<<"Reaction"<<JX;
                                                TNEX[JX]=TNEX[JX]+1.e00;
                                                //cout<<IEX<<L<<M<<LS<<MS;
                                                get(IPSP ,L)=get(ISPEX  ,IEX,3,LS,MS); //L is now the new molecule that splits
                                                get(IPSP ,M)=get(ISPEX  ,IEX,4,LS,MS);
                                                LSI=LS;
                                                MSI=MS;
                                                //any additional vibrational modes must be set to zero
                                                IVM=get(ISPV  ,LS);
                                                NMC=IPCP[L];
                                                NVM=get(ISPV  ,NMC);
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        get (IPVIB , KV,L)=0;
                                                    }
                                                }
                                                IVM=get(ISPV  ,MS);
                                                NMC=IPCP[M];
                                                NVM=get(ISPV  ,NMC);
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        get (IPVIB , KV,M)=0;
                                                    }
                                                }
                                                //put all pre-collision energies into the relative translational energy and adjust for the reaction energy
                                                ECT=0.5e00*get (SPM , 1,LS,MS)*VRR;
                                                if(get(ISPR ,1,LS)>0)
                                                    ECT=ECT+PROT[L];
                                                if(MELE>1)
                                                    ECT=ECT+PELE[L];
                                                if(get(ISPV  ,LS)>0)
                                                {
                                                    for(KV=1;KV<=get(ISPV  ,LS);KV++)
                                                    {
                                                        JI=get (IPVIB , KV,L);
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        ECT=ECT+(double)(JI)*BOLTZ*get(SPVM ,1,KV,LS);
                                                    }
                                                }
                                                if(get(ISPR ,1,MS)>0)
                                                    ECT=ECT+PROT[M];
                                                if(get(ISPR ,1,MS)) AA = PROT[M] ;
                                                if(MELE>1)
                                                    ECT=ECT+PELE[M];
                                                if(get(ISPV  ,MS)>0)
                                                {
                                                    for(KV=1;KV<=get(ISPV  ,MS);KV++)
                                                    {
                                                        JI=get (IPVIB , KV,M);
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        ECT=ECT+(double)(JI)*BOLTZ*get(SPVM ,1,KV,MS);
                                                    }
                                                }
                                                ECT=ECT+get(SPEX  ,3,IEX,LS,MS);
                                                if(ECT<0.0)
                                                {
                                                    //printf ("-VE ECT %f\n",ECT);
                                                    //printf ("REACTION %d",JJ," BETWEEN %d",LS," & %d\n",MS);
                                                    // cout<<"-VE ECT "<<ECT<<endl;
                                                    // cout<<"REACTION "<<JJ<<" BETWEEN "<<LS<<" "<<MS<<endl;
                                                    //dout
                                                    //cin.get();
                                                    return ;
                                                }
                                                if(get(SPEX  ,3,IEX,LS,MS)<0.e00)
                                                {
                                                    get (TREACL , 3,LS)=get (TREACL , 3,LS)-1;
                                                    get (TREACL , 3,MS)=get (TREACL , 3,MS)-1;
                                                    LS=get(IPSP ,L) ;
                                                    MS=get(IPSP ,M) ;
                                                    get (TREACG , 3,LS)=get (TREACG , 3,LS)+1;
                                                    get (TREACG , 3,MS)=get (TREACG , 3,MS)+1;
                                                }
                                                else
                                                {
                                                    get (TREACL , 4,LS)=get (TREACL , 4,LS)-1;
                                                    get (TREACL , 4,MS)=get (TREACL , 4,MS)-1;
                                                    LS=get(IPSP ,L) ;
                                                    MS=get(IPSP ,M) ;
                                                    get (TREACG , 4,LS)=get (TREACG , 4,LS)+1;
                                                    get (TREACG , 4,MS)=get (TREACG , 4,MS)+1;
                                                }
                                                RML=get (SPM , 1,LS,MS)/get (SP , 5,MS);
                                                RMM=get (SPM , 1,LS,MS)/get (SP , 5,LS);
                                                //calculate the new VRR to match ECT using the new molecular masses
                                                VRR=2.e00*ECT/get (SPM , 1,LS,MS);
                                                if(get(ISPV  ,LS)>0)
                                                {
                                                    for(KV=1;get(ISPV  ,LS);KV++)
                                                    {
                                                        if(get (IPVIB , KV,L)<0)
                                                        {
                                                            get (IPVIB , KV,L)=-99999;
                                                        }
                                                        else
                                                        {
                                                            get (IPVIB , KV,L)=0;
                                                        }
                                                    }
                                                }
                                                if(get(ISPR ,1,LS)>0)
                                                    PROT[L]=0;
                                                if(MELE>1)
                                                    PELE[L]=0.e00;
                                                if(get(ISPV  ,MS)>0)
                                                {
                                                    for(KV=1;get(ISPV  ,MS);KV++)
                                                    {
                                                        if(get (IPVIB , KV,M)<0)
                                                        {
                                                            get (IPVIB , KV,M)=-99999;
                                                        }
                                                        else
                                                        {
                                                            get (IPVIB , KV,M)=0;
                                                        }
                                                    }
                                                }
                                                if(get(ISPR ,1,MS)>0)
                                                    PROT[M]=0;
                                                if(MELE>1)
                                                    PELE[M]=0.e00;
                                                //set vibrational level of product molecule in exothermic reaction to enforce detailed balance
                                                if(get(SPEX  ,3,IEX,LSI,MSI)>0.e00)
                                                {
                                                    //exothermic exchange or chain reaction
                                                    IK=-1; //becomes 0 when the level is chosen
                                                    NK=-1;
                                                    //dout
                                                    //                                                        RANDOM_NUMBER(RANF);
                                                    RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                    //NTRY=0;
                                                    while(IK<0)
                                                    {
                                                        // NTRY=NTRY+1;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout>>"NTRY VibProd"<<NTRY<<endl;
                                                        // }
                                                        NK=NK+1;
                                                        BB=(get (VAR , 8,NN)-get(SPEX  ,4,IEX,LSI,MSI))*  (get (SPREX , 2,IEX,LSI,MSI,NK+1)-get (SPREX , 1,IEX,LSI,MSI,NK+1))/(get(SPEX  ,5,IEX,LSI,MSI)-get(SPEX  ,4,IEX,LSI,MSI))+get (SPREX , 1,IEX,LSI,MSI,NK+1);
                                                        if(RANF<BB)
                                                            IK=NK;
                                                    }
                                                    if(get(NSLEV ,1 , LS)>0)
                                                    {
                                                        IK+=get(NSLEV ,1,LS);
                                                        get(NSLEV ,1,LS)=0;
                                                    }
                                                    KV=get(ISPEX  ,IEX,7,LSI,MSI);
                                                    get (IPVIB , KV,L)=IK;
                                                    EVIB=(double)(IK)*BOLTZ*get(SPVM ,1,KV,LS);
                                                    ECT=ECT-EVIB;
                                                    if(ECT<0.e00)
                                                    {
                                                        //NTRY=0;
                                                        while(ECT<0.e00)
                                                        {
                                                            //NTRY+=1;
                                                            // if(NTRY>100)
                                                            //     cout<<"NTRY ECT<0"<<NTRY<<endl;
                                                            get (IPVIB , KV,L)=get (IPVIB , KV,L)-1;
                                                            get(NSLEV ,1,LS)+=1;
                                                            ECT=ECT+BOLTZ*get(SPVM ,1,KV,LS);
                                                        }
                                                    }
                                                }
                                                else
                                                {
                                                    //for endothermic reaction, select vibration from vib. dist. at macroscopic temperature
                                                    //normal L-B selection would be from the excessively low energy after the endo. reaction
                                                    KV=get(ISPEX  ,IEX,5,LS,MS);
                                                    //dout
                                                    SVIB( LS,get (VAR , 8,NN),IK,KV);
                                                    if(get(NSLEV ,2,LS)>0)
                                                    {
                                                        IK=IK+get(NSLEV ,2,LS);
                                                        get(NSLEV ,2,LS)=0;
                                                    }
                                                    get (IPVIB , KV,L)=IK;
                                                    EVIB=(double)(IK)*BOLTZ*get(SPVM ,1,KV,LS);
                                                    ECT=ECT-EVIB;
                                                    if(ECT<0.e00)
                                                    {
                                                        //NTRY=0;
                                                        while(ECT<0.e00)
                                                        {
                                                            //NTRY+=1;
                                                            get (IPVIB , KV,L)-=1;
                                                            get(NSLEV ,2,LS)+=1;
                                                            ECT=ECT+BOLTZ*get(SPVM ,1,KV,LS);
                                                            // if(NTRY>100)
                                                            // {
                                                            //cout<<"NTRY ECT<0#2"<<NTRY<<endl;
                                                            // get (IPVIB , KV,L]=0;
                                                            //   ECT+=EVIB;
                                                            //   NSLEV[2,LS]=0;
                                                            // }
                                                        }
                                                    }
                                                }
                                                //set rotational energy of molecule L to equilibrium at the macroscopic temperature
                                                SROT( LS,get (VAR , 8,NN),PROT[L]);
                                                if(SLER[LS]>1.e-21)
                                                {
                                                    PROT[L]+=SLER[LS];
                                                    SLER[LS]=1.e-21;
                                                }
                                                ECT-=PROT[L] ;
                                                ABA=PROT[L] ;
                                                if(ECT<0.e00)
                                                {
                                                    //NTRY=0;
                                                    while(ECT<0.e00)
                                                    {
                                                        //NTRY+=1;
                                                        BB=0.5e00*PROT[L];
                                                        SLER[LS]+=BB;
                                                        PROT[L]=BB;
                                                        ECT+=BB;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout<<"NTRY ECT<0#3"<<NTRY<<L<<endl;
                                                        //   ECT+=ABA;
                                                        //   PROT[L]=0;
                                                        //   SLER[LS]=1.e-21;
                                                        // }
                                                    }
                                                }
                                                //calculate the new VRR to match ECT using the new molecular masses
                                                VRR=2.e00*ECT/get (SPM , 1,LS,MS);
                                            }
                                        }
                                    }
                            
                                        //end of reactions other than the deferred dissociation action in the DISSOCIATION subroutine
                                    if(IREC==0 && IDISS==0)
                                    {
                                        //recombined redistribution already made and there is a separate subroutine for dissociation
                                        //Larsen-Borgnakke serial redistribution
                                        ECT=0.5e00*get (SPM , 1,LS,MS)*VRR ;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            if(NSP==1)
                                            {
                                                K=L;KS=LS;JS=MS;
                                            }
                                            else
                                            {
                                                K=M; KS=MS; JS=LS;
                                            }
                                            //now electronic energy for this molecule
                                            if(MELE>1)
                                            {
                                                B=1.e00/get(QELC ,3,1,KS);
                                                //dout
                                                //RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                if(B>RANF)
                                                {
                                                    NPS=0;
                                                    ECC=ECT+PELE[K];
                                                    if(get(NELL  ,KS)==1){
                                                        NPS=get(QELC ,1,1,KS); //number of possible states is at least the degeneracy of the ground state
                                                    }
                                                    if(get(NELL  ,KS)>1)
                                                    {
                                                        for(NEL=1;NEL<=get(NELL  ,KS);NEL++)
                                                        {
                                                            if(ECC>BOLTZ*get(QELC ,2,NEL,KS))
                                                                NPS=NPS+get(QELC ,1,NEL,KS);
                                                        }
                                                        II=0;
                                                        //NTRY=0;
                                                        while(II==0)
                                                        {
                                                            //NTRY+=1;
                                                            // if(NTRY>100)
                                                            //           cout<<"NTRY ElecEn"<<NTRY<<endl;
                                                            //dout
                                                            //                                                                    RANDOM_NUMBER(RANF);
                                                            RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                            NSTATE=ceil(RANF*NPS);//random state, now determine the energy level
                                                            NAS=0;
                                                            NLEVEL=-1;
                                                            for(NEL=1;NEL<=get(NELL  ,KS);NEL++)
                                                            {
                                                                NAS= NAS+get(QELC ,1,NEL,KS);
                                                                if(NSTATE<=NAS && NLEVEL<0)
                                                                    NLEVEL=NEL;
                                                            }
                                                            //dout
                                                            //                                                                    RANDOM_NUMBER(RANF);
                                                            RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                            if((1.e00/(B*get(QELC ,3,NLEVEL,KS)))<RANF)
                                                            {
                                                                II=1;
                                                            }
                                                            else
                                                            {
                                                                if(ECC>BOLTZ*get(QELC ,2,NLEVEL,KS))
                                                                {
                                                                    PROB=powf(1.e00-BOLTZ*get(QELC ,2,NLEVEL,KS)/ECC,(1.5e00-get (SPM , 3,KS,JS)));
                                                                    //dout
                                                                    //                                                                            RANDOM_NUMBER(RANF);
                                                                    RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                    if(PROB>RANF)
                                                                    {
                                                                        II=1;
                                                                        PELE[K]=BOLTZ*get(QELC ,2,NLEVEL,KS);
                                                                    }
                                                                }
                                                            }
                                                        }
                                                        ECT=ECC-PELE[K];
                                                    }
                                                }
                                            }
                                            //now the vibrational energy for this molecule
                                            if(MMVM>0 && IEX==0)
                                            {
                                                if(get(ISPV  ,KS)>0)
                                                {
                                                    for(KV=1;KV<=get(ISPV  ,KS);KV++)
                                                    {
                                                        if(get (IPVIB , KV,K)>=0 && IDISS==0) //do not redistribute to a dissociating molecule marked for removal
                                                        {
                                                            EVIB=(double)(get (IPVIB , KV,K))*BOLTZ*get(SPVM ,1,KV,KS);
                                                            ECC=ECT+EVIB;
                                                            MAXLEV=ECC/(BOLTZ*get(SPVM ,1,KV,KS));
                                                            if(get(SPVM ,3,KV,KS)>0.0)
                                                            {   
                                                                B=get(SPVM ,4,KV,KS)/get(SPVM ,3,KV,KS);
                                                                A=get(SPVM ,4,KV,KS)/get (VAR , 8,NN);
                                                               ZV = powf(A,get (SPM , 3,KS,JS))*powf((get(SPVM ,2,KV,KS)*powf(B,-get (SPM , 3,KS,JS))),((powf(A,0.3333333e00)-1.e00)/(powf(B,0.33333e00)-1.e00)));
                                                               
                                                            }
                                                            else
                                                                ZV=get(SPVM ,2,KV,KS);
                                                            //                                                                    RANDOM_NUMBER(RANF) //dout
                                                            RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                            if(1.e00/ZV>RANF ||IREC==1)
                                                            {
                                                                II=0;
                                                                NSTEP=0;
                                                                while(II==0 && NSTEP<100000)
                                                                {
                                                                    NSTEP+=1;
                                                                    if(NSTEP>99000)
                                                                    {
                                                                        printf("%d %f %d\n",NSTEP,ECC,MAXLEV);
                                                                        //cout<<NSTEP<<" "<<ECC<<" "<<MAXLEV<<endl;
                                                                        //dout
                                                                        return ;
                                                                    }
                                                                    //                                                                            RANDOM_NUMBER(RANF);
                                                                    RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                    IV=RANF*(MAXLEV+0.99999999e00);
                                                                    get (IPVIB , KV,K)=IV;
                                                                    EVIB=(double)(IV)*BOLTZ*get(SPVM ,1,KV,KS);
                                                                    if(EVIB<ECC)
                                                                    {
                                                                        PROB=powf(1.e00-EVIB/ECC,1.5e00-get(SPVM ,3,KS,JS));
                                                                        //PROB is the probability ratio of eqn (3.28)
                                                                        //                                                                                RANDOM_NUMBER(RANF);
                                                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                                        if(PROB>RANF)
                                                                            II=1;
                                                                    }
                                                                }
                                                                ECT=ECC-EVIB;
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                            //now rotation of this molecule
                                            //dout
                                            if(get(ISPR ,1,KS) > 0)
                                            {
                                                if(get(ISPR ,2,KS)==0 && get(ISPR ,2,JS)==0)
                                                {
                                                    B=1.e00/get (SPM , 7,KS,JS);
                                                }
                                                else
                                                    B=1.e00/(get(SPR ,1,KS))+get(SPR ,2,KS)*get (VAR , 8,NN)+get(SPR ,3,KS)*powf(get (VAR , 8,NN),2);
                                                //                                                        RANDOM_NUMBER(RANF);
                                                RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                if(B>RANF|| IREC==1)
                                                {
                                                    ECC=ECT+PROT[K];
                                                    if(get(ISPR ,1,KS)==2)
                                                    {
                                                        //                                                                RANDOM_NUMBER(RANF);
                                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                                        ERM=1.e00-powf(RANF,(1.e00/(2.5e00-get (SPM , 3,KS,JS))));//eqn(5.46)
                                                    }
                                                    else
                                                        LBS( 0.5e00*get(ISPR ,1,KS)-1.e00,1.5e00-get (SPM , 3,KS,JS),ERM);
                                                    PROT[K]=ERM*ECC;
                                                    ECT=ECC-PROT[K];
                                                }
                                            }
                                        }
                                        //adjust VR for the change in energy
                                        VR=sqrtf(2.e00*ECT/get (SPM , 1,LS,MS));
                                    }//end of L-B redistribution
                                    if(fabsf(get (SPM , 8,LS,MS)-1.0)<0.001)
                                    {
                                        //use the VHS logic
                                        //                                                RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*RANF-1.e00;
                                        //B is the cosine of a random elevation angle
                                        A=sqrtf(1.e00-B*B);
                                        VRCP[1]=B*VR;
                                        //                                                RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*RANF;
                                        //C is a random azimuth angle;
                                        VRCP[2]=A*(double)cos(C)*VR;
                                        VRCP[3]=A*(double)sin(C)*VR;
                                    }
                                    else
                                    {
                                        //use the VSS logic
                                        //the VRCP terms do not allow properly for the change in VR - see new book  !STILL TO BE FIXED
                                        VRA=VR/VRI;
                                        //                                                RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*powf(RANF,get (SP , 4,1))-1.e00;
                                        // B is the cosine of the deflection angle for the VSS model
                                        A=sqrtf(1.e00-B*B);
                                        //                                                RANDOM_NUMBER(RANF);
                                        RANF=((double)rand()/(double)RAND_MAX);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*RANF;
                                        OC=(double)cos(C);
                                        SD=(double)sin(C);
                                        D=sqrtf(powf(VRC[2],2)+powf(VRC[3],2));
                                        VRCP[1]=(B*VRC[1]+A*SD*D)*VRA;
                                        VRCP[2]=(B*VRC[2]+A*(VRI*VRC[3]*OC-VRC[1]*VRC[2]*SD)/D)*VRA;
                                        VRCP[3]=(B*VRC[3]+A*(VRI*VRC[2]*OC+VRC[1]*VRC[3]*SD)/D)*VRA;
                                        //the post-collision rel. velocity components are based on eqn (3.18)
                                    }
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        get(PV  ,KK,L)=VCM[KK]+RMM*VRCP[KK];
                                        get(PV  ,KK,M)=VCM[KK]-RMM*VRCP[KK];
                                    }
                                    IPCP[L]=M;
                                    IPCP[M]=L;
                                    //call energy(0,E2)
                                    // !              IF (Dfabs(E2-E1) > 1.D-14) read(*,*)
                                }////collision occurrence
                            }
                            
                            
                        }//separate simplegas / mixture coding
                    }
                }
            }
        }
    //remove any recombined atoms
    
}


void COLLISIONS()
{   
    
    int N=NCCELLS;
    
    d_allocate(N , COLL_TOTCOL) ;
    for(int i=0 ; i<N+2 ; i++)  get (COLL_TOTCOL , i)=0e00 ;
    
    
  
    for(N=1;N<=NCCELLS;N++){
        cuda_collisions(N);
    }
    
    

   

    //std::cout<<"printf: "<< duration <<'\n';
    
    for(N=1;N<=NCCELLS;N++){
        TOTCOL=TOTCOL+get (COLL_TOTCOL , N);
    }
    for(int N=1;N<=NM;N++)
    {
        if(get(IPCELL  ,N)<0)
            REMOVE_MOL(N); 
    }
    return;
} 

void SETXT()
{
    //generate TECPLOT files for displaying an x-t diagram of an unsteady flow
    //this employs ordered data, therefore the cells MUST NOT BE ADAPTED
    //N.B. some custom coding for particular problems
    //
    //
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    //OUTPUT output;
    //
    
    // IMPLICIT NONE
    //
    int N,M,IOUT;
    double A,C;
    double *VALINT;
    // REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:) :: VALINT
    //
    //VALINT(N,M) the interpolated values at sampling cell M boundaries and extrapolated values at boundaries
    //    N=1 distance
    //    N=2 time
    //    N=3 number density
    //    N=4 radial velocity
    //    N=5 pressure (nkT)
    //    N=6 temperature
    //    N=7 h2o fraction (Sec. 7.9 only)
    //
    //the variables in VALINT may be altered for particular problems
    //
    d_allocate(7 , NCELLS+2 , VALINT) ;
    
    
    // ALLOCATE (VALINT(6,NCELLS+1),STAT=ERROR)
    //
    //777 FORMAT(12G14.6)
    //24[]
    
    //Internal options
    IOUT=0;    //0 for dimensioned output, 1 for non-dimensional output
    //
    A=1.e00;   //dt/dt for selection of v velocity component in TECPLOT to draw particle paths as "streamlines"
    //
    if(FTIME < 0.5e00*DTM){
        //Headings and zero time record
        //        IF (ERROR /= 0) THEN
        //        WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR VALINT ARRAY',ERROR
        //        ENDIF
        NLINE=1;
        file_9<< "J in tecplot file = "<<NLINE*(NCELLS+1)<<endl;
        //  WRITE (18,*) 'VARIABLES = "Distance","Time","n","u","p","T","H2O","A"'   //for combustion wave output(Sec. 7.9)
        
        file_18<<"VARIABLES = 'Distance','Time','n','u','p','T','A' "<<endl;
        file_18<<"ZONE I= "<<NCELLS+1<<", J=  (set to number of output intervals+1), F=POINT"<<endl;
        //
        for(N=1;N<=NCELLS+1;N++){
            get (VALINT , 1,N)=XB[1]+(N-1)*DDIV;    //distance
            get (VALINT , 1,N)=get (VALINT , 1,N);         //time
            get (VALINT , 2,N)=0.0;
            get (VALINT , 3,N)=FND[1];
            get (VALINT , 4,N)=0;
            get (VALINT , 5,N)=FND[1]*BOLTZ*FTMP[1];
            get (VALINT , 6,N)=FTMP[1];
            //   VALINT(7,N)=FSP(6,1)   //FSP(6 for combustion wave
            if((get (VALINT , 1,N) > XS) && (ISECS == 1)){
                get (VALINT , 3,N)=FND[2];
                get (VALINT , 5,N)=FND[2]*BOLTZ*FTMP[2];
                get (VALINT , 6,N)=FTMP[2];
                //      VALINT(7,N)=FSP(6,2)
            }
            if(IOUT == 1){
                get (VALINT , 3,N)=1.e00;
                get (VALINT , 5,N)=1.e00;
                get (VALINT , 6,N)=1.e00;
            }
            for(M=1;M<=6;M++)
                file_18<<get (VALINT , M,N)<<"\t";//WRITE (18,777) (VALINT(M,N),M=1,6),A
            file_18<<A<<endl;
        }
    }
    else{
        NLINE=NLINE+1;
        cout<<"J in tecplot file = "<<NLINE<<endl;
        if(IVB == 0) C=DDIV;
        if(IVB == 1) C=(XB[2]+VELOB*FTIME-XB[1])/double(NDIV);
        for(N=1;N<=NCELLS+1;N++){
            get (VALINT , 1,N)=XB[1]+(N-1)*C;
            get (VALINT , 2,N)=FTIME;
            if((N > 1) && (N < NCELLS+1)){
                get (VALINT , 3,N)=0.5e00*(get (VAR , 3,N)+get (VAR , 3,N-1));
                get (VALINT , 4,N)=0.5e00*(get (VAR , 5,N)+get (VAR , 5,N-1));
                get (VALINT , 5,N)=0.5e00*(get (VAR , 18,N)+get (VAR , 18,N-1));
                get (VALINT , 6,N)=0.5e00*(get (VAR , 11,N)+get (VAR , 11,N-1));
                //     VALINT(7,N)=0.5D00*(VARSP(1,N,6)+VARSP(1,N-1,6))   //H2O fraction for Sec 7.9
            }
        }
        for(N=3;N<=6;N++)
            get (VALINT , N,1)=0.5e00*(3.e00*get (VALINT , N,2)-get (VALINT , N,3));
        
        //
        for(N=3;N<=6;N++)
            get (VALINT , N,NCELLS+1)=0.5e00*(3.e00*get (VALINT , N,NCELLS)-get (VALINT , N,NCELLS-1));
        
        //
        for(N=1;N<=NCELLS+1;N++){
            if(IOUT == 1){
                get (VALINT , 1,N)=(get (VALINT , 1,N)-XB[1])/(XB[2]-XB[1]);
                get (VALINT , 2,N)=get (VALINT , 2,N)/TNORM;
                get (VALINT , 3,N)=get (VALINT , 3,N)/FND[1];
                get (VALINT , 4,N)=get (VALINT , 4,N)/VMPM;
                get (VALINT , 5,N)=get (VALINT , 5,N)/(FND[1]*BOLTZ*FTMP[1]);
                get (VALINT , 6,N)=get (VALINT , 6,N)/FTMP[1];
            }
            for(M=1;M<=6;M++)
                file_18<<get (VALINT , M,N)<<"\t";//WRITE (18,777) (get (VALINT , M,N),M=1,6),A       //
            file_18<<A<<endl;
        }
    }
    //
    return;
}